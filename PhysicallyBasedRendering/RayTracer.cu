#include "hip/hip_runtime.h"
﻿#include "RayTracer.cuh"
#include "Octree.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm\glm.hpp>
#include <glm\gtx\component_wise.hpp>
#include <stdio.h>
#include <glm\gtc\matrix_transform.hpp>
#include <hip/hip_math_constants.h>
#include <math.h>
#include <thrust\device_vector.h>
#include <queue>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <ctime>
#include <stdio.h>


texture<float4, 2, hipReadModeElementType> albedoTex;
texture<float4, 2, hipReadModeElementType> normalTex;
texture<float4, 2, hipReadModeElementType> aoTex;
texture<float4, 2, hipReadModeElementType> metallicTex;
texture<float4, 2, hipReadModeElementType> roughnessTex;

texture<float4, 2, hipReadModeElementType> backgroundTex;



const int WINDOW_HEIGHT = 1024;
const int WINDOW_WIDTH = 1024;

const int RAY_X_NUM = 32;
const int RAY_Y_NUM = 32;

const int QUEUE_SIZE = 32;

const int DEPTH = 2;

const int SAMPLE_NUM = 1;

using std::cout;
using std::endl;
using std::max;
using std::min;

// TODO LIST
// 1. 에너지 보존 for reflect and refract 
// 4. marching cube로 나온 fluid가 뒷면이 culling 되어 있는 문제가 있음
// ggx distribution이라고 외우자
__device__ float DistributionGGX(vec3 N, vec3 H, float roughness)
{
	float a = roughness * roughness;
	float a2 = a * a;
	float NdotH = max(dot(N, H), 0.0f);
	float NdotH2 = NdotH * NdotH;

	float nominator = a2;
	float denominator = (NdotH2 * (a2 - 1.0) + 1.0);
	denominator = glm::pi<float>() * denominator * denominator;

	return nominator / denominator;
}

__device__ float GeometrySchlickGGX(float NdotV, float roughness)
{
	float r = (roughness + 1.0);
	float k = (r * r) / 8.0;

	float nominator = NdotV;
	float denominator = NdotV * (1.0 - k) + k;

	return nominator / denominator;
}

// smith geometry라고 외우자
// geometry shadowing 빛이 어떤 표면으로 갈 때 다른 표면에 막혀 가지 못하는 경우
// geometry obstruction 빛이 어떤 표면에서 눈으로 갈 때 다른 표면에 막혀 가지 못하는 경우
// 이 두가지를 모두 고려해야 해서 ggx1 * ggx2
// 0.8이 안 막히고, 0.8이 안 막힌다면 결국은 0.8 * 0.8
__device__ float GeometrySmith(vec3 N, vec3 V, vec3 L, float roughness)
{
	float NdotV = max(dot(N, V), 0.0f);
	float NdotL = max(dot(N, L), 0.0f);

	float ggx2 = GeometrySchlickGGX(NdotV, roughness);
	float ggx1 = GeometrySchlickGGX(NdotL, roughness);

	return ggx1 * ggx2;
}

// cosTheta가 작을 수록 큰 값이 들어간다
// 즉 90도에 가까운 곳에서 볼 수록 빛이 쎄진다는 것이다.
// 90이면 그냥 1임
// 각도가 높아지면 점점 약해지고 F0값에 가까워짐
__device__ vec3 fresnelSchlick(float cosTheta, vec3 F0)
{
	return F0 + (1.0f - F0) * pow(1.0f - cosTheta, 5.0f);
}

__device__ vec3 calculateEta(float refractiveIndex)
{
	return vec3(powf(1.0f - (1.0f / refractiveIndex), 2.0f) / powf(1.0f + (1.0f / refractiveIndex), 2.0f));
}

__device__ vec3 Interpolation(Triangle triangle, vec3 position, vec3& N, vec2& uv)
{
	vec3 v0 = triangle.v1 - triangle.v0;
	vec3 v1 = triangle.v2 - triangle.v0;
	vec3 v2 = position - triangle.v0;

	float d00 = dot(v0, v0);
	float d01 = dot(v0, v1);
	float d11 = dot(v1, v1);
	float d20 = dot(v2, v0);
	float d21 = dot(v2, v1);
	float denom = d00*d11 - d01*d01;

	float v = (d11*d20 - d01*d21) / denom;
	float w = (d00*d21 - d01*d20) / denom;
	float u = 1.0f - v - w;

	N = u * triangle.v0normal + v * triangle.v1normal + w * triangle.v2normal;
	uv = u * triangle.v0uv + v * triangle.v1uv + w * triangle.v2uv;
}

// ray와 sphere가 intersect하는지 검사하는 함수
__device__ bool RaySphereIntersect(Ray ray, Sphere sphere, float& dist)
{
	glm::vec3 s = ray.origin - sphere.origin;

	float a = dot(ray.dir, ray.dir);
	float bPrime = dot(s, ray.dir);
	float c = dot(s, s) - sphere.radius * sphere.radius;

	float D = bPrime * bPrime - a * c;
	if (D >= 0 && bPrime <= 0)
	{
		float t1 = (-bPrime + sqrt(D)) / a;
		float t2 = (-bPrime - sqrt(D)) / a;
		dist = t1 > t2 ? t2 : t1;
		return dist > 0.0001f;
	}
	else
		return false;
}

// ray와 triangle이 intersect하는지 검사하는 함수
__device__ bool RayTriangleIntersect(Ray ray, Triangle triangle, float& dist)
{
	glm::vec3 v0v1 = triangle.v1 - triangle.v0;
	glm::vec3 v0v2 = triangle.v2 - triangle.v0;
	glm::vec3 pvec = glm::cross(ray.dir, v0v2);

	float det = dot(v0v1, pvec);

	// back face culling
	if (det < 0.001f)
		return false;

	/*if (fabsf(det) < 0.01f)
		return false;*/

	float invDet = 1 / det;

	glm::vec3 tvec = ray.origin - triangle.v0;
	float u = glm::dot(tvec, pvec) * invDet;
	if (u < 0 || u > 1)
		return false;

	glm::vec3 qvec = cross(tvec, v0v1);
	float v = dot(ray.dir, qvec) * invDet;
	if (v < 0 || u + v > 1)
		return false;

	dist = dot(v0v2, qvec) * invDet;

	return dist > 0.001f;
}

//bool RayPlaneIntersect(Ray ray, vec4 plane, float& t)
//{
//	t = -dot(plane, vec4(ray.origin, 1.0)) / glm::dot(glm::vec3(plane), ray.dir);
//	return t > 0.0;
//}
//
//bool RayRectIntersect(Ray ray, Rect rect, float& t)
//{
//	bool intersect = RayPlaneIntersect(ray, rect.plane, t);
//	if (intersect)
//	{
//		vec3 pos = ray.origin + ray.dir*t;
//		vec3 lpos = pos - rect.center;
//
//		float x = dot(lpos, rect.dirx);
//		float y = dot(lpos, rect.diry);
//
//		if (abs(x) > rect.halfx || abs(y) > rect.halfy)
//			intersect = false;
//	}
//
//	return intersect;
//}

__device__ bool RayAABBIntersect(Ray ray, AABB box)
{
	float tmin, tmax, tymin, tymax, tzmin, tzmax;

	glm::vec3 invdir = 1.0f / ray.dir;
	int sign[3];
	sign[0] = invdir.x < 0;
	sign[1] = invdir.y < 0;
	sign[2] = invdir.z < 0;

	tmin = (box.bounds[sign[0]].x - ray.origin.x) * invdir.x;
	tmax = (box.bounds[1 - sign[0]].x - ray.origin.x) * invdir.x;
	tymin = (box.bounds[sign[1]].y - ray.origin.y) * invdir.y;
	tymax = (box.bounds[1 - sign[1]].y - ray.origin.y) * invdir.y;

	if ((tmin > tymax) || (tymin > tmax))
		return false;

	if (tymin > tmin)
		tmin = tymin;
	if (tymax < tmax)
		tmax = tymax;

	tzmin = (box.bounds[sign[2]].z - ray.origin.z) * invdir.z;
	tzmax = (box.bounds[1 - sign[2]].z - ray.origin.z) * invdir.z;

	if ((tmin > tzmax) || (tzmin > tmax))
		return false;

	return true;
}

__device__ bool RayAABBsIntersect(Ray ray, AABB* boxes, int boxNum)
{
	bool isIntersect = false;

	for (int it = 0; it < boxNum; it++)
	{
		if (RayAABBIntersect(ray, boxes[it]))
			isIntersect = true;
	}
	return isIntersect;
}


__device__ float RayTraversal(OctreeNode* root, Ray ray, float& minDist)
{
	if (root == nullptr)
		return false;

	if (ray.dir.x < 0)
	{
		ray.origin.x = root->bnd.bounds[0].x + root->bnd.bounds[1].x - ray.origin.x;
		ray.dir.x = -ray.dir.x;
	}
	if (ray.dir.y < 0)
	{
		ray.origin.y = root->bnd.bounds[0].y + root->bnd.bounds[1].y - ray.origin.y;
		ray.dir.y = -ray.dir.y;
	}
	if (ray.dir.z < 0)
	{
		ray.origin.z = root->bnd.bounds[0].z + root->bnd.bounds[1].z - ray.origin.z;
		ray.dir.z = -ray.dir.z;
	}

	double divx = 1 / ray.dir.x;
	double divy = 1 / ray.dir.y;
	double divz = 1 / ray.dir.z;

	double tx0 = (root->bnd.bounds[0].x - ray.origin.x) * divx;
	double tx1 = (root->bnd.bounds[1].x - ray.origin.x) * divx;
	double ty0 = (root->bnd.bounds[0].y - ray.origin.y) * divy;
	double ty1 = (root->bnd.bounds[1].y - ray.origin.y) * divy;
	double tz0 = (root->bnd.bounds[0].z - ray.origin.z) * divz;
	double tz1 = (root->bnd.bounds[1].z - ray.origin.z) * divz;

	float tmin = max(max(tx0, ty0), tz0);
	float tmax = min(min(tx1, ty1), tz1);

	if (tmin <= tmax)
	{
		//return true;

		if (tmin < minDist)
			return true;
		else
			return false;
	}
	else
		return false;

}

__device__ float KDRayTraversal(gpukdtreeNode root, Ray ray)
{


	if (ray.dir.x < 0)
	{
		ray.origin.x = root.nodeAABB.bounds[0].x + root.nodeAABB.bounds[1].x - ray.origin.x;
		ray.dir.x = -ray.dir.x;
	}
	if (ray.dir.y < 0)
	{
		ray.origin.y = root.nodeAABB.bounds[0].y + root.nodeAABB.bounds[1].y - ray.origin.y;
		ray.dir.y = -ray.dir.y;
	}
	if (ray.dir.z < 0)
	{
		ray.origin.z = root.nodeAABB.bounds[0].z + root.nodeAABB.bounds[1].z - ray.origin.z;
		ray.dir.z = -ray.dir.z;
	}

	double divx = 1 / ray.dir.x;
	double divy = 1 / ray.dir.y;
	double divz = 1 / ray.dir.z;

	double tx0 = (root.nodeAABB.bounds[0].x - ray.origin.x) * divx;
	double tx1 = (root.nodeAABB.bounds[1].x - ray.origin.x) * divx;
	double ty0 = (root.nodeAABB.bounds[0].y - ray.origin.y) * divy;
	double ty1 = (root.nodeAABB.bounds[1].y - ray.origin.y) * divy;
	double tz0 = (root.nodeAABB.bounds[0].z - ray.origin.z) * divz;
	double tz1 = (root.nodeAABB.bounds[1].z - ray.origin.z) * divz;

	float tmin = max(max(tx0, ty0), tz0);
	float tmax = min(min(tx1, ty1), tz1);

	if (tmin <= tmax)
	{
		return true;
	}
	else
		return false;

}

__device__ void RayTreeTraversal(OctreeNode* root, 
								Ray ray, 
								int& minIdx, 
								float& tmpDist, 
								Triangle* triangles, 
								const float& rayThreshold, 
								float& minDist)
{
	//// recursive
	//if (RayTraversal(root, ray))
	//{
	//	//int a = 0;

	//	if (root->children[0] != nullptr)
	//	{
	//		for (int i = 0; i < 8; i++)
	//		{
	//			if (root->children[i] != nullptr)
	//			{
	//				RayTreeTraversal(root->children[i], ray, minIdx, tmpDist, triangles, rayThreshold, minDist);
	//				//a++;
	//			}
	//		}
	//	}

	//	else
	//	{
	//		for (int i = 0; i < root->triangleIdx.size(); i++)
	//		{
	//			//idx->push_back(newIdx.operator[](i));
	//			if (RayTriangleIntersect(ray, triangles[root->triangleIdx.operator[](i)], tmpDist))
	//			{
	//				if (tmpDist > rayThreshold && tmpDist < minDist)
	//				{
	//					minDist = tmpDist;
	//					minIdx = root->triangleIdx.operator[](i);
	//				}
	//			}
	//		}
	//		
	//	}
	//}
	//return;

	OctreeNode* stack[64];
	OctreeNode** stackPtr = stack;
	*stackPtr++ = NULL;


	OctreeNode* node = root;
	do
	{
		OctreeNode* children[8];
		for (int i = 0; i < 8; i++)
			children[i] = node->children[i];

		bool intersect[8];
		//float dist[8];
		for (int i = 0; i < 8; i++)
			intersect[i] = RayTraversal(children[i], ray, minDist);

		for (int i = 0; i < 8; i++)
			if (intersect[i] && children[i]->children[0] == nullptr)
			{
				for (int j = 0; j < children[i]->triangleIdx.size(); j++)
				{
					if (RayTriangleIntersect(ray, triangles[children[i]->triangleIdx[j]], tmpDist))
					{
						if (tmpDist > rayThreshold && tmpDist < minDist)
						{
							minDist = tmpDist;
							minIdx = children[i]->triangleIdx.operator[](j);
						}
					}	
				}
			}
		


		bool traverse[8];
		for (int i = 0; i < 8; i++)
			traverse[i] = intersect[i] && children[i]->children[0] != nullptr;


		if (!traverse[0] && !traverse[1] && !traverse[2] && !traverse[3] && !traverse[4] && !traverse[5] && !traverse[6] && !traverse[7])
			node = *--stackPtr;
		else
		{
			int a = 0;
			for (int i = 0; i < 8; i++)
			{
				if (traverse[i])
				{
					node = children[i];
					a = i;
					break;
				}
			}

			for (int i = a+1; i < 8; i++)
			{
				if (traverse[i])
				{
					*stackPtr++ = children[i];
				}
			}
		}
	
	} while (node != NULL);
}

__device__ void RayKDTreeTraversal(gpukdtree* root,
									Ray ray,
									int& minIdx,
									float& tmpDist,
									Triangle* triangles,
									const float& rayThreshold,
									float& minDist)
{
	int currentid, leftid, rightid, cid;
	DeviceStack<int> treestack;
	treestack.push(0);
	while (!treestack.empty())
	{
		currentid = treestack.pop();

		//test node intersection
		if (KDRayTraversal(root->nodes.data[currentid], ray)) {
			leftid = root->nodes.data[currentid].leftChild;
			rightid = root->nodes.data[currentid].rightChild;
			//// leaf node
			if (leftid == -1 && rightid == -1) {
				/*if (dkdtree::Intersect_nodeTriangles_Ray(ray, currentid, tmpDist, cid, root->nodes.data, triangles, root->triangleNodeAssociation.data)) {
					if (tmpDist<minDist) {
						minDist = tmpDist;
						minIdx = cid;
					}
				}*/
				continue;
			}
			// middle node
			if (leftid != -1)
				treestack.push(leftid);
			if (rightid != -1)
				treestack.push(rightid);
		}
	}

}

//ray의 원점과 가장 가까운 곳에서 intersect하는 triangle의 id를 가져오는 함수
//octree 사용
__device__ int OTFindNearestTriangleIdx(Ray ray, Triangle* triangles, OctreeNode* root, float& dist)
{
	const float rayThreshold = 0.01f;
	float minDist = 99999.0f;
	int minIdx = -1;
	float tmpDist;

	RayTreeTraversal(root, ray, minIdx, tmpDist, triangles, rayThreshold, minDist);	//	전체 삼각형 중 해당 ray가 지나가는 node에 있는 것만 골라낸다.

	dist = minDist;
	return minIdx;
}

__device__ int KDFindNearestTriangleIdx(Ray ray, Triangle* triangles, gpukdtree* root, float& dist)
{
	const float rayThreshold = 0.01f;
	float minDist = 99999.0f;
	int minIdx = -1;
	float tmpDist;

	RayKDTreeTraversal(root, ray, minIdx, tmpDist, triangles, rayThreshold, minDist);	//	전체 삼각형 중 해당 ray가 지나가는 node에 있는 것만 골라낸다.

	dist = minDist;
	return minIdx;
}


 __device__ int findnearesttriangleidx(Ray ray, Triangle* triangles, int trianglenum, float& dist)
{

	const float raythreshold = 0.01f;
	float mindist = 9999999.0f;
	int minidx = -1;
	float tmpdist;

	for (int i = 0; i < trianglenum; ++i)
	{
		if (RayTriangleIntersect(ray, triangles[i], tmpdist))
		{
			if (tmpdist > raythreshold && tmpdist < mindist)
			{
				mindist = tmpdist;
				minidx = i;
			}
		}
	}

	dist = mindist;
	return minidx;
}

// ray의 원점과 가장 가까운 곳에서 intersect하는 sphere의 id를 가져오는 함수
__device__ int FindNearestSphereIdx(Ray ray, Sphere* spheres, int sphereNum, float& dist)
{
	const float rayThreshold = 0.0001f;
	float minDist = 999999.0f;
	int minIdx = -1;
	float tmpDist;

	for (int i = 0; i < sphereNum; ++i)
	{
		if (RaySphereIntersect(ray, spheres[i], tmpDist))
		{
			if (tmpDist > rayThreshold && tmpDist < minDist)
			{
				minDist = tmpDist;
				minIdx = i;
			}
		}
	}

	dist = minDist;
	return minIdx;
}

// window의 픽셀의 위치가 각각 x, y로 입력됨
__device__ Ray GenerateCameraRay(int y, int x, glm::mat4 cameraModelMatrix, int rayX, int rayY)
{
	Ray ray;

	// 각 픽셀의 중앙을 가르키는 값 생성, 0~1의 값으로 Normalizing
	// antialiasing
	float NDCy = (y + 0.33333f + 0.33333f*rayY) / WINDOW_HEIGHT;
	float NDCx = (x + 0.33333f + 0.33333f*rayX) / WINDOW_WIDTH;

	// no antialiasing
	/*float NDCy = (y + 0.5f) / WINDOW_HEIGHT;
	float NDCx = (x + 0.5f) / WINDOW_WIDTH;*/

	// window 종횡비
	float aspectRatio = WINDOW_WIDTH / WINDOW_HEIGHT;

	// 시야각 설정
	float fov = 45.0f;

	// unProject
	float xx = (NDCx * 2.0f - 1.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f) * aspectRatio;
	float yy = (NDCy * 2.0f - 1.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f);

	// world space에서의 ray 정보를 계산
	ray.origin = glm::vec3(cameraModelMatrix * glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
	ray.dir = normalize(vec3(cameraModelMatrix * vec4(glm::vec3(xx, yy, -1.0), 0.0f)));
	ray.decay = 1.0f;
	ray.depth = 1;

	// 만들어진 ray를 return
	return ray;
}

__device__ void Enqueue(Ray* rayQueue, Ray ray, int& rear)
{
	rayQueue[rear] = ray;
	rear = (rear + 1) % QUEUE_SIZE;
}

__device__ void Dequeue(Ray* rayQueue, int& front)
{
	front = (front + 1) % QUEUE_SIZE;
}

__device__ Ray GetQueueFront(Ray* rayQueue, const int front)
{
	return rayQueue[front];
}

__device__ bool IsQueueEmpty(const int front, const int rear)
{
	return front == rear;
}

__device__ bool IsLight(const vec3 emission)
{
	return emission.x > 0.0f || emission.y > 0.0f || emission.z > 0.0f;
}

__device__ bool IsLighted(
	vec3 hitPoint,
	Light light,
	Triangle* triangles,
	const int triangleNum,
	const int nearestTriangleIdx,
	Sphere* spheres,
	const int sphereNum,
	const int nearestSphereIdx)
{
	// shadow ray 생성, origin은 hit point, 방향은 hit point부터 광원까지의 방향
	Ray shadowRay;
	shadowRay.origin = hitPoint;
	shadowRay.dir = normalize(light.pos - hitPoint);
	float distance = glm::distance(light.pos, hitPoint);

	float distToTriangle;

	for (int t_i = 0; t_i < triangleNum; ++t_i)
	{
		// 처음 hit한 triangle은 제외
		if (nearestTriangleIdx != t_i)
		{
			// shadow
			if (RayTriangleIntersect(shadowRay, triangles[t_i], distToTriangle))
			{
				// 앞쪽의 dir만 봄, 매우 가까운 곳은 그림자 아님
				if (distToTriangle > 0.01f && distToTriangle < glm::distance(light.pos, hitPoint))
				{
					return false;
				}
			}
		}
	}

	float distToSphere;

	for (int s_i = 0; s_i < sphereNum; ++s_i)
	{
		// 광원은 0임, 광원을 제외한 경우에만 그림자 생김
		if (nearestSphereIdx != s_i && s_i != 0)
		{
			if (RaySphereIntersect(shadowRay, spheres[s_i], distToSphere))
			{
				// 앞쪽의 dir만 봄, 매우 가까운 곳은 그림자 아님
				if (distToSphere > 0.01f && distToSphere < glm::distance(light.pos, hitPoint))
				{
					return false;
				}
			}
		}
	}

	return true;
}

// ray가 hit 했다면 true를 리턴하고 hit한 곳의 정보를 가져오는 함수
__device__ bool GetHitPointInfo(
	Ray nowRay,
	Triangle* triangles,
	int triangleNum,
	int& nearestTriangleIdx,
	Sphere* spheres,
	int sphereNum,
	int& nearestSphereIdx,
	vec3& hitPoint, 
	int& materialId, 
	vec3& N,
	vec2& uv,
	OctreeNode* root,
	gpukdtree* kdroot)
{
	float distToTriangle, distToSphere, distToAreaLight = 0.0f;
	
	//옥트리
	//nearestTriangleIdx = KDFindNearestTriangleIdx(nowRay, triangles, kdroot, distToTriangle);
	nearestTriangleIdx = OTFindNearestTriangleIdx(nowRay, triangles, root, distToTriangle);
	//nearestTriangleIdx = FindNearestTriangleIdx(nowRay, triangles, triangleNum, distToTriangle);
	nearestSphereIdx = FindNearestSphereIdx(nowRay, spheres, sphereNum, distToSphere);

	// 아무곳도 intersect를 못했다거나 뒤쪽에 있다면
	if ((nearestTriangleIdx == -1 || distToTriangle < 0.0f) &&
		(nearestSphereIdx == -1 || distToSphere < 0.0f) &&
		(distToAreaLight <= 0.0f))
		return false;

	if (distToSphere > distToTriangle)
	{
		Triangle nearestTriangle = triangles[nearestTriangleIdx];
		hitPoint = nowRay.origin + nowRay.dir * distToTriangle;
		materialId = nearestTriangle.materialId;
		Interpolation(nearestTriangle, hitPoint, N, uv);
	}
	else
	{
		Sphere nearestSphere = spheres[nearestSphereIdx];
		hitPoint = nowRay.origin + nowRay.dir * distToSphere;
		materialId = nearestSphere.materialId;
		N = glm::normalize(hitPoint - nearestSphere.origin);
		// no uv...
	}

	return true;
}


__device__ vec4 RayTraceColor(
	Ray ray,
	int rayIndex,
	Ray* rayQueue,
	Triangle* triangles,
	int triangleNum,
	Sphere* spheres,
	int sphereNum,
	Light* lights,
	int lightNum,
	Material* materials,
	int matNum,
	float* randomNums,
	int depth,
	OctreeNode* root,
	gpukdtree* kdroot)
{
	vec3 sumLo = vec3(0.0f, 0.0f, 0.0f);
	int front = 0, rear = 0;

	Enqueue(rayQueue, ray, rear);

	vec3 V = -ray.dir;

	while (!IsQueueEmpty(front, rear))
	{
		Ray nowRay;
		nowRay = GetQueueFront(rayQueue, front);
		Dequeue(rayQueue, front);

		vec3 hitPoint = glm::vec3(0.0f);
		// hit한 object의 material id
		int materialId = 0;
		// normal vector
		vec3 N = glm::vec3(0.0f);
		vec2 uv = glm::vec2(0.0f);
		int nearestTriangleIdx = 0;
		int nearestSphereIdx = 0;

		// hit point의 정보를 가져옴
		if (GetHitPointInfo(
			nowRay,
			triangles,
			triangleNum,
			nearestTriangleIdx,
			spheres,
			sphereNum,
			nearestSphereIdx,
			hitPoint,
			materialId,
			N,
			uv,
			root,
			kdroot))
		{

			// ∫Ω(kd c / π + ks DFG / 4(ωo⋅n)(ωi⋅n)) Li(p,ωi) n⋅ωi dωi
			// radiance * (1.0f * textureColor/pi + 0.0f) * lightcolor * NdotL
			vec3 albedo;
			vec3 emission;
			vec3 F0;
			float4 texNormal;
			float ao;
			float metallic;
			float roughness;

			vec3 kS;
			vec3 kD;

			// sphere
			if (materials[materialId].texId == 0)
			{
				float4 texRGBA;
				texRGBA = tex2D(albedoTex, uv.x, uv.y);
				albedo = glm::pow(glm::vec3(texRGBA.x, texRGBA.y, texRGBA.z), vec3(2.2));

				texNormal = tex2D(normalTex, uv.x, uv.y);
				ao = tex2D(aoTex, uv.x, uv.y).x;
				metallic = tex2D(metallicTex, uv.x, uv.y).x;
				roughness = tex2D(roughnessTex, uv.x, uv.y).x;

				glm::vec3 texNormalVec = glm::vec3(
					texNormal.x * 2.0f - 1.0f,
					texNormal.y * 2.0f - 1.0f,
					texNormal.z * 2.0f - 1.0f);

				glm::mat3 TBN = glm::mat3(
					triangles[nearestTriangleIdx].tangent,
					triangles[nearestTriangleIdx].bitangent,
					N);

				// TBN의 inverse
				N = glm::normalize(texNormalVec);

				N = TBN * N;
			}
			// plane
			else if (materials[materialId].texId == 1)
			{
				float4 texRGBA;
				texRGBA = tex2D(backgroundTex, uv.x, uv.y);
				albedo = glm::pow(glm::vec3(texRGBA.x, texRGBA.y, texRGBA.z), vec3(2.2));

				ao = materials[materialId].ambient;
				metallic = materials[materialId].metallic;
				roughness = materials[materialId].roughness;
				emission = materials[materialId].emission;
			}
			// fluid
			else
			{
				albedo = materials[materialId].albedo;
				ao = materials[materialId].ambient;
				metallic = materials[materialId].metallic;
				roughness = materials[materialId].roughness;
				emission = materials[materialId].emission;
			}

			// fluid라면
			if (materials[materialId].refractiveIndex != 0.0f)
			{
				F0 = calculateEta(materials[materialId].refractiveIndex);
			}
			else
			{
				// metallic이면 F0가 큼, 아니면 작음
				F0 = glm::mix(vec3(0.04f), albedo, metallic);
			}

			vec3 Lo = vec3(0.0f);
			for (int k = 0; k < lightNum; k++)
			{
				vec3 L = glm::normalize(lights[k].pos - hitPoint);
				vec3 H = glm::normalize(V + L);

				float distance = glm::distance(lights[k].pos, hitPoint);
				float attenuation = 1.0 / (distance*distance);

				vec3 radiance = lights[k].color * attenuation;

				float NDF = DistributionGGX(N, H, roughness);
				float G = GeometrySmith(N, V, L, roughness);
				vec3 F = fresnelSchlick(glm::max(glm::dot(H, V), 0.0f), F0);

				vec3 nominator = NDF * G * F;
				float denominator = 4 * glm::max(glm::dot(N, V), 0.0f) * glm::max(glm::dot(N, L), 0.0f) + 0.001f;
				vec3 specular = nominator / denominator;

				kS = F;

				kD = vec3(1.0) - kS;
				kD *= (1.0f - metallic);

				vec3 diffuse = kD * albedo / glm::pi<float>();

				float NdotL = glm::clamp(glm::dot(N, L), 0.0f, 1.0f);

				if (!IsLighted(hitPoint, lights[k], triangles, triangleNum, nearestTriangleIdx,
					spheres, sphereNum, nearestSphereIdx))
				{
					// brdf * radiance * NdotL
					Lo += (diffuse + specular) * radiance * NdotL * 0.1f;
				}
				else
				{
					// brdf * radiance * NdotL
					Lo += (diffuse + specular) * radiance * NdotL;
				}
			}

			vec3 ambient = vec3(0.03) * albedo * ao;

			// Light Sampling
			if (nowRay.depth == 1)
			{
				if (IsLight(emission))
				{
					sumLo += emission;
				}
				else
				{
					sumLo += (ambient + Lo) * nowRay.decay;
				}
			}
			else
			{
				float distance = glm::distance(hitPoint, nowRay.origin);
				float attenuation = 1.0f / (distance * distance);
				sumLo += emission * attenuation * nowRay.decay / (float)SAMPLE_NUM;
			}

			// Path Tracing, BRDF Sampling
			// 광원에 닿았으면
			/*if (nowRay.depth == 1)
			{
				if (IsLight(emission))
				{
					sumLo += emission * (nowRay.decay / SAMPLE_NUM);
				}
			}
			else
			{
				float distance = glm::distance(hitPoint, nowRay.origin);
				float attenuation = 1.0f / (distance * distance);
				sumLo += emission * attenuation * (nowRay.decay / SAMPLE_NUM);
			}*/

			//////////////////////////////////////////////////////////////////////////////////////////분리선

			if (nowRay.depth < DEPTH)
			{
				for (int j = 0; j < SAMPLE_NUM; ++j)
				{
					float r = sqrtf(1.0f -
						randomNums[(rayIndex * SAMPLE_NUM + j) * 2] *
						randomNums[(rayIndex * SAMPLE_NUM + j) * 2]);
					float phi = 2 * glm::pi<float>() * randomNums[(rayIndex * SAMPLE_NUM + j) * 2 + 1];

					vec3 randomVec = normalize(vec3(
						cosf(phi)*r,
						randomNums[(rayIndex * SAMPLE_NUM + j) * 2],
						sinf(phi)*r));

					glm::mat3 TNB = glm::mat3(
						triangles[nearestTriangleIdx].tangent,
						N,
						triangles[nearestTriangleIdx].bitangent);
					vec3 reflectRandomVec = TNB * randomVec;

					
					Ray reflectRay;
					// 여기서 kS.r을 쓴 이유는 reflect ray 하나만 쓰기 때문에 한 것
					// Ray Tracing
					/*reflectRay.dir = normalize(reflect(nowRay.dir, N));
					reflectRay.decay = kS.r * nowRay.decay / SAMPLE_NUM;*/
					
					// Path Tracing
					reflectRay.dir = normalize(reflectRandomVec);
					reflectRay.decay = nowRay.decay *
						glm::clamp(dot(N, reflectRay.dir), 0.0f, 1.0f);
					
					reflectRay.depth = nowRay.depth + 1;
					reflectRay.origin = hitPoint + reflectRay.dir * 0.08f;

					Enqueue(rayQueue, reflectRay, rear);

	/*				glm::mat3 refractTNB = glm::mat3(
						triangles[nearestTriangleIdx].tangent,
						normalize(refract(nowRay.dir, N, 1.0f / materials[materialId].refractiveIndex)),
						-triangles[nearestTriangleIdx].bitangent);
					vec3 refractRandomVec = ;*/

					Ray refractRay;
					// 현재 빛의 감쇠 정도와 물체의 재질에 따라 refract ray의 감쇠 정도가 정해짐
					refractRay.dir = normalize(refract(
						nowRay.dir, N, 1.0f / materials[materialId].refractiveIndex));
					refractRay.decay = nowRay.decay * kD.r / SAMPLE_NUM;

					// 투명한 Object이기 때문에 kD가 refract decay로 들어간 거임

					refractRay.depth = nowRay.depth + 1;
					refractRay.origin = hitPoint + refractRay.dir * 0.08f;

					Enqueue(rayQueue, refractRay, rear);
				}
			}
		}
	}

	// hdr
	sumLo = sumLo / (sumLo + vec3(1.0));
	// gamma correction
	sumLo = glm::pow(sumLo, vec3(1.0 / 2.2));

	vec4 color = glm::vec4(sumLo, 1.0f);

	return color;
}



__global__ void RayTraceD(
	glm::vec4* data,
	const int gridX,
	const int gridY,
	glm::mat4 view,
	Triangle* triangles, int triangleNum,
	Sphere* spheres, int sphereNum,
	Light* lights, int lightNum,
	Material* materials, int matNum,
	float* randomNums,
	OctreeNode* root,
	gpukdtree* kdroot)
{
	//unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int x = (blockIdx.x + gridY * RAY_Y_NUM) * WINDOW_HEIGHT + (threadIdx.x + gridX * RAY_X_NUM);
	glm::vec4 color = glm::vec4(0.0f);

	Ray rayQueue[QUEUE_SIZE];
	
	for (int i = 0; i < 2; i++)
	{
		for (int j = 0; j < 2; j++)
		{
			Ray ray = GenerateCameraRay(blockIdx.x + gridY * RAY_Y_NUM, threadIdx.x + gridX * RAY_X_NUM, view, i, j);

			// NOTICE for문을 돌릴 때 iter를 변수로 하니까 검은 화면이 나옴
			// y, x로 들어가고
			// 0, 0 좌표는 좌하단
			color += RayTraceColor(
				ray,
				blockIdx.x * blockDim.x + threadIdx.x,
				rayQueue,
				triangles,
				triangleNum,
				spheres,
				sphereNum,
				lights,
				lightNum,
				materials,
				matNum,
				randomNums,
				DEPTH,
				root,
				kdroot);
		}
	}

	//color = glm::vec4(randomNums[x%1024]);

	data[x] = color / 4.0f;
}

__global__ void random(float* result, int seed)
{
	hiprandState_t state;
	const int randomMax = 10000;

	hiprand_init(seed, blockIdx.x, 0, &state);
	int randNum = hiprand(&state) % randomMax;

	// theta 범위는 0 ~ 1
	result[blockIdx.x] = (float)randNum / (float)randomMax;
}

void RayTrace(
	glm::vec4* data,
	const int gridX,
	const int gridY,
	glm::mat4 view,
	const vector<Triangle>& triangles,
	const vector<Sphere>& spheres,
	const vector<Light>& lights,
	const vector<Material>& materials,
	const vector<float>& randomThetaPi,
	OctreeNode* root,
	gpukdtree* kdroot)
{
	thrust::device_vector<Triangle> t = triangles;
	thrust::device_vector<Sphere> s = spheres;
	thrust::device_vector<Light> l = lights;
	thrust::device_vector<Material> m = materials;
	thrust::device_vector<float> rnums = randomThetaPi;

	hipDeviceSetLimit(hipLimitMallocHeapSize, 5000000000 * sizeof(float));
	
	/*vec3 min = vec3(-30, -30, -30);
	vec3 max = vec3(30, 30, 30);

	
	int tnum = t.size();

	printf("Num Triangles: %d\n", tnum);

	OctreeNode* root = BuildOctree((Triangle *)triangles.data(), tnum, 1000, min, max);

	OctreeNode* octree = OTHostToDevice(root);*/

	//cout << "ray trace device start" << endl;

	RayTraceD << <RAY_Y_NUM, RAY_X_NUM >> > (
		data,
		gridX,
		gridY,
		view,
		t.data().get(),
		t.size(),
		s.data().get(),
		s.size(),
		l.data().get(),
		l.size(),
		m.data().get(),
		m.size(),
		rnums.data().get(),
		root,
		kdroot
	);
}

void LoadCudaTextures()
{
	Texture2D texFile;
	texFile.LoadFixedTexture("Texture/RustedIron/albedo.png");
	texFile.SetParameters(GL_LINEAR_MIPMAP_LINEAR, GL_LINEAR_MIPMAP_LINEAR, GL_CLAMP_TO_EDGE, GL_CLAMP_TO_EDGE);
	float* texArray = texFile.GetTexImage(GL_RGBA);

	unsigned int size = 2048 * 2048 * 4 * sizeof(float);

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	hipArray* cuArray;
	//hipMipmappedArray* cuMipmappedArray;

	hipMallocArray(&cuArray, &channelDesc, 2048, 2048);
	//hipMalloc3DArray()

	hipMemcpyToArray(cuArray, 0, 0, texArray, size, hipMemcpyHostToDevice);

	albedoTex.addressMode[0] = hipAddressModeWrap;
	albedoTex.addressMode[1] = hipAddressModeWrap;
	albedoTex.filterMode = hipFilterModeLinear;
	albedoTex.normalized = true;

	hipBindTextureToArray(albedoTex, cuArray, channelDesc);
	delete texArray;

	//////////////////////////////////////////////////////////////////////////////

	texFile.LoadFixedTexture("Texture/RustedIron/normal.png");
	texFile.SetParameters(GL_LINEAR_MIPMAP_LINEAR, GL_LINEAR_MIPMAP_LINEAR, GL_CLAMP_TO_EDGE, GL_CLAMP_TO_EDGE);
	texArray = texFile.GetTexImage(GL_RGBA);

	size = 2048 * 2048 * 4 * sizeof(float);

	cuArray;
	hipMallocArray(&cuArray, &channelDesc, 2048, 2048);

	hipMemcpyToArray(cuArray, 0, 0, texArray, size, hipMemcpyHostToDevice);

	normalTex.addressMode[0] = hipAddressModeWrap;
	normalTex.addressMode[1] = hipAddressModeWrap;
	normalTex.filterMode = hipFilterModeLinear;
	normalTex.normalized = true;

	hipBindTextureToArray(normalTex, cuArray, channelDesc);
	delete texArray;

	//////////////////////////////////////////////////////////////////////////////
	//channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	texFile.LoadFixedTexture("Texture/RustedIron/ao.png");
	texFile.SetParameters(GL_LINEAR_MIPMAP_LINEAR, GL_LINEAR_MIPMAP_LINEAR, GL_CLAMP_TO_EDGE, GL_CLAMP_TO_EDGE);
	texArray = texFile.GetTexImage(GL_RGBA);

	size = 2048 * 2048 * 4 * sizeof(float);

	cuArray;
	hipMallocArray(&cuArray, &channelDesc, 2048, 2048);

	hipMemcpyToArray(cuArray, 0, 0, texArray, size, hipMemcpyHostToDevice);

	aoTex.addressMode[0] = hipAddressModeWrap;
	aoTex.addressMode[1] = hipAddressModeWrap;
	aoTex.filterMode = hipFilterModeLinear;
	aoTex.normalized = true;

	hipBindTextureToArray(aoTex, cuArray, channelDesc);
	delete texArray;

	//////////////////////////////////////////////////////////////////////////////

	texFile.LoadFixedTexture("Texture/RustedIron/metallic.png");
	texFile.SetParameters(GL_LINEAR_MIPMAP_LINEAR, GL_LINEAR_MIPMAP_LINEAR, GL_CLAMP_TO_EDGE, GL_CLAMP_TO_EDGE);
	texArray = texFile.GetTexImage(GL_RGBA);

	size = 2048 * 2048 * 4 * sizeof(float);

	cuArray;
	hipMallocArray(&cuArray, &channelDesc, 2048, 2048);

	hipMemcpyToArray(cuArray, 0, 0, texArray, size, hipMemcpyHostToDevice);

	metallicTex.addressMode[0] = hipAddressModeWrap;
	metallicTex.addressMode[1] = hipAddressModeWrap;
	metallicTex.filterMode = hipFilterModeLinear;
	metallicTex.normalized = true;

	hipBindTextureToArray(metallicTex, cuArray, channelDesc);
	delete texArray;

	//////////////////////////////////////////////////////////////////////////////

	texFile.LoadFixedTexture("Texture/RustedIron/roughness.png");
	texFile.SetParameters(GL_LINEAR_MIPMAP_LINEAR, GL_LINEAR_MIPMAP_LINEAR, GL_CLAMP_TO_EDGE, GL_CLAMP_TO_EDGE);
	texArray = texFile.GetTexImage(GL_RGBA);

	size = 2048 * 2048 * 4 * sizeof(float);

	cuArray;
	hipMallocArray(&cuArray, &channelDesc, 2048, 2048);

	hipMemcpyToArray(cuArray, 0, 0, texArray, size, hipMemcpyHostToDevice);

	roughnessTex.addressMode[0] = hipAddressModeWrap;
	roughnessTex.addressMode[1] = hipAddressModeWrap;
	roughnessTex.filterMode = hipFilterModeLinear;
	roughnessTex.normalized = true;

	hipBindTextureToArray(roughnessTex, cuArray, channelDesc);
	delete texArray;

	//////////////////////////////////////////////////////////////////////////////

	texFile.LoadFixedTexture("Texture/Background/stripe.png");
	texFile.SetParameters(GL_LINEAR_MIPMAP_LINEAR, GL_LINEAR_MIPMAP_LINEAR, GL_CLAMP_TO_EDGE, GL_CLAMP_TO_EDGE);
	texArray = texFile.GetTexImage(GL_RGBA);

	size = 2048 * 2048 * 4 * sizeof(float);

	cuArray;
	hipMallocArray(&cuArray, &channelDesc, 2048, 2048);

	hipMemcpyToArray(cuArray, 0, 0, texArray, size, hipMemcpyHostToDevice);

	backgroundTex.addressMode[0] = hipAddressModeWrap;
	backgroundTex.addressMode[1] = hipAddressModeWrap;
	backgroundTex.filterMode = hipFilterModeLinear;
	backgroundTex.normalized = true;

	hipBindTextureToArray(backgroundTex, cuArray, channelDesc);
	delete texArray;
}