#include "hip/hip_runtime.h"
﻿#include "RayTracer.cuh"
#include "Octree.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm\glm.hpp>
#include <glm\gtx\component_wise.hpp>
#include <stdio.h>
#include <glm\gtc\matrix_transform.hpp>
#include <hip/hip_math_constants.h>
#include <math.h>
#include <thrust\device_vector.h>
#include <queue>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <stdio.h>

texture<float4, 2, hipReadModeElementType> albedoTex;
texture<float4, 2, hipReadModeElementType> normalTex;
texture<float4, 2, hipReadModeElementType> aoTex;
texture<float4, 2, hipReadModeElementType> metallicTex;
texture<float4, 2, hipReadModeElementType> roughnessTex;

texture<float4, 2, hipReadModeElementType> backgroundTex;

struct Ray
{
	// Ray의 원점
	vec3 origin;
	// Ray의 방향
	vec3 dir;
	
	float decay;
};

const int WINDOW_HEIGHT = 1024;
const int WINDOW_WIDTH = 1024;

const int RAY_X_NUM = 64;
const int RAY_Y_NUM = 64;

const int QUEUE_SIZE = 30;

const int DEPTH = 2;

const int SAMPLE_NUM = 1;

using std::cout;
using std::endl;
using std::max;
using std::min;

// TODO LIST
// 1. 에너지 보존 for reflect and refract 
// 2. path tracing
// ggx distribution이라고 외우자
__device__ float DistributionGGX(vec3 N, vec3 H, float roughness)
{
	float a = roughness * roughness;
	float a2 = a * a;
	float NdotH = max(dot(N, H), 0.0f);
	float NdotH2 = NdotH * NdotH;

	float nominator = a2;
	float denominator = (NdotH2 * (a2 - 1.0) + 1.0);
	denominator = glm::pi<float>() * denominator * denominator;

	return nominator / denominator;
}

__device__ float GeometrySchlickGGX(float NdotV, float roughness)
{
	float r = (roughness + 1.0);
	float k = (r * r) / 8.0;

	float nominator = NdotV;
	float denominator = NdotV * (1.0 - k) + k;

	return nominator / denominator;
}

// smith geometry라고 외우자
// geometry shadowing 빛이 어떤 표면으로 갈 때 다른 표면에 막혀 가지 못하는 경우
// geometry obstruction 빛이 어떤 표면에서 눈으로 갈 때 다른 표면에 막혀 가지 못하는 경우
// 이 두가지를 모두 고려해야 해서 ggx1 * ggx2
// 0.8이 안 막히고, 0.8이 안 막힌다면 결국은 0.8 * 0.8
__device__ float GeometrySmith(vec3 N, vec3 V, vec3 L, float roughness)
{
	float NdotV = max(dot(N, V), 0.0f);
	float NdotL = max(dot(N, L), 0.0f);

	float ggx2 = GeometrySchlickGGX(NdotV, roughness);
	float ggx1 = GeometrySchlickGGX(NdotL, roughness);

	return ggx1 * ggx2;
}

// cosTheta가 작을 수록 큰 값이 들어간다
// 즉 90도에 가까운 곳에서 볼 수록 빛이 쎄진다는 것이다.
// 90이면 그냥 1임
// 각도가 높아지면 점점 약해지고 F0값에 가까워짐
__device__ vec3 fresnelSchlick(float cosTheta, vec3 F0)
{
	return F0 + (1.0f - F0) * pow(1.0f - cosTheta, 5.0f);
}

__device__ vec3 calculateEta(float refractiveIndex)
{
	return vec3(powf(1.0f - (1.0f / refractiveIndex), 2.0f) / powf(1.0f + (1.0f / refractiveIndex), 2.0f));
}

__device__ vec3 Interpolation(Triangle triangle, vec3 position, vec3& N, vec2& uv)
{
	vec3 v0 = triangle.v1 - triangle.v0;
	vec3 v1 = triangle.v2 - triangle.v0;
	vec3 v2 = position - triangle.v0;

	float d00 = dot(v0, v0);
	float d01 = dot(v0, v1);
	float d11 = dot(v1, v1);
	float d20 = dot(v2, v0);
	float d21 = dot(v2, v1);
	float denom = d00*d11 - d01*d01;

	float v = (d11*d20 - d01*d21) / denom;
	float w = (d00*d21 - d01*d20) / denom;
	float u = 1.0f - v - w;

	N = u * triangle.v0normal + v * triangle.v1normal + w * triangle.v2normal;
	uv = u * triangle.v0uv + v * triangle.v1uv + w * triangle.v2uv;
}

// ray와 sphere가 intersect하는지 검사하는 함수
__device__ bool RaySphereIntersect(Ray ray, Sphere sphere, float& dist)
{
	glm::vec3 s = ray.origin - sphere.origin;
	float minDist = 0.001f;

	float a = dot(ray.dir, ray.dir);
	float bPrime = dot(s, ray.dir);
	float c = dot(s, s) - sphere.radius * sphere.radius;

	float D = bPrime * bPrime - a * c;
	if (D >= 0 && bPrime <= 0)
	{
		float t1 = (-bPrime + sqrt(D)) / a;
		float t2 = (-bPrime - sqrt(D)) / a;
		dist = t1 > t2 ? t2 : t1;
		return dist > minDist;
	}
	else
		return false;
}

// ray와 triangle이 intersect하는지 검사하는 함수
__device__ bool RayTriangleIntersect(Ray ray, Triangle triangle, float& dist)
{
	glm::vec3 v0v1 = triangle.v1 - triangle.v0;
	glm::vec3 v0v2 = triangle.v2 - triangle.v0;
	glm::vec3 pvec = glm::cross(ray.dir, v0v2);

	float det = dot(v0v1, pvec);

	// back face culling
	if (det < 0.01f)
		return false;

	/*if (fabsf(det) < 0.01f)
		return false;*/

	float invDet = 1 / det;

	glm::vec3 tvec = ray.origin - triangle.v0;
	float u = glm::dot(tvec, pvec) * invDet;
	if (u < 0 || u > 1)
		return false;

	glm::vec3 qvec = cross(tvec, v0v1);
	float v = dot(ray.dir, qvec) * invDet;
	if (v < 0 || u + v > 1)
		return false;

	dist = dot(v0v2, qvec) * invDet;

	return true;
}

//bool RayPlaneIntersect(Ray ray, vec4 plane, float& t)
//{
//	t = -dot(plane, vec4(ray.origin, 1.0)) / glm::dot(glm::vec3(plane), ray.dir);
//	return t > 0.0;
//}
//
//bool RayRectIntersect(Ray ray, Rect rect, float& t)
//{
//	bool intersect = RayPlaneIntersect(ray, rect.plane, t);
//	if (intersect)
//	{
//		vec3 pos = ray.origin + ray.dir*t;
//		vec3 lpos = pos - rect.center;
//
//		float x = dot(lpos, rect.dirx);
//		float y = dot(lpos, rect.diry);
//
//		if (abs(x) > rect.halfx || abs(y) > rect.halfy)
//			intersect = false;
//	}
//
//	return intersect;
//}

__device__ bool RayAABBIntersect(Ray ray, AABB box)
{
	float tmin, tmax, tymin, tymax, tzmin, tzmax;

	glm::vec3 invdir = 1.0f / ray.dir;
	int sign[3];
	sign[0] = invdir.x < 0;
	sign[1] = invdir.y < 0;
	sign[2] = invdir.z < 0;

	tmin = (box.bounds[sign[0]].x - ray.origin.x) * invdir.x;
	tmax = (box.bounds[1 - sign[0]].x - ray.origin.x) * invdir.x;
	tymin = (box.bounds[sign[1]].y - ray.origin.y) * invdir.y;
	tymax = (box.bounds[1 - sign[1]].y - ray.origin.y) * invdir.y;

	if ((tmin > tymax) || (tymin > tmax))
		return false;

	if (tymin > tmin)
		tmin = tymin;
	if (tymax < tmax)
		tmax = tymax;

	tzmin = (box.bounds[sign[2]].z - ray.origin.z) * invdir.z;
	tzmax = (box.bounds[1 - sign[2]].z - ray.origin.z) * invdir.z;

	if ((tmin > tzmax) || (tzmin > tmax))
		return false;

	return true;
}

__device__ bool RayAABBsIntersect(Ray ray, AABB* boxes, int boxNum)
{
	bool isIntersect = false;

	for (int it = 0; it < boxNum; it++)
	{
		if (RayAABBIntersect(ray, boxes[it]))
			isIntersect = true;
	}
	return isIntersect;
}

// ray의 원점과 가장 가까운 곳에서 intersect하는 triangle의 id를 가져오는 함수
__device__ int FindNearestTriangleIdx(Ray ray, Triangle* triangles, int triangleNum, float& dist)
{
	const float rayThreshold = 0.01f;
	float minDist = 9999999.0f;
	int minIdx = -1;
	float tmpDist;

	for (int i = 0; i < triangleNum; ++i)
	{
		if (RayTriangleIntersect(ray, triangles[i], tmpDist))
		{
			if (tmpDist > rayThreshold && tmpDist < minDist)
			{
				minDist = tmpDist;
				minIdx = i;
			}
		}
	}

	dist = minDist;
	return minIdx;
}

// ray의 원점과 가장 가까운 곳에서 intersect하는 sphere의 id를 가져오는 함수
__device__ int FindNearestSphereIdx(Ray ray, Sphere* spheres, int sphereNum, float& dist)
{
	const float rayThreshold = 0.0001f;
	float minDist = 9999999.0f;
	int minIdx = -1;
	float tmpDist;

	for (int i = 0; i < sphereNum; ++i)
	{
		if (RaySphereIntersect(ray, spheres[i], tmpDist))
		{
			if (tmpDist > rayThreshold && tmpDist < minDist)
			{
				minDist = tmpDist;
				minIdx = i;
			}
		}
	}

	dist = minDist;
	return minIdx;
}

// window의 픽셀의 위치가 각각 x, y로 입력됨
__device__ Ray GenerateCameraRay(int y, int x, glm::mat4 cameraModelMatrix, int rayX, int rayY)
{
	Ray ray;

	// 각 픽셀의 중앙을 가르키는 값 생성, 0~1의 값으로 Normalizing
	// antialiasing
	float NDCy = (y + 0.33333f + 0.33333f*rayY) / WINDOW_HEIGHT;
	float NDCx = (x + 0.33333f + 0.33333f*rayX) / WINDOW_WIDTH;

	/*float NDCy = (y + 0.5f) / WINDOW_HEIGHT;
	float NDCx = (x + 0.5f) / WINDOW_WIDTH;*/

	// window 종횡비
	float aspectRatio = WINDOW_WIDTH / WINDOW_HEIGHT;

	// 시야각 설정
	float fov = 45.0f;

	// unProject
	float xx = (NDCx * 2.0f - 1.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f) * aspectRatio;
	float yy = (NDCy * 2.0f - 1.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f);

	// world space에서의 ray 정보를 계산
	ray.origin = glm::vec3(cameraModelMatrix * glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
	ray.dir = normalize(vec3(cameraModelMatrix * vec4(glm::vec3(xx, yy, -1.0), 0.0f)));

	// 만들어진 ray를 return
	return ray;
}

__device__ void Enqueue(Ray* rayQueue, Ray ray, int& rear)
{
	rear = (rear + 1) % QUEUE_SIZE;
	rayQueue[rear] = ray;
}

__device__ void Dequeue(Ray* rayQueue, int& front)
{
	Ray ray = rayQueue[front];
	front = (front + 1) % QUEUE_SIZE;
}

__device__ Ray GetQueueFront(Ray* rayQueue, const int front)
{
	return rayQueue[(front + 1) % QUEUE_SIZE];
}

__device__ bool IsQueueEmpty(const int front, const int rear)
{
	return front == rear;
}

__device__ bool IsLighted(
	vec3 hitPoint,
	Light light,
	Triangle* triangles,
	const int triangleNum,
	const int nearestTriangleIdx,
	Sphere* spheres,
	const int sphereNum,
	const int nearestSphereIdx)
{
	// shadow ray 생성, origin은 hit point, 방향은 hit point부터 광원까지의 방향
	Ray shadowRay;
	shadowRay.origin = hitPoint;
	shadowRay.dir = normalize(light.pos - hitPoint);
	float distance = glm::distance(light.pos, hitPoint);

	float distToTriangle;

	for (int t_i = 0; t_i < triangleNum; ++t_i)
	{
		// 처음 hit한 triangle은 제외
		if (nearestTriangleIdx != t_i)
		{
			// shadow
			if (RayTriangleIntersect(shadowRay, triangles[t_i], distToTriangle))
			{
				// 앞쪽의 dir만 봄
				if (distToTriangle > 0.01f && distToTriangle < glm::distance(light.pos, hitPoint))
				{
					return false;
				}
			}
		}
	}

	float distToSphere;

	for (int s_i = 0; s_i < sphereNum; ++s_i)
	{
		// 광원은 0임, 광원을 제외한 경우에만 그림자 생김
		if (nearestSphereIdx != s_i && s_i != 0)
		{
			if (RaySphereIntersect(shadowRay, spheres[s_i], distToSphere))
			{
				// 앞쪽의 dir만 봄
				if (distToSphere > 0.01f && distToSphere < glm::distance(light.pos, hitPoint))
				{
					return false;
				}
			}
		}
	}

	return true;
}

// ray가 hit 했다면 true를 리턴하고 hit한 곳의 정보를 가져오는 함수
__device__ bool GetHitPointInfo(
	Ray nowRay,
	Triangle* triangles,
	int triangleNum,
	int& nearestTriangleIdx,
	Sphere* spheres,
	int sphereNum,
	int& nearestSphereIdx,
	vec3& hitPoint, 
	int& materialId, 
	vec3& N,
	vec2& uv)
{
	float distToTriangle, distToSphere, distToAreaLight = 0.0f;
	nearestTriangleIdx = FindNearestTriangleIdx(nowRay, triangles, triangleNum, distToTriangle);
	nearestSphereIdx = FindNearestSphereIdx(nowRay, spheres, sphereNum, distToSphere);

	// 아무곳도 intersect를 못했다거나 뒤쪽에 있다면
	if ((nearestTriangleIdx == -1 || distToTriangle < 0.0f) &&
		(nearestSphereIdx == -1 || distToSphere < 0.0f) &&
		(distToAreaLight <= 0.0f))
		return false;

	if (distToSphere > distToTriangle)
	{
		Triangle nearestTriangle = triangles[nearestTriangleIdx];
		hitPoint = nowRay.origin + nowRay.dir * distToTriangle;
		materialId = nearestTriangle.materialId;
		Interpolation(nearestTriangle, hitPoint, N, uv);
	}
	else
	{
		Sphere nearestSphere = spheres[nearestSphereIdx];
		hitPoint = nowRay.origin + nowRay.dir * distToSphere;
		materialId = nearestSphere.materialId;
		N = glm::normalize(hitPoint - nearestSphere.origin);
		// no uv...
	}

	return true;
}

__device__ vec4 RayTraceColor(
	Ray ray,
	Ray* rayQueue,
	float* randomNums,
	AABB* objects,
	int objNum,
	Triangle* triangles,
	int triangleNum,
	Sphere* spheres,
	int sphereNum,
	Light* lights,
	int lightNum,
	Material* materials,
	int matNum,
	int depth)
{
	vec4 color = vec4(0.0f);
	int front = 0, rear = 0;

	Enqueue(rayQueue, ray, rear);

	int nowDepth = 1;
	vec3 V = -ray.dir;

	for (int i = 1; i < depth; ++i)
	{
		int target = rear;

		while (!IsQueueEmpty(target, front))
		{
			Ray nowRay;
			nowRay = GetQueueFront(rayQueue, front);
			Dequeue(rayQueue, front);

			if (!RayAABBsIntersect(nowRay, objects, objNum))
				continue;

			vec4 lightedColor = glm::vec4(0.0f);
			vec3 hitPoint = glm::vec3(0.0f);
			// hit한 object의 material id
			int materialId = 0;
			// normal vector
			vec3 N = glm::vec3(0.0f);
			vec2 uv = glm::vec2(0.0f);
			int nearestTriangleIdx = 0;
			int nearestSphereIdx = 0;

			// hit point의 정보를 가져옴
			if (GetHitPointInfo(
				nowRay,
				triangles,
				triangleNum,
				nearestTriangleIdx,
				spheres,
				sphereNum,
				nearestSphereIdx,
				hitPoint,
				materialId,
				N,
				uv))
			{

				// ∫Ω(kd c / π + ks DFG / 4(ωo⋅n)(ωi⋅n)) Li(p,ωi) n⋅ωi dωi
				// radiance * (1.0f * textureColor/pi + 0.0f) * lightcolor * NdotL
				vec3 albedo;
				vec3 emission;
				vec3 F0;
				float4 texNormal;
				float ao;
				float metallic;
				float roughness;

				vec3 kS;
				vec3 kD;

				if (materials[materialId].texId == 0)
				{
					float4 texRGBA;
					texRGBA = tex2D(albedoTex, uv.x, uv.y);
					albedo = glm::pow(glm::vec3(texRGBA.x, texRGBA.y, texRGBA.z), vec3(2.2));

					texNormal = tex2D(normalTex, uv.x, uv.y);
					ao = tex2D(aoTex, uv.x, uv.y).x;
					metallic = tex2D(metallicTex, uv.x, uv.y).x;
					roughness = tex2D(roughnessTex, uv.x, uv.y).x;

					glm::vec3 texNormalVec = glm::vec3(
						texNormal.x * 2.0f - 1.0f,
						texNormal.y * 2.0f - 1.0f,
						texNormal.z * 2.0f - 1.0f);

					glm::mat3 TBN = glm::mat3(
						triangles[nearestTriangleIdx].tangent,
						triangles[nearestTriangleIdx].bitangent,
						N);

					// TBN의 inverse
					N = glm::normalize(texNormalVec);

					N = TBN * N;
				}
				else if (materials[materialId].texId == 1)
				{
					float4 texRGBA;
					texRGBA = tex2D(backgroundTex, uv.x, uv.y);
					albedo = glm::pow(glm::vec3(texRGBA.x, texRGBA.y, texRGBA.z), vec3(2.2));

					ao = materials[materialId].ambient;
					metallic = materials[materialId].metallic;
					roughness = materials[materialId].roughness;
					emission = materials[materialId].emission;
				}
				else
				{
					albedo = materials[materialId].albedo;
					ao = materials[materialId].ambient;
					metallic = materials[materialId].metallic;
					roughness = materials[materialId].roughness;
					emission = materials[materialId].emission;
				}

				if (materials[materialId].refractiveIndex != 0.0f)
				{
					F0 = calculateEta(materials[materialId].refractiveIndex);
				}
				else
				{
					F0 = glm::mix(vec3(0.04f), albedo, metallic);
				}

				vec3 Lo = vec3(0.0f);
				for (int k = 0; k < lightNum; k++)
				{
					if (!IsLighted(hitPoint, lights[k], triangles, triangleNum, nearestTriangleIdx,
						spheres, sphereNum, nearestSphereIdx))
					{
						continue;
					}

					vec3 L = glm::normalize(lights[k].pos - hitPoint);
					vec3 H = glm::normalize(V + L);

					float distance = glm::distance(lights[k].pos, hitPoint);
					float attenuation = 1.0 / (distance*distance);

					vec3 radiance = lights[k].color * attenuation;

					float NDF = DistributionGGX(N, H, roughness);
					float G = GeometrySmith(N, V, L, roughness);
					vec3 F = fresnelSchlick(glm::max(glm::dot(H, V), 0.0f), F0);

					vec3 nominator = NDF * G * F;
					float denominator = 4 * glm::max(glm::dot(N, V), 0.0f) * glm::max(glm::dot(N, L), 0.0f) + 0.001f;
					vec3 specular = nominator / denominator;

					kS = F;
					kD = vec3(1.0) - kS;
					kD *= 1.0f - metallic;

					float NdotL = glm::clamp(glm::dot(N, L), 0.0f, 1.0f);

					Lo += (kD*albedo / glm::pi<float>() + specular) * radiance * NdotL;
				}

				vec3 ambient = vec3(0.03) * albedo * ao;

				vec3 tmpColor = ambient + Lo + emission;

				// hdr
				tmpColor = tmpColor / (tmpColor + vec3(1.0));
				// gamma correction
				tmpColor = glm::pow(tmpColor, vec3(1.0 / 2.2));

				lightedColor += glm::vec4(tmpColor, 1.0f);

				color += lightedColor * nowRay.decay;

				//////////////////////////////////////////////////////////////////////////////////////////분리선

				for (int j = 0; j < SAMPLE_NUM; ++j)
				{
					// theta, phi
					vec3 randomVec = vec3(
						cosf(randomNums[j * 2])*sinf(randomNums[j * 2 + 1]),
						sinf(randomNums[j * 2]),
						cosf(randomNums[j * 2])*cosf(randomNums[j * 2 + 1]));
					
					glm::mat3 TNB = glm::mat3(
						triangles[nearestTriangleIdx].tangent,
						N,
						triangles[nearestTriangleIdx].bitangent);
					randomVec = randomVec * TNB;


					Ray reflectRay;
					// reflect ray의 시작점은 hit point
					reflectRay.origin = hitPoint;
					//reflectRay.dir = normalize(randomVec);
					reflectRay.dir = normalize(reflect(nowRay.dir, N));

					// 현재 빛의 감쇠 정도와 물체의 재질에 따라 reflect ray의 감쇠 정도가 정해짐 
					reflectRay.decay = kS.r * ray.decay / SAMPLE_NUM;

					Enqueue(rayQueue, reflectRay, rear);
				}

				// refract는 ray tracing
				Ray refractRay;
				// refract ray의 시작점은 hit point
				refractRay.origin = hitPoint;
				refractRay.dir = normalize(refract(nowRay.dir, N, 1.0f / materials[materialId].refractiveIndex));
				// 현재 빛의 감쇠 정도와 물체의 재질에 따라 refract ray의 감쇠 정도가 정해짐
				refractRay.decay = kD.r * ray.decay;

				Enqueue(rayQueue, refractRay, rear);
			}
		}

		nowDepth++;
	}

	// 나오지 못한 queue들 나오게 하기
	while (!IsQueueEmpty(front, rear))
	{
		Ray nowRay;
		nowRay = GetQueueFront(rayQueue, front);
		Dequeue(rayQueue, front);

		if (!RayAABBsIntersect(nowRay, objects, objNum))
			continue;

		vec4 lightedColor = glm::vec4(0.0f);
		vec3 hitPoint = glm::vec3(0.0f);
		int materialId = 0;
		vec3 N = glm::vec3(0.0f);
		vec2 uv = glm::vec2(0.0f);
		int nearestTriangleIdx = 0;
		int nearestSphereIdx = 0;

		if (GetHitPointInfo(
			nowRay,
			triangles,
			triangleNum,
			nearestTriangleIdx,
			spheres,
			sphereNum,
			nearestSphereIdx,
			hitPoint,
			materialId,
			N,
			uv))
		{
			vec3 albedo;
			vec3 emission;
			vec3 F0;
			float4 texNormal;
			float ao;
			float metallic;
			float roughness;

			vec3 kS;
			vec3 kD;

			if (materials[materialId].texId == 0)
			{
				float4 texRGBA;
				texRGBA = tex2D(albedoTex, uv.x, uv.y);
				albedo = glm::pow(glm::vec3(texRGBA.x, texRGBA.y, texRGBA.z), vec3(2.2));

				texNormal = tex2D(normalTex, uv.x, uv.y);
				ao = tex2D(aoTex, uv.x, uv.y).x;
				metallic = tex2D(metallicTex, uv.x, uv.y).x;
				roughness = tex2D(roughnessTex, uv.x, uv.y).x;

				glm::vec3 texNormalVec = glm::vec3(
					texNormal.x * 2.0f - 1.0f,
					texNormal.y * 2.0f - 1.0f,
					texNormal.z * 2.0f - 1.0f);

				glm::mat3 TBN = glm::mat3(
					triangles[nearestTriangleIdx].tangent,
					triangles[nearestTriangleIdx].bitangent,
					N);

				// TBN의 inverse
				N = glm::normalize(texNormalVec);

				N = TBN * N;
			}
			else if (materials[materialId].texId == 1)
			{
				float4 texRGBA;
				texRGBA = tex2D(backgroundTex, uv.x, uv.y);
				albedo = glm::pow(glm::vec3(texRGBA.x, texRGBA.y, texRGBA.z), vec3(2.2));

				ao = materials[materialId].ambient;
				metallic = materials[materialId].metallic;
				roughness = materials[materialId].roughness;
				emission = materials[materialId].emission;
			}
			else
			{
				albedo = materials[materialId].albedo;
				ao = materials[materialId].ambient;
				metallic = materials[materialId].metallic;
				roughness = materials[materialId].roughness;
				emission = materials[materialId].emission;
			}

			if (materials[materialId].refractiveIndex != 0.0f)
			{
				F0 = calculateEta(materials[materialId].refractiveIndex);
			}
			else
			{
				F0 = glm::mix(vec3(0.04f), albedo, metallic);
			}

			vec3 Lo = vec3(0.0f);
			for (int k = 0; k < lightNum; k++)
			{
				if (!IsLighted(hitPoint, lights[k], triangles, triangleNum, nearestTriangleIdx,
					spheres, sphereNum, nearestSphereIdx))
				{
					continue;
				}

				vec3 L = glm::normalize(lights[k].pos - hitPoint);
				vec3 H = glm::normalize(V + L);

				float distance = glm::distance(lights[k].pos, hitPoint);
				float attenuation = 1.0 / (distance*distance);

				vec3 radiance = lights[k].color * attenuation;

				float NDF = DistributionGGX(N, H, roughness);
				float G = GeometrySmith(N, V, L, roughness);
				vec3 F = fresnelSchlick(glm::max(glm::dot(H, V), 0.0f), F0);

				vec3 nominator = NDF*G*F;
				float denominator = 4 * glm::max(glm::dot(N, V), 0.0f) * glm::max(glm::dot(N, L), 0.0f) + 0.001f;
				vec3 specular = nominator / denominator;

				kS = F;
				kD = vec3(1.0) - kS;
				kD *= (1.0f - metallic);

				float NdotL = glm::clamp(glm::dot(N, L), 0.0f, 1.0f);

				Lo += (kD*albedo / glm::pi<float>() + specular) * radiance * NdotL;
			}

			vec3 ambient = vec3(0.03) * albedo * ao;

			vec3 tmpColor = ambient + Lo + emission;

			// hdr
			tmpColor = tmpColor / (tmpColor + vec3(1.0));
			// gamma correction
			tmpColor = glm::pow(tmpColor, vec3(1.0 / 2.2));

			lightedColor += glm::vec4(tmpColor, 1.0f);

			color += lightedColor * nowRay.decay;
		}
	}

	return color;
}

__global__ void RayTraceD(
	glm::vec4* data,
	float* randomNums,
	const int gridX,
	const int gridY,
	glm::mat4 view,
	OctreeNode* root,
	AABB* boundingboxes, int boxNum,
	Triangle* triangles, int triangleNum,
	Sphere* spheres, int sphereNum,
	Light* lights, int lightNum,
	Material* materials, int matNum)
{
	//unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int x = (blockIdx.x + gridY * RAY_Y_NUM) * WINDOW_HEIGHT + (threadIdx.x + gridX * RAY_X_NUM);
	glm::vec4 color = glm::vec4(0.0f);

	Ray rayQueue[QUEUE_SIZE];
	
	for (int i = 0; i < 2; i++)
	{
		for (int j = 0; j < 2; j++)
		{
			Ray ray = GenerateCameraRay(blockIdx.x + gridY * RAY_Y_NUM, threadIdx.x + gridX * RAY_X_NUM, view, i, j);

			ray.decay = 1.0f;

			// NOTICE for문을 돌릴 때 iter를 변수로 하니까 검은 화면이 나옴
			// y, x로 들어가고
			// 0, 0 좌표는 좌하단
			color += RayTraceColor(
				ray,
				rayQueue,
				randomNums,
				boundingboxes,
				boxNum,
				triangles,
				triangleNum,
				spheres,
				sphereNum,
				lights,
				lightNum,
				materials,
				matNum,
				DEPTH);
		}
	}
	data[x] = color / 4.0f;
}

__global__ void random(float* result)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	hiprandState_t state;
	const int randomMax = 10000;

	hiprand_init(0, 0, 0, &state);
	int randNum = hiprand(&state) % randomMax;

	// theta 범위는 0 ~ 6.28
	if (x % 2 == 0)
		result[x] = (float)randNum / (float)randomMax * glm::pi<float>() * 2;
	// phi 범위는 0 ~ 3.14
	else
		result[x] = (float)randNum / (float)randomMax * glm::pi<float>();
}

void RayTrace(
	glm::vec4* data,
	const int gridX,
	const int gridY,
	glm::mat4 view,
	OctreeNode* root,
	const vector<AABB>& boundingboxes,
	const vector<Triangle>& triangles,
	const vector<Sphere>& spheres,
	const vector<Light>& lights,
	const vector<Material>& materials)
{
	thrust::device_vector<AABB> b = boundingboxes;
	thrust::device_vector<Triangle> t = triangles;
	thrust::device_vector<Sphere> s = spheres;
	thrust::device_vector<Light> l = lights;
	thrust::device_vector<Material> m = materials;

	hipDeviceSetLimit(hipLimitMallocHeapSize, 5000000000 * sizeof(float));

	float* randomThetaPi;
	// 엄밀히 말하면 sample^depth개의 random variable이 필요하지만 sample의 제곱으로 함
	hipMalloc((void**)&randomThetaPi, sizeof(float) * SAMPLE_NUM * SAMPLE_NUM);

	random << <SAMPLE_NUM, SAMPLE_NUM>> > (randomThetaPi);

	vector<Triangle> tss;
	OctreeNode* d_root = BuildOctree(tss);

	RayTraceD << <RAY_Y_NUM, RAY_X_NUM >> > (
		data,
		randomThetaPi,
		gridX,
		gridY,
		view,
		d_root,
		b.data().get(),
		b.size(),
		t.data().get(),
		t.size(),
		s.data().get(),
		s.size(),
		l.data().get(),
		l.size(),
		m.data().get(),
		m.size()
	);

	hipFree(randomThetaPi);
}

void LoadCudaTextures()
{
	Texture2D texFile;
	texFile.LoadFixedTexture("Texture/RustedIron/albedo.png");
	texFile.SetParameters(GL_CLAMP_TO_EDGE, GL_CLAMP_TO_EDGE, GL_LINEAR, GL_LINEAR);
	float* texArray = texFile.GetTexImage(GL_RGBA);

	unsigned int size = 2048 * 2048 * 4 * sizeof(float);

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	hipArray* cuArray;
	hipMallocArray(&cuArray, &channelDesc, 2048, 2048);

	hipMemcpyToArray(cuArray, 0, 0, texArray, size, hipMemcpyHostToDevice);

	albedoTex.addressMode[0] = hipAddressModeWrap;
	albedoTex.addressMode[1] = hipAddressModeWrap;
	albedoTex.filterMode = hipFilterModeLinear;
	albedoTex.normalized = true;

	hipBindTextureToArray(albedoTex, cuArray, channelDesc);
	delete texArray;

	//////////////////////////////////////////////////////////////////////////////

	texFile.LoadFixedTexture("Texture/RustedIron/normal.png");
	texFile.SetParameters(GL_CLAMP_TO_EDGE, GL_CLAMP_TO_EDGE, GL_LINEAR, GL_LINEAR);
	texArray = texFile.GetTexImage(GL_RGBA);

	size = 2048 * 2048 * 4 * sizeof(float);

	cuArray;
	hipMallocArray(&cuArray, &channelDesc, 2048, 2048);

	hipMemcpyToArray(cuArray, 0, 0, texArray, size, hipMemcpyHostToDevice);

	normalTex.addressMode[0] = hipAddressModeWrap;
	normalTex.addressMode[1] = hipAddressModeWrap;
	normalTex.filterMode = hipFilterModeLinear;
	normalTex.normalized = true;

	hipBindTextureToArray(normalTex, cuArray, channelDesc);
	delete texArray;

	//////////////////////////////////////////////////////////////////////////////
	//channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	texFile.LoadFixedTexture("Texture/RustedIron/ao.png");
	texFile.SetParameters(GL_CLAMP_TO_EDGE, GL_CLAMP_TO_EDGE, GL_LINEAR, GL_LINEAR);
	texArray = texFile.GetTexImage(GL_RGBA);

	size = 2048 * 2048 * 4 * sizeof(float);

	cuArray;
	hipMallocArray(&cuArray, &channelDesc, 2048, 2048);

	hipMemcpyToArray(cuArray, 0, 0, texArray, size, hipMemcpyHostToDevice);

	aoTex.addressMode[0] = hipAddressModeWrap;
	aoTex.addressMode[1] = hipAddressModeWrap;
	aoTex.filterMode = hipFilterModeLinear;
	aoTex.normalized = true;

	hipBindTextureToArray(aoTex, cuArray, channelDesc);
	delete texArray;

	//////////////////////////////////////////////////////////////////////////////

	texFile.LoadFixedTexture("Texture/RustedIron/metallic.png");
	texFile.SetParameters(GL_CLAMP_TO_EDGE, GL_CLAMP_TO_EDGE, GL_LINEAR, GL_LINEAR);
	texArray = texFile.GetTexImage(GL_RGBA);

	size = 2048 * 2048 * 4 * sizeof(float);

	cuArray;
	hipMallocArray(&cuArray, &channelDesc, 2048, 2048);

	hipMemcpyToArray(cuArray, 0, 0, texArray, size, hipMemcpyHostToDevice);

	metallicTex.addressMode[0] = hipAddressModeWrap;
	metallicTex.addressMode[1] = hipAddressModeWrap;
	metallicTex.filterMode = hipFilterModeLinear;
	metallicTex.normalized = true;

	hipBindTextureToArray(metallicTex, cuArray, channelDesc);
	delete texArray;

	//////////////////////////////////////////////////////////////////////////////

	texFile.LoadFixedTexture("Texture/RustedIron/roughness.png");
	texFile.SetParameters(GL_CLAMP_TO_EDGE, GL_CLAMP_TO_EDGE, GL_LINEAR, GL_LINEAR);
	texArray = texFile.GetTexImage(GL_RGBA);

	size = 2048 * 2048 * 4 * sizeof(float);

	cuArray;
	hipMallocArray(&cuArray, &channelDesc, 2048, 2048);

	hipMemcpyToArray(cuArray, 0, 0, texArray, size, hipMemcpyHostToDevice);

	roughnessTex.addressMode[0] = hipAddressModeWrap;
	roughnessTex.addressMode[1] = hipAddressModeWrap;
	roughnessTex.filterMode = hipFilterModeLinear;
	roughnessTex.normalized = true;

	hipBindTextureToArray(roughnessTex, cuArray, channelDesc);
	delete texArray;

	//////////////////////////////////////////////////////////////////////////////

	texFile.LoadFixedTexture("Texture/Background/stripe.png");
	texFile.SetParameters(GL_CLAMP_TO_EDGE, GL_CLAMP_TO_EDGE, GL_LINEAR, GL_LINEAR);
	texArray = texFile.GetTexImage(GL_RGBA);

	size = 2048 * 2048 * 4 * sizeof(float);

	cuArray;
	hipMallocArray(&cuArray, &channelDesc, 2048, 2048);

	hipMemcpyToArray(cuArray, 0, 0, texArray, size, hipMemcpyHostToDevice);

	backgroundTex.addressMode[0] = hipAddressModeWrap;
	backgroundTex.addressMode[1] = hipAddressModeWrap;
	backgroundTex.filterMode = hipFilterModeLinear;
	backgroundTex.normalized = true;

	hipBindTextureToArray(backgroundTex, cuArray, channelDesc);
	delete texArray;
}