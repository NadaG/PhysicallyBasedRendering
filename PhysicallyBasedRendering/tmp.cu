
#include <hip/hip_runtime.h>
//#include "RayTracingRenderer.h"
//#include "LectureSceneRenderer.h"
//#include "PBRRenderer.h"
//#include "LTCRenderer.h"
//#include "FluidRenderer.h"
//
//#include "WindowManager.h"
//#include "Debug.h"
//#include "RayTracer.cuh"
//
//#include "Octree.cuh"
//
//#define STB_IMAGE_IMPLEMENTATION
//#include "stb_image.h"
//
//using namespace std;
//
//enum Scene
//{
//	PBR_SCENE = 0,
//	FLUID_SCENE = 1,
//	LTC_SCENE = 2,
//	SMOKE_SCENE = 3,
//	STARBURST_SCENE = 4,
//	TEMPORALGLARE_SCENE = 5,
//	RAYTRACING_SCENE = 6
//};
//
//// TODO movement 셋팅하고 사용하는 부분 너무 복잡함, 함수로 빼내던가 해야함
//// TODO scene object update하고 render 하는 부분 이상함, renderer에서 여러 object를 다 가져와야하는 문제 해결
//// TODO shader 불러오고 사용하고 그리는 부분 중복되는 부분이 너무 많은데 그 부분들 수정하기
//
//struct BTNode
//{
//	int data = NULL;
//
//	BTNode *left = nullptr;
//	BTNode *right = nullptr;
//};
//
//
//BTNode* CreateNode(int data)
//{
//	BTNode *node = new BTNode;
//	node->data = data;
//
//	return node;
//}
//
//
//void InsertLeft(BTNode *parent, BTNode *left)
//{
//	if (parent->left != nullptr)
//		return;
//
//	parent->left = left;
//}
//
//void InsertRight(BTNode *parent, BTNode *right)
//{
//	if (parent->right != nullptr)
//		return;
//
//	parent->right = right;
//}
//
//void Traversal(BTNode *node)
//{
//	cout << "data : " << node->data << endl;
//
//	if (node->left != nullptr)
//		Traversal(node->left);
//	if (node->right != nullptr)
//		Traversal(node->right);
//
//}
//
//void DeleteTree(BTNode *node)
//{
//	if (node->left != nullptr)
//		DeleteTree(node->left);
//
//	if (node->right != nullptr)
//		DeleteTree(node->right);
//
//	delete node;
//}
//
//
//const int BTSize = sizeof(BTNode);
//
//BTNode* BTHostToDevice(BTNode *node)
//{
//	if (node == nullptr)
//		return nullptr;
//
//	node->left = BTHostToDevice(node->left);
//	node->right = BTHostToDevice(node->right);
//
//	BTNode* gnode;
//	cudaMalloc((void**)&gnode, BTSize);
//	cudaMemcpy(gnode, node, BTSize, cudaMemcpyHostToDevice);
//
//	return gnode;
//}
//
//BTNode* BTDeviceToHost(BTNode *node)
//{
//	if (node == nullptr)
//		return nullptr;
//
//	BTNode* cnode = new BTNode;
//	cudaMemcpy(cnode, node, BTSize, cudaMemcpyDeviceToHost);
//
//	cnode->left = BTDeviceToHost(cnode->left);
//	cnode->right = BTDeviceToHost(cnode->right);
//
//	return cnode;
//}
//
//__global__
//void Mull(BTNode *node)
//{
//	if (threadIdx.x == 0)
//		node->data = node->data * 0;
//	else if (threadIdx.x == 1)
//		node->left->data = node->left->data * 1;
//	else if (threadIdx.x == 2)
//		node->right->data = node->right->data * 2;
//}
//
//
//
//int main(int argc, char **argv)
//{
//	BTNode *node = CreateNode(1);
//
//	InsertLeft(node, CreateNode(2));
//	InsertRight(node, CreateNode(3));
//
//	Traversal(node);
//
//	BTNode *gnode = BTHostToDevice(node);
//
//	dim3 dimBlock(3, 1);
//	dim3 dimGrid(1, 1);
//	
//
//	Mull << < dimGrid, dimBlock >> >(gnode);
//	//Traversal(node);
//
//	//cudaMemcpy(node, gnode, BTSize, cudaMemcpyDeviceToHost);
//	/*cudaMemcpy(node->hostRight, gnode->deviceRight, BTSize, cudaMemcpyDeviceToHost);
//	if (gnode->deviceLeft == nullptr)
//		cout << "dd" << endl;*/
//	//cout << gnode->data << endl;
//	//cout << node->hostLeft << endl;
//
//	node = BTDeviceToHost(gnode);
//	Traversal(node);
//
//
//	tmpfunc();
//
//	/*Scene scene = RAYTRACING_SCENE;
//
//	WindowManager::GetInstance()->Initialize();
//
//	WindowManager::GetInstance()->WindowHint(GLFW_SAMPLES, 4);
//	WindowManager::GetInstance()->WindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
//	WindowManager::GetInstance()->WindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
//	WindowManager::GetInstance()->WindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
//	WindowManager::GetInstance()->WindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
//	GLFWwindow* window = WindowManager::GetInstance()->CreateMyWindow(1024, 1024, "OpenGL");
//
//	InputManager::GetInstance()->Initialize(window);
//
//	TO Refacto 어딘가로 버려버릴것
//	glewExperimental = true;
//
//	if (glewInit() != GLEW_OK)
//	{
//	fprintf(stderr, "Failed to initialize GLEW\n");
//	return 0;
//	}
//
//	Renderer* renderer;
//	SceneManager* sceneManager;
//
//	switch (scene)
//	{
//	case RAYTRACING_SCENE:
//	{
//	sceneManager = new RayTracingSceneManager();
//	renderer = new RayTracingRenderer(sceneManager);
//	break;
//	}
//	case FLUID_SCENE:
//	{
//	sceneManager = new FluidSceneManager();
//	renderer = new FluidRenderer(sceneManager);
//	break;
//	}
//	case LTC_SCENE:
//	{
//	sceneManager = new LTCSceneManager();
//	renderer = new LTCRenderer(sceneManager);
//	break;
//	}
//	case PBR_SCENE:
//	{
//	sceneManager = new PBRSceneManager();
//	renderer = new PBRRenderer(sceneManager);
//	break;
//	}
//	default:
//	sceneManager = new RayTracingSceneManager();
//	renderer = new RayTracingRenderer(sceneManager);
//	break;
//	}
//
//	sceneManager->InitializeObjects();
//
//	renderer->Initialize(window);
//	renderer->InitializeRender();
//
//	do
//	{
//	sceneManager->Update();
//	renderer->Render();
//	InputManager::GetInstance()->PollEvents();
//	glfwSwapBuffers(window);
//	}
//	while (InputManager::GetInstance()->IsKey(GLFW_KEY_ESCAPE) != GLFW_PRESS &&
//	!WindowManager::GetInstance()->WindowShouldClose());
//
//	renderer->TerminateRender();
//
//	delete renderer;
//	delete sceneManager;
//
//	WindowManager::GetInstance()->Terminate();
//
//
//
//	return 0;*/
//}
//
////3 channel
////albedo, microsurface, metallic
////refracted light(diffuse)
////depth and color would be very
////
////metallic(conductvie) reflect many lights and energy less lost
////dielectric(insulation) refract many lights lost many energy and wide
////
////refracted light는 물체의 성질에 따른 것이다.
////따라서 albedo에서 정해진다.
////안에 전자가 많을 경우 refracted light의 비중이 줄어들고(metallic)
////안에 전자가 적을 경우 refracted light의 비중이 높아진다.
////
////reflected light는 빛 자체의 색만을 가진다,
////왜냐면 refracted light는 물체에서 색깔이 흡수되지만 reflected light는 아니기 때문이다.
////roughness channel이나 glossiness channel로 표현한다.
////roughness channel에서는 하얀색일수록 거칠다는 뜻이다.
////smoothness