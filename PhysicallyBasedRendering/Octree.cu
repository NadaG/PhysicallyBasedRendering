#include "hip/hip_runtime.h"
#include "Octree.cuh"
#include "Model.h"
#include <algorithm>

using std::min;
using std::max;

void tmpfunc()
{
	Model model;
	model.Load("Obj/torus.obj");

	vector<Triangle> triangles = model.GetTriangles();
	for (int i = 0; i < triangles.size(); i++)
	{
		cout << triangles[i].v0.x << endl;
	}

}

const int OTSize = sizeof(OctreeNode);

OctreeNode* OTHostToDevice(OctreeNode* root)
{
	if (root == nullptr)
		return nullptr;

	for (int i = 0; i < 8; i++)
		root->children[i] = OTHostToDevice(root->children[i]);

	int* gtriangleIdxData;
	hipMalloc((void**)&gtriangleIdxData, sizeof(int)*root->triangleIdx.size());
	hipMemcpy(gtriangleIdxData, root->triangleIdx.data, sizeof(int)*root->triangleIdx.size(), hipMemcpyHostToDevice);

	root->triangleIdx.data = gtriangleIdxData;

	OctreeNode* gnode;
	hipMalloc((void**)&gnode, OTSize);
	hipMemcpy(gnode, root, OTSize, hipMemcpyHostToDevice);

	//gnode->triangleIdx.data = gtriangleIdxData;

	return gnode;
}

void Subdivide(OctreeNode* root)
{
	for (int i = 0; i < 8; i++)
		root->children[i] = new OctreeNode;

	//	left top back
	root->children[0]->bnd.bounds[0].x = root->bnd.bounds[0].x;
	root->children[0]->bnd.bounds[1].x = (root->bnd.bounds[0].x + root->bnd.bounds[1].x) / 2;
	root->children[0]->bnd.bounds[0].y = (root->bnd.bounds[0].y + root->bnd.bounds[1].y) / 2;
	root->children[0]->bnd.bounds[1].y = root->bnd.bounds[1].y;
	root->children[0]->bnd.bounds[0].z = root->bnd.bounds[0].z;
	root->children[0]->bnd.bounds[1].z = (root->bnd.bounds[0].z + root->bnd.bounds[1].z) / 2;
	//	right top back
	root->children[1]->bnd.bounds[0].x = (root->bnd.bounds[0].x + root->bnd.bounds[1].x) / 2;
	root->children[1]->bnd.bounds[1].x = root->bnd.bounds[1].x; 
	root->children[1]->bnd.bounds[0].y = (root->bnd.bounds[0].y + root->bnd.bounds[1].y) / 2;
	root->children[1]->bnd.bounds[1].y = root->bnd.bounds[1].y;
	root->children[1]->bnd.bounds[0].z = root->bnd.bounds[0].z;
	root->children[1]->bnd.bounds[1].z = (root->bnd.bounds[0].z + root->bnd.bounds[1].z) / 2;
	//	left bottom back
	root->children[2]->bnd.bounds[0].x = root->bnd.bounds[0].x;
	root->children[2]->bnd.bounds[1].x = (root->bnd.bounds[0].x + root->bnd.bounds[1].x) / 2;
	root->children[2]->bnd.bounds[0].y = root->bnd.bounds[0].y;
	root->children[2]->bnd.bounds[1].y = (root->bnd.bounds[0].y + root->bnd.bounds[1].y) / 2;
	root->children[2]->bnd.bounds[0].z = root->bnd.bounds[0].z;
	root->children[2]->bnd.bounds[1].z = (root->bnd.bounds[0].z + root->bnd.bounds[1].z) / 2;
	//	right bottom back
	root->children[3]->bnd.bounds[0].x = (root->bnd.bounds[0].x + root->bnd.bounds[1].x) / 2;
	root->children[3]->bnd.bounds[1].x = root->bnd.bounds[1].x;
	root->children[3]->bnd.bounds[0].y = root->bnd.bounds[0].y; 
	root->children[3]->bnd.bounds[1].y = (root->bnd.bounds[0].y + root->bnd.bounds[1].y) / 2;
	root->children[3]->bnd.bounds[0].z = root->bnd.bounds[0].z;
	root->children[3]->bnd.bounds[1].z = (root->bnd.bounds[0].z + root->bnd.bounds[1].z) / 2;
 

	//	left top front
	root->children[4]->bnd.bounds[0].x = root->bnd.bounds[0].x;
	root->children[4]->bnd.bounds[1].x = (root->bnd.bounds[0].x + root->bnd.bounds[1].x) / 2;
	root->children[4]->bnd.bounds[0].y = (root->bnd.bounds[0].y + root->bnd.bounds[1].y) / 2;
	root->children[4]->bnd.bounds[1].y = root->bnd.bounds[1].y;
	root->children[4]->bnd.bounds[0].z = (root->bnd.bounds[0].z + root->bnd.bounds[1].z) / 2;
	root->children[4]->bnd.bounds[1].z = root->bnd.bounds[1].z; 
	//	right top front
	root->children[5]->bnd.bounds[0].x = (root->bnd.bounds[0].x + root->bnd.bounds[1].x) / 2;
	root->children[5]->bnd.bounds[1].x = root->bnd.bounds[1].x;
	root->children[5]->bnd.bounds[0].y = (root->bnd.bounds[0].y + root->bnd.bounds[1].y) / 2;
	root->children[5]->bnd.bounds[1].y = root->bnd.bounds[1].y;
	root->children[5]->bnd.bounds[0].z = (root->bnd.bounds[0].z + root->bnd.bounds[1].z) / 2;
	root->children[5]->bnd.bounds[1].z = root->bnd.bounds[1].z; 
	//	left bottom front
	root->children[6]->bnd.bounds[0].x = root->bnd.bounds[0].x;
	root->children[6]->bnd.bounds[1].x = (root->bnd.bounds[0].x + root->bnd.bounds[1].x) / 2;
	root->children[6]->bnd.bounds[0].y = root->bnd.bounds[0].y;
	root->children[6]->bnd.bounds[1].y = (root->bnd.bounds[0].y + root->bnd.bounds[1].y) / 2;
	root->children[6]->bnd.bounds[0].z = (root->bnd.bounds[0].z + root->bnd.bounds[1].z) / 2;
	root->children[6]->bnd.bounds[1].z = root->bnd.bounds[1].z; 
	//	right bottom front
	root->children[7]->bnd.bounds[0].x = (root->bnd.bounds[0].x + root->bnd.bounds[1].x) / 2;
	root->children[7]->bnd.bounds[1].x = root->bnd.bounds[1].x;
	root->children[7]->bnd.bounds[0].y = root->bnd.bounds[0].y;
	root->children[7]->bnd.bounds[1].y = (root->bnd.bounds[0].y + root->bnd.bounds[1].y) / 2;
	root->children[7]->bnd.bounds[0].z = (root->bnd.bounds[0].z + root->bnd.bounds[1].z) / 2;
	root->children[7]->bnd.bounds[1].z = root->bnd.bounds[1].z; 
}

void DeleteOctree(OctreeNode *root)
{
	for (int i = 0; i < 8; i++)
	{
		if (root->children[i] != nullptr)
			Subdivide(root->children[i]);
	}

	delete root;
}

bool TriangleExist(OctreeNode* node, Triangle triangle)
{
	vec3 o;		//	삼각형의 중심
	o.x = (triangle.v0.x + triangle.v1.x + triangle.v2.x) / 3;
	o.y = (triangle.v0.y + triangle.v1.y + triangle.v2.y) / 3;
	o.z = (triangle.v0.z + triangle.v1.z + triangle.v2.z) / 3;

	float a = length(o - triangle.v0);
	float b = length(o - triangle.v1);
	float c = length(o - triangle.v2);

	//	중심과 꼭지점과의 거리 중 가장 큰 것을 bounding sphere의 반지름으로 계산한다
	float rad = std::max(a, b);
	rad = std::max(rad, c);

	vec3 bo;	//	node의 중심
	bo.x = (node->bnd.bounds[0].x + node->bnd.bounds[1].x) / 2;
	bo.y = (node->bnd.bounds[0].y + node->bnd.bounds[1].y) / 2;
	bo.z = (node->bnd.bounds[0].z + node->bnd.bounds[1].z) / 2;

	if (length(bo - o) > rad + length(node->bnd.bounds[1] - bo) + 0.001f)
		return false;
	else
		return true;

	if (node->bnd.bounds[1].x < triangle.tbb.bounds[0].x || node->bnd.bounds[0].x > triangle.tbb.bounds[1].x) return false;
	if (node->bnd.bounds[1].y < triangle.tbb.bounds[0].y || node->bnd.bounds[0].y > triangle.tbb.bounds[1].y) return false;
	if (node->bnd.bounds[1].z < triangle.tbb.bounds[0].z || node->bnd.bounds[0].z > triangle.tbb.bounds[1].z) return false;
	return true;

}

void SpaceDivision(OctreeNode* root, Triangle* triangles, Ovector* idx, int limit)
{
	Ovector *newIdx = new Ovector;

	for (int i = 0; i < idx->size(); i++)
	{
		if (TriangleExist(root, triangles[idx->operator[](i)]))
			newIdx->push_back(idx->operator[](i));
	}

	if (newIdx->size() > limit)
	{
		//cout << newIdx->size() << endl;
		Subdivide(root);

		for (int i = 0; i < 8; i++)
			SpaceDivision(root->children[i], triangles, newIdx, limit);
	}
	else if (newIdx->size() > 0)
	{
		root->triangleIdx = *newIdx;
	}
}


//void SpaceDivision(OctreeNode* root, vector<Triangle> triangles, int limit)
//{
//	vector<Triangle> *newTri = new vector<Triangle>();
//
//	for (int i = 0; i < triangles.size(); i++)
//	{
//		if (TriangleExist(root, triangles[i]))
//			newTri->push_back(triangles[i]);
//	}
//
//	if (newTri->size() > limit)
//	{
//		Subdivide(root);
//
//		for (int i = 0; i < 8; i++)
//			SpaceDivision(root->children[i], *newTri, limit);
//	}
//	else if (newTri->size() > 0)
//	{
//		root->triangles = *newTri;
//	}
//}

//void SpaceDivision(OctreeNode* root, Triangle* triangles, int numTriangles, int maxTriangles)
//{
//	Triangle* tri = (Triangle*)malloc(sizeof(Triangle) * numTriangles);
//
//	int index = 0;
//	for (int i = 0; i < numTriangles; i++)
//	{
//		if (TriangleExist(root, triangles[i]))
//		{
//			tri[index] = triangles[i];
//			index++;
//		}
//	}
//
//	if (index> maxTriangles)
//	{
//		Subdivide(root);
//
//		for (int i = 0; i < 8; i++)
//			SpaceDivision(root->children[i], *newTri, maxTriangles);
//	}
//	else if (newTri->size() > 0)
//	{
//		root->triangles = *newTri;
//	}
//}


OctreeNode* BuildOctree(Triangle* triangles, int numTriangles, int limit, vec3 min, vec3 max)
{
	OctreeNode* root = new OctreeNode;
	Ovector* idx = new Ovector;

	for (int i = 0; i < numTriangles; i++)
	{
		idx->push_back(i);
	}

	root->bnd.bounds[0] = min;
	root->bnd.bounds[1] = max;

	SpaceDivision(root, triangles, idx, limit);
	cout << "end" << endl;

	return root;
}