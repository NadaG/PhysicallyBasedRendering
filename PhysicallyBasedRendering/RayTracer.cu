#include "hip/hip_runtime.h"
#include "RayTracer.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hip/hip_math_constants.h>
#include <math.h>

struct Ray
{
	float3 origin;
	float3 dir;
};

struct Sphere
{
	float3 origin;
	float radius;
};

struct Triangle
{
	float3 v0;
	float3 v1;
	float3 v2;
};

struct Matrix44
{
	float m[4][4];
};

const int WINDOW_HEIGHT = 1024;
const int WINDOW_WIDTH = 1024;

__device__ float3 operator+(const float3& a, const float3& b)
{
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator-(const float3& a, const float3& b)
{
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ float3 operator-(const float3& a)
{
	return make_float3(-a.x, -a.y, -a.z);
}

__device__ float3 operator*(const float3& a, const float& b)
{
	return make_float3(a.x * b, a.y * b, a.z * b);
}

__device__ float3 operator*(const float& a, const float3& b)
{
	return b * a;
}

__device__ float4 operator*(const float4& a, const Matrix44& mat)
{
	return make_float4(
		mat.m[0][0] * a.x + mat.m[0][1] * a.y + mat.m[0][2] * a.z + mat.m[0][3] * a.w,
		mat.m[1][0] * a.x + mat.m[1][1] * a.y + mat.m[1][2] * a.z + mat.m[1][3] * a.w,
		mat.m[2][0] * a.x + mat.m[2][1] * a.y + mat.m[2][2] * a.z + mat.m[2][3] * a.w,
		mat.m[3][0] * a.x + mat.m[3][1] * a.y + mat.m[3][2] * a.z + mat.m[3][3] * a.w);
}

__device__ float mymax(const float x, const float y)
{
	return x > y ? x : y;
}

__device__ float dot(const float3& a, const float3& b)
{
	return a.x*b.x + a.y*b.y + a.z*b.z;
}

__device__ float3 cross(const float3& a, const float3& b)
{
	return make_float3(a.y*b.z - a.z*b.y, -(a.x*b.z - a.z*b.x), a.x*b.y - a.y*b.x);
}

__device__ float3 reflect(const float3& i, const float3& n)
{
	return i - 2.0f * dot(n, i) * n;
}

__device__ float3 normalize(float3 v)
{
	float3 normalizedV;
	float length = (float)sqrt(dot(v, v));
	normalizedV.x = v.x / length;
	normalizedV.y = v.y / length;
	normalizedV.z = v.z / length;
	return normalizedV;
}

__device__ Matrix44 operator-(const Matrix44& mat)
{
}

__device__ bool RaySphereIntersect(Ray ray, Sphere sphere, float& dist)
{
	float3 s = ray.origin - sphere.origin;

	float a = dot(ray.dir, ray.dir);
	float bPrime = dot(s, ray.dir);
	float c = dot(s, s) - sphere.radius * sphere.radius;

	float D = bPrime * bPrime - a * c;
	if (D >= 0 && bPrime <= 0)
	{
		float t1 = (-bPrime + sqrt(D)) / a;
		float t2 = (-bPrime - sqrt(D)) / a;
		dist = t1 > t2 ? t2 : t1;
		return true;
	}
	else
		return false;
}

__device__ bool RayTriangleIntersect(Ray ray, Triangle triangle, float& dist)
{
	float3 v0v1 = triangle.v1 - triangle.v0;
	float3 v0v2 = triangle.v2 - triangle.v0;
	float3 pvec = cross(ray.dir, v0v2);

	float det = dot(v0v1, pvec);

	float epsilon = 0.0001f;

	if (det < epsilon) 
		return false;
	
	if (fabs(det) < epsilon)
		return false;

	float invDet = 1 / det;

	float3 tvec = ray.origin - triangle.v0;
	float u = dot(tvec, pvec) * invDet;
	if (u < 0 || u > 1)
		return false;

	float3 qvec = cross(tvec, v0v1);
	float v = dot(ray.dir, qvec) * invDet;
	if (v < 0 || u + v > 1)
		return false;

	float t = dot(v0v2, qvec) * invDet;

	return true;
}

// Camera functions
///////////////////
__device__ Ray GenerateCameraRay(int y, int x)
{
	Ray ray;

	float NDCy = (y + 0.5f) / WINDOW_HEIGHT;
	float NDCx = (x + 0.5f) / WINDOW_WIDTH;

	float aspectRatio = WINDOW_WIDTH / WINDOW_HEIGHT;

	float fov = 45.0f;

	float xx = (((float)(x + 0.5f) / (float)WINDOW_WIDTH) * 2.0f - 1.0f) * 
		tan(fov / 2 * 3.141592653f / 180.0f) * aspectRatio;
	float yy = (1.0f - ((float)(y + 0.5f) / (float)WINDOW_HEIGHT) * 2.0f) * tan(fov / 2 * 3.141592653f / 180.0f);

	//// -1 ~ 1
	ray.origin = make_float3(0.0f, 0.0f, 0.0f);
	ray.dir = normalize(make_float3(xx, yy, -1.0));

	//// view matrix�� translate ���и� ������
	//ray.origin = (-view*vec4(vec3(0.0), 1)).xyz;
	//// view matrix�� rotate ������ ������
	//ray.dir = normalize((view*vec4(ray.dir, 0)).xyz);
	//// view matrix�� camera ������ x, y, z���� column���� �α� ������(normalize��) scale ������ �ǹ̰� ���� 

	return ray;
}

__global__ void RayTraceD(float4* data, Matrix44 mat)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	Ray ray = GenerateCameraRay(blockIdx.x, threadIdx.x);

	Sphere sphere;
	float4 translatedPos = make_float4(0.0f, 0.0f, -2.0f, 1.0f);
	sphere.origin = make_float3(translatedPos.x, translatedPos.y, translatedPos.z);
	sphere.radius = 0.1f;

	Sphere sphere2;
	sphere2.origin = make_float3(10.0f, 0.0f, -20.0f);
	sphere2.radius = 5.0f;

	Triangle triangle;
	triangle.v0 = make_float3(0.0f, 0.0f, -5.0f);
	triangle.v1 = make_float3(1.0f, 0.0f, -5.0f);
	triangle.v2 = make_float3(1.0f, 1.0f, -5.0f);

	float distToSphere, distToTriangle;

	float3 lightPos = make_float3(10.0f, 0.0f, 0.0f);

	if (RaySphereIntersect(ray, sphere, distToSphere))
	{
		float3 hitPoint = ray.origin + ray.dir * distToSphere;
		float3 L = normalize(lightPos - hitPoint);
		float3 N = normalize(hitPoint - sphere.origin);

		float3 ambient = make_float3(0.2, 0.2, 0.2);

		float3 diffuse = make_float3(0.1, 0.4, 0.2) * mymax(0, dot(N, L));

		float3 V = -ray.dir;

		float3 specular = make_float3(0.1, 0.4, 0.2) * mymax(0, pow(mymax(dot(normalize(reflect(-L, N)), V), 0.0), 16));

		float3 col = ambient + diffuse + specular;

		data[x] = make_float4(col.x, col.y, col.z, 1.0f);
	}
	else if (RaySphereIntersect(ray, sphere2, distToSphere))
	{
		float3 hitPoint = ray.origin + ray.dir * distToSphere;
		float3 L = normalize(lightPos - hitPoint);
		float3 N = normalize(hitPoint - sphere2.origin);

		float3 ambient = make_float3(0.2, 0.2, 0.2);

		float3 diffuse = make_float3(0.1, 0.4, 0.2) * mymax(0, dot(N, L));

		float3 V = -ray.dir;

		float3 specular = make_float3(0.1, 0.4, 0.2) * mymax(0, pow(mymax(dot(normalize(reflect(-L, N)), V), 0.0), 16));

		float3 col = ambient + diffuse + specular;

		data[x] = make_float4(col.x, col.y, col.z, 1.0f);
	}
	else if (RayTriangleIntersect(ray, triangle, distToTriangle))
	{
		data[x] = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
	}
	else
	{
		data[x] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
	}

	
}

void RayTrace(float4* data, Matrix44 mat)
{
	RayTraceD << <WINDOW_HEIGHT, WINDOW_WIDTH >> > (data, mat);
}