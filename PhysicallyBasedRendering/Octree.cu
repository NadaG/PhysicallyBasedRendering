#include "hip/hip_runtime.h"
#include "Octree.cuh"
#include <algorithm>

using std::min;
using std::max;

__global__ void BuildOctreeD(Triangle* triangles, int triangleNum)
{
}

OctreeNode* HostToDevice(OctreeNode* root)
{
	OctreeNode *h_root, *d_root;
	h_root = new OctreeNode[1];

	memcpy(h_root, root, sizeof(OctreeNode));

	for (int i = 0; i < 8; i++)
	{
		hipMalloc(&(h_root[0].children[i]), sizeof(OctreeNode));
		hipMemcpy(h_root[0].children[i], root[0].children[i], sizeof(OctreeNode), hipMemcpyHostToDevice);
	}

	hipMalloc((void**)&d_root, sizeof(OctreeNode));
	hipMemcpy(d_root, h_root, sizeof(OctreeNode), hipMemcpyHostToDevice);

	return d_root;
}

// host memory�� triangle�� �̿��� octree�� build�ϰ� 
// build�� octree�� device memory�� �Ű� return�ϴ� �Լ�
OctreeNode* BuildOctree(const vector<Triangle>& triangles)
{	
	OctreeNode* root;
	root = new OctreeNode;
	for (int i = 0; i < 8; i++)
	{
		root->children[i] = new OctreeNode;
	}

	root->bnd.bounds[0] = glm::vec3(0.0f, -1.0f, -1.0f);
	root->bnd.bounds[1] = glm::vec3(1.0f, 1.0f, 1.0f);

	return HostToDevice(root);
}

bool IsInNode(OctreeNode* node, Triangle triangle)
{
	AABB triangleAABB;

	triangleAABB.bounds[0].x = min(min(triangle.v0.x, triangle.v1.x), triangle.v2.x);
	triangleAABB.bounds[0].y = min(min(triangle.v0.y, triangle.v1.y), triangle.v2.y);
	triangleAABB.bounds[0].z = min(min(triangle.v0.z, triangle.v1.z), triangle.v2.z);

	triangleAABB.bounds[1].x = max(max(triangle.v0.x, triangle.v1.x), triangle.v2.x);
	triangleAABB.bounds[1].y = max(max(triangle.v0.y, triangle.v1.y), triangle.v2.y);
	triangleAABB.bounds[1].z = max(max(triangle.v0.z, triangle.v1.z), triangle.v2.z);

	if (triangleAABB.bounds[0].x > node->bnd.bounds[1].x) return false;
	if (triangleAABB.bounds[1].x < node->bnd.bounds[0].x) return false;
	if (triangleAABB.bounds[0].y > node->bnd.bounds[1].y) return false;
	if (triangleAABB.bounds[1].y < node->bnd.bounds[0].y) return false;
	if (triangleAABB.bounds[0].z > node->bnd.bounds[1].z) return false;
	if (triangleAABB.bounds[1].z < node->bnd.bounds[0].z) return false;

	return true;
}