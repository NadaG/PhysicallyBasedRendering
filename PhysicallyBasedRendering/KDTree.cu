#include "hip/hip_runtime.h"
﻿#include "KDTree.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm\glm.hpp>
#include <glm\gtx\component_wise.hpp>
#include <stdio.h>
#include <glm\gtc\matrix_transform.hpp>
#include <hip/hip_math_constants.h>
#include <math.h>
#include <thrust\device_vector.h>
#include <queue>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <ctime>
#include <stdio.h>


__global__ void ComputeAABB(Triangle* T, int triangleNum)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= triangleNum)
		return;

	T[idx].tbb.bounds[0].x = glm::min(T[idx].v2.x, glm::min(T[idx].v0.x, T[idx].v1.x));
	T[idx].tbb.bounds[1].x = glm::max(T[idx].v2.x, glm::max(T[idx].v0.x, T[idx].v1.x));

	T[idx].tbb.bounds[0].y = glm::min(T[idx].v2.y, glm::min(T[idx].v0.y, T[idx].v1.y));
	T[idx].tbb.bounds[1].y = glm::max(T[idx].v2.y, glm::max(T[idx].v0.y, T[idx].v1.y));

	T[idx].tbb.bounds[0].z = glm::min(T[idx].v2.z, glm::min(T[idx].v0.z, T[idx].v1.z));
	T[idx].tbb.bounds[1].z = glm::max(T[idx].v2.z, glm::max(T[idx].v0.z, T[idx].v1.z));
}

KDTreeNode* BuildKDTree(const vector<Triangle>& T)
{
	thrust::device_vector<Triangle> t = T;
	int block = t.size() / 1024 + 1;

	ComputeAABB << < block, 1024 >> > (t.data().get(), t.size());
	cout << "Compute Tri AABB" << endl;

	//// node list
	//Nvector* nodeList = new Nvector();
	//hipMalloc((void**)&(nodeList->nodeTriangleList), sizeof(int)*t.size());

	//Nvector* deviceNodeList;
	//hipMalloc((void**)&deviceNodeList, sizeof(Nvector));
	//hipMemcpy(deviceNodeList, nodeList, sizeof(Nvector), hipMemcpyHostToDevice);

	//// active list
	//Nvector* activeList = new Nvector();
	//hipMalloc((void**)&(activeList->nodeTriangleList), sizeof(int)*t.size());

	//Nvector* deviceActiveList;
	//hipMalloc((void**)&deviceActiveList, sizeof(Nvector));
	//hipMemcpy(deviceActiveList, activeList, sizeof(Nvector), hipMemcpyHostToDevice);

	//// small list
	//Nvector* smallList = new Nvector();
	//hipMalloc((void**)&(smallList->nodeTriangleList), sizeof(int)*t.size());

	//Nvector* deviceSmallList;
	//hipMalloc((void**)&deviceSmallList, sizeof(Nvector));
	//hipMemcpy(deviceSmallList, smallList, sizeof(Nvector), hipMemcpyHostToDevice);

	//// next list
	//Nvector* nextList = new Nvector();
	//hipMalloc((void**)&(nextList->nodeTriangleList), sizeof(int)*t.size());

	//Nvector* deviceNextList;
	//hipMalloc((void**)&deviceNextList, sizeof(Nvector));
	//hipMemcpy(deviceNextList, nextList, sizeof(Nvector), hipMemcpyHostToDevice);


	//KDTreeNode* root;
	//hipMalloc((void**)&root, sizeof(KDTreeNode));
	
	Nvector* nodeList = new Nvector();
	nodeList->nodeTriangleList = new int[t.size()];

	Nvector* activeList = new Nvector();
	activeList->nodeTriangleList = new int[t.size()];

	Nvector* smallList = new Nvector();
	smallList->nodeTriangleList = new int[t.size()];

	Nvector* nextList = new Nvector();
	nextList->nodeTriangleList = new int[t.size()];

	KDTreeNode* root = new KDTreeNode();
	root->firstTriangle = 0;
	root->triangleNum = t.size();
	root->bnd.bounds[0] = vec3(-60, -60, -60);
	root->bnd.bounds[1] = vec3(60, 60, 60);
	root->chunkSize = t.size() / CHUNKSIZE + 1;

	activeList->push_back(*root);
	for (int i = 0; i < t.size(); i++)
	{
		activeList->nodeTriangleList[i] = i;
	}

	// Large node stage
	int aa = 1;
	while (!activeList->empty() && aa == 1)
	{
		nodeList->append(activeList);
		delete nextList;
		nextList = new Nvector();

		ProcessLargeNodes(activeList, smallList, nextList, t.data().get(), t.size());

		/*Nvector* tmp = activeList;
		activeList = nextList;
		nextList = tmp;*/

		aa = 0;
	}
	

	cout << "KDend" << endl;
	
	delete nodeList;
	delete activeList;
	delete smallList;
	delete nextList;
	delete root;


	return nullptr;
}

__global__ void ChunkingTriangle(Nvector* activeList, ChunkNode* chunkList)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < activeList->size())
	{
		int size = activeList->operator[](idx).chunkSize;

		for (int i = 0; i < size; i++)
		{
			ChunkNode newChunk;
			newChunk.node = &activeList->operator[](idx);
			newChunk.firstTriangle = activeList->operator[](idx).firstTriangle + i*CHUNKSIZE;
			//newChunk.firstTriangle = 100;

			if (i == size - 1)
				newChunk.triangleNum = activeList->operator[](idx).triangleNum - CHUNKSIZE*i;
			else
				newChunk.triangleNum = CHUNKSIZE;

			// 자신보다 index가 앞인 노드들의 chunk 수를 모두 더해서 chunk list에서 현재 노드의 시작 위치를 알아낸다.
			int startIdx = 0;
			for (int j = 0; j < idx; j++)
			{
				startIdx += activeList->operator[](j).chunkSize;
			}

			chunkList[startIdx + i] = newChunk;
	
			//chunkList[0].triangleNum = size;
		}
	}
}


__global__ void ComputeChunkAABB(ChunkNode* chunkList, int chunkNum, Triangle* T)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < chunkNum)
	{
		vec3 max = { -99999,-99999,-99999 };
		vec3 min = { 99999,99999,99999 };

		for (int i = 0; i < CHUNKSIZE; i++)
		{
			chunkList[idx].cbb.bounds[0].x = thrust::min(min.x, T[chunkList[idx].firstTriangle + i].tbb.bounds[0].x);
			chunkList[idx].cbb.bounds[0].y = thrust::min(min.y, T[chunkList[idx].firstTriangle + i].tbb.bounds[0].y);
			chunkList[idx].cbb.bounds[0].z = thrust::min(min.z, T[chunkList[idx].firstTriangle + i].tbb.bounds[0].z);

			chunkList[idx].cbb.bounds[1].x = thrust::max(max.x, T[chunkList[idx].firstTriangle + i].tbb.bounds[1].x);
			chunkList[idx].cbb.bounds[1].y = thrust::max(max.y, T[chunkList[idx].firstTriangle + i].tbb.bounds[1].y);
			chunkList[idx].cbb.bounds[1].z = thrust::max(max.z, T[chunkList[idx].firstTriangle + i].tbb.bounds[1].z);

			min.x = chunkList[idx].cbb.bounds[0].x;
			min.y = chunkList[idx].cbb.bounds[0].y;
			min.z = chunkList[idx].cbb.bounds[0].z;

			max.x = chunkList[idx].cbb.bounds[1].x;
			max.y = chunkList[idx].cbb.bounds[1].y;
			max.z = chunkList[idx].cbb.bounds[1].z;

		}	
	}
}


__global__ void SegmentedReduction(ChunkNode* chunkList, int gap, int cnum)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx*gap * 2 > cnum - 1 || idx*gap * 2 + gap > cnum-1)
		return;

	if (chunkList[idx*gap * 2].node == chunkList[idx*gap * 2 + gap].node)
	{
		chunkList[idx*gap * 2].cbb.bounds[0].x = thrust::min(chunkList[idx*gap * 2].cbb.bounds[0].x, chunkList[idx*gap * 2 + gap].cbb.bounds[0].x);
		chunkList[idx*gap * 2].cbb.bounds[0].y = thrust::min(chunkList[idx*gap * 2].cbb.bounds[0].y, chunkList[idx*gap * 2 + gap].cbb.bounds[0].y);
		chunkList[idx*gap * 2].cbb.bounds[0].z = thrust::min(chunkList[idx*gap * 2].cbb.bounds[0].z, chunkList[idx*gap * 2 + gap].cbb.bounds[0].z);

		chunkList[idx*gap * 2].cbb.bounds[1].x = thrust::max(chunkList[idx*gap * 2].cbb.bounds[1].x, chunkList[idx*gap * 2 + gap].cbb.bounds[1].x);
		chunkList[idx*gap * 2].cbb.bounds[1].y = thrust::max(chunkList[idx*gap * 2].cbb.bounds[1].y, chunkList[idx*gap * 2 + gap].cbb.bounds[1].y);
		chunkList[idx*gap * 2].cbb.bounds[1].z = thrust::max(chunkList[idx*gap * 2].cbb.bounds[1].z, chunkList[idx*gap * 2 + gap].cbb.bounds[1].z);
	}
	else
	{	
		if (chunkList[idx*gap * 2 + gap].node == chunkList[0].node)
		{
			chunkList[idx*gap * 2].node->tbb.bounds[0].x = thrust::min(chunkList[idx*gap * 2].node->tbb.bounds[0].x, chunkList[idx*gap * 2].cbb.bounds[0].x);
			chunkList[idx*gap * 2].node->tbb.bounds[0].y = thrust::min(chunkList[idx*gap * 2].node->tbb.bounds[0].y, chunkList[idx*gap * 2].cbb.bounds[0].y);
			chunkList[idx*gap * 2].node->tbb.bounds[0].z = thrust::min(chunkList[idx*gap * 2].node->tbb.bounds[0].z, chunkList[idx*gap * 2].cbb.bounds[0].z);

			chunkList[idx*gap * 2].node->tbb.bounds[1].x = thrust::max(chunkList[idx*gap * 2].node->tbb.bounds[1].x, chunkList[idx*gap * 2].cbb.bounds[1].x);
			chunkList[idx*gap * 2].node->tbb.bounds[1].y = thrust::max(chunkList[idx*gap * 2].node->tbb.bounds[1].y, chunkList[idx*gap * 2].cbb.bounds[1].y);
			chunkList[idx*gap * 2].node->tbb.bounds[1].z = thrust::max(chunkList[idx*gap * 2].node->tbb.bounds[1].z, chunkList[idx*gap * 2].cbb.bounds[1].z);

			chunkList[idx*gap * 2] = chunkList[idx*gap * 2 + gap];
		}
		else
		{
			chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].x = thrust::min(chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].x, chunkList[idx*gap * 2 + gap].cbb.bounds[0].x);
			chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].y = thrust::min(chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].y, chunkList[idx*gap * 2 + gap].cbb.bounds[0].y);
			chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].z = thrust::min(chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].z, chunkList[idx*gap * 2 + gap].cbb.bounds[0].z);

			chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].x = thrust::max(chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].x, chunkList[idx*gap * 2 + gap].cbb.bounds[1].x);
			chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].y = thrust::max(chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].y, chunkList[idx*gap * 2 + gap].cbb.bounds[1].y);
			chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].z = thrust::max(chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].z, chunkList[idx*gap * 2 + gap].cbb.bounds[1].z);
		}
	}

	chunkList[0].node->tbb = chunkList[0].cbb;
}


void ProcessLargeNodes(Nvector* activeList, Nvector* smallList, Nvector* nextList, Triangle* T, int triangleNum)
{
	//	copy active list to GPU memory
	int* devNTL;
	hipMalloc((void**)&devNTL, sizeof(int)*triangleNum);
	hipMemcpy(devNTL, activeList->nodeTriangleList, sizeof(int)*triangleNum, hipMemcpyHostToDevice);

	KDTreeNode* devData;
	hipMalloc((void**)&devData, sizeof(KDTreeNode)*activeList->size());
	hipMemcpy(devData, activeList->data, sizeof(KDTreeNode)*activeList->size(), hipMemcpyHostToDevice);

	Nvector* tmp = new Nvector();
	tmp->nodeTriangleList = devNTL;
	tmp->data = devData;
	tmp->capacity = activeList->capacity;
	tmp->sz = activeList->sz;

	Nvector* devActiveList;
	hipMalloc((void**)&devActiveList, sizeof(Nvector));
	hipMemcpy(devActiveList, tmp, sizeof(Nvector), hipMemcpyHostToDevice);

	

	///////////////////////////////////////////////
	//	1st step, group triangles into chunks

	ChunkNode* chunkList;

	//	active list에 존재하는 모든 chunk의 개수를 구한다.
	int cnum = 0;
	for (int i = 0; i < activeList->size(); i++)
	{
		cnum += activeList->operator[](i).chunkSize;
	}
	hipMalloc((void**)&chunkList, sizeof(ChunkNode)*cnum);
	
	int block = activeList->size();

	ChunkingTriangle << < block, 1 >> > (devActiveList, chunkList);

	
	///////////////////////////////////////////////


	///////////////////////////////////////////////
	//	2nd step, compute per-node bounding box

	ComputeChunkAABB << < cnum, 1 >> > (chunkList, cnum, T);
	
	int a = 0;
	if (cnum % 2 == 0)
		a = cnum / 2;
	else
		a = cnum / 2 + 1;
	for (int gap = 0; gap < a; gap++)
	{
		SegmentedReduction << < 1, cnum >> > (chunkList, pow(2, gap), cnum);
	}

	///////////////////////////////////////////////


	///////////////////////////////////////////////
	//	3rd step, split large node

	//SplitLargeNode << < 1, 1 >> > ();

	///////////////////////////////////////////////





	ChunkNode* hostList = new ChunkNode[cnum];
	hipMemcpy(hostList, chunkList, sizeof(ChunkNode)*cnum, hipMemcpyDeviceToHost);

	cout << "idx " <<hostList[33].triangleNum << endl;

	for (int i = 0; i < 34; i++)
	{
		//cout << "AABB " << hostList[i].cbb.bounds[1].y << endl;
	}

	
	//hipMemcpy(hostList, chunkList, sizeof(ChunkNode)*cnum, hipMemcpyDeviceToHost);
	KDTreeNode* node11 = new KDTreeNode();
	hipMemcpy(node11, hostList[0].node, sizeof(KDTreeNode), hipMemcpyDeviceToHost);

	//cout << "==========================" << endl;
	//cout << "AABB " << hostList[0].cbb.bounds[0].x << endl;
	//cout << "AABB " << hostList[0].cbb.bounds[0].y << endl;
	//cout << "AABB " << hostList[0].cbb.bounds[0].z << endl;

	//cout << "AABB "<<hostList[0].cbb.bounds[1].x << endl;
	//cout << "AABB " << hostList[0].cbb.bounds[1].y << endl;
	//cout << "AABB " << hostList[0].cbb.bounds[1].z << endl;

	cout << "==========================" << endl;
	cout << "AABB " << node11->tbb.bounds[0].x << endl;
	cout << "AABB " << node11->tbb.bounds[0].y << endl;
	cout << "AABB " << node11->tbb.bounds[0].z << endl;

	cout << "AABB " << node11->tbb.bounds[1].x << endl;
	cout << "AABB " << node11->tbb.bounds[1].y << endl;
	cout << "AABB " << node11->tbb.bounds[1].z << endl;

	hipFree(chunkList);
	delete hostList;
	hipFree(devNTL);
	hipFree(devData);
	hipFree(devActiveList);

	tmp->nodeTriangleList = nullptr;
	tmp->data = nullptr;
	delete tmp;
}  