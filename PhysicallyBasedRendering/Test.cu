#include "hip/hip_runtime.h"
#include "Test.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void makernel(float* data)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	data[i] = 7.0f;
}

int hello(float* data)
{
	float* ddata;
	hipMalloc((void**)&ddata, 3 * sizeof(float));

	makernel << <1, 3 >> > (ddata);

	hipMemcpy(data, ddata, 3 * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(ddata);
	hipDeviceSynchronize();
	return 0;
}