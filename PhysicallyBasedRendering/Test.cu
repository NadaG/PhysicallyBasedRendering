#include "hip/hip_runtime.h"
#include "Test.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void TestFunctionD(float* data)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	data[i] = 7.0f;
}

int TestFunction(float* data)
{
	float* ddata;
	hipMalloc((void**)&ddata, 3 * sizeof(float));

	TestFunctionD << <1, 3 >> > (ddata);

	hipMemcpy(data, ddata, 3 * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(ddata);
	hipDeviceSynchronize();
	return 0;
}

__global__ void pboTestD(float* data)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	switch (x % 4)
	{
	case 0:
		data[x] = 1.0f;
		break;
	case 1:
		data[x] = 0.0f;
		break;
	case 2:
		data[x] = 1.0f;
		break;
	default:
		data[x] = 1.0f;
		break;
	}
}

void pboTest(float* data)
{
	pboTestD << <1024 * 4, 1024 >> > (data);
}