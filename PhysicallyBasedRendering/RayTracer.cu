#include "hip/hip_runtime.h"
﻿#include "RayTracer.cuh"
#include "Octree.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm\glm.hpp>
#include <glm\gtx\component_wise.hpp>
#include <stdio.h>
#include <glm\gtc\matrix_transform.hpp>
#include <hip/hip_math_constants.h>
#include <math.h>
#include <thrust\device_vector.h>
#include <queue>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

struct Ray
{
	// Ray의 원점
	vec3 origin;
	// Ray의 방향
	vec3 dir;
	// 0: primary, 1: reflect, 2: refract
	int rayType;

	float decay;
};

const int WINDOW_HEIGHT = 1024;
const int WINDOW_WIDTH = 1024;

const int RAY_X_NUM = 256;
const int RAY_Y_NUM = 256;

const int QUEUE_SIZE = 8;

using std::cout;
using std::endl;


__device__ vec3 CastRay(vec3 P, vec3 N) 
{
	hiprandState localState;
	hiprand_init(0, 0, 0, &localState);
	for (int n = 0; n < 100; ++n) 
	{
		float theta = (hiprand_uniform(&localState) - 0.5f)*glm::pi<float>();
		float phi = (hiprand_uniform(&localState) - 0.5f)*glm::pi<float>();

	}
}

__device__ vec3 Interpolation(Triangle triangle, vec3 position, vec3& N, vec2& uv)
{
	vec3 v0 = triangle.v1 - triangle.v0;
	vec3 v1 = triangle.v2 - triangle.v0;
	vec3 v2 = position - triangle.v0;

	float d00 = dot(v0, v0);
	float d01 = dot(v0, v1);
	float d11 = dot(v1, v1);
	float d20 = dot(v2, v0);
	float d21 = dot(v2, v1);
	float denom = d00*d11 - d01*d01;

	float v = (d11*d20 - d01*d21) / denom;
	float w = (d00*d21 - d01*d20) / denom;
	float u = 1.0f - v - w;

	N = u * triangle.v0normal + v * triangle.v1normal + w * triangle.v2normal;
	uv = u * triangle.v0uv + v * triangle.v1uv + w * triangle.v2uv;
}

// ray와 sphere가 intersect하는지 검사하는 함수
__device__ bool RaySphereIntersect(Ray ray, Sphere sphere, float& dist)
{
	glm::vec3 s = ray.origin - sphere.origin;
	float minDist = 0.001f;

	float a = dot(ray.dir, ray.dir);
	float bPrime = dot(s, ray.dir);
	float c = dot(s, s) - sphere.radius * sphere.radius;

	float D = bPrime * bPrime - a * c;
	if (D >= 0 && bPrime <= 0)
	{
		float t1 = (-bPrime + sqrt(D)) / a;
		float t2 = (-bPrime - sqrt(D)) / a;
		dist = t1 > t2 ? t2 : t1;
		return dist > minDist;
	}
	else
		return false;
}

// ray와 triangle이 intersect하는지 검사하는 함수
__device__ bool RayTriangleIntersect(Ray ray, Triangle triangle, float& dist)
{
	glm::vec3 v0v1 = triangle.v1 - triangle.v0;
	glm::vec3 v0v2 = triangle.v2 - triangle.v0;
	glm::vec3 pvec = glm::cross(ray.dir, v0v2);

	float det = dot(v0v1, pvec);

	// back face culling
	if (det < 0.01f)
		return false;

	if (fabsf(det) < 0.01f)
		return false;

	float invDet = 1 / det;

	glm::vec3 tvec = ray.origin - triangle.v0;
	float u = glm::dot(tvec, pvec) * invDet;
	if (u < 0 || u > 1)
		return false;

	glm::vec3 qvec = cross(tvec, v0v1);
	float v = dot(ray.dir, qvec) * invDet;
	if (v < 0 || u + v > 1)
		return false;

	dist = dot(v0v2, qvec) * invDet;

	return true;
}

//bool RayPlaneIntersect(Ray ray, vec4 plane, float& t)
//{
//	t = -dot(plane, vec4(ray.origin, 1.0)) / glm::dot(glm::vec3(plane), ray.dir);
//	return t > 0.0;
//}
//
//bool RayRectIntersect(Ray ray, Rect rect, float& t)
//{
//	bool intersect = RayPlaneIntersect(ray, rect.plane, t);
//	if (intersect)
//	{
//		vec3 pos = ray.origin + ray.dir*t;
//		vec3 lpos = pos - rect.center;
//
//		float x = dot(lpos, rect.dirx);
//		float y = dot(lpos, rect.diry);
//
//		if (abs(x) > rect.halfx || abs(y) > rect.halfy)
//			intersect = false;
//	}
//
//	return intersect;
//}

__device__ bool RayAABBIntersect(Ray ray, AABB box)
{
	float tmin, tmax, tymin, tymax, tzmin, tzmax;

	glm::vec3 invdir = 1.0f / ray.dir;
	int sign[3];
	sign[0] = invdir.x < 0;
	sign[1] = invdir.y < 0;
	sign[2] = invdir.z < 0;

	tmin = (box.bounds[sign[0]].x - ray.origin.x) * invdir.x;
	tmax = (box.bounds[1 - sign[0]].x - ray.origin.x) * invdir.x;
	tymin = (box.bounds[sign[1]].y - ray.origin.y) * invdir.y;
	tymax = (box.bounds[1 - sign[1]].y - ray.origin.y) * invdir.y;

	if ((tmin > tymax) || (tymin > tmax))
		return false;

	if (tymin > tmin)
		tmin = tymin;
	if (tymax < tmax)
		tmax = tymax;

	tzmin = (box.bounds[sign[2]].z - ray.origin.z) * invdir.z;
	tzmax = (box.bounds[1 - sign[2]].z - ray.origin.z) * invdir.z;

	if ((tmin > tzmax) || (tzmin > tmax))
		return false;

	return true;
}

__device__ bool RayAABBsIntersect(Ray ray, AABB* boxes, int boxNum)
{
	bool isIntersect = false;

	for (int it = 0; it < boxNum; it++)
	{
		if (RayAABBIntersect(ray, boxes[it]))
			isIntersect = true;
	}
	return isIntersect;
}

// ray의 원점과 가장 가까운 곳에서 intersect하는 triangle의 id를 가져오는 함수
__device__ int FindNearestTriangleIdx(Ray ray, Triangle* triangles, int triangleNum, float& dist)
{
	const float rayThreshold = 0.0001f;
	float minDist = 9999999.0f;
	int minIdx = -1;
	float tmpDist;

	for (int i = 0; i < triangleNum; ++i)
	{
		if (RayTriangleIntersect(ray, triangles[i], tmpDist))
		{
			if (tmpDist > rayThreshold && tmpDist < minDist)
			{
				minDist = tmpDist;
				minIdx = i;
			}
		}
	}

	dist = minDist;
	return minIdx;
}

// ray의 원점과 가장 가까운 곳에서 intersect하는 sphere의 id를 가져오는 함수
__device__ int FindNearestSphereIdx(Ray ray, Sphere* spheres, int sphereNum, float& dist)
{
	const float rayThreshold = 0.0001f;
	float minDist = 9999999.0f;
	int minIdx = -1;
	float tmpDist;

	for (int i = 0; i < sphereNum; ++i)
	{
		if (RaySphereIntersect(ray, spheres[i], tmpDist))
		{
			if (tmpDist > rayThreshold && tmpDist < minDist)
			{
				minDist = tmpDist;
				minIdx = i;
			}
		}
	}

	dist = minDist;
	return minIdx;
}

// window의 픽셀의 위치가 각각 x, y로 입력됨
__device__ Ray GenerateCameraRay(int y, int x, glm::mat4 cameraModelMatrix)
{
	Ray ray;

	// 각 픽셀의 중앙을 가르키는 값 생성, 0~1의 값으로 Normalizing
	float NDCy = (y + 0.5f) / WINDOW_HEIGHT;
	float NDCx = (x + 0.5f) / WINDOW_WIDTH;

	// window 종횡비
	float aspectRatio = WINDOW_WIDTH / WINDOW_HEIGHT;

	// 시야각 설정
	float fov = 45.0f;

	// unProject
	float xx = (NDCx * 2.0f - 1.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f) * aspectRatio;
	float yy = (NDCy * 2.0f - 1.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f);

	// world space에서의 ray 정보를 계산
	ray.origin = glm::vec3(cameraModelMatrix * glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
	ray.dir = normalize(vec3(cameraModelMatrix * vec4(glm::vec3(xx, yy, -1.0), 0.0f)));

	// 만들어진 ray를 return
	return ray;
}

__device__ void Enqueue(Ray* rayQueue, Ray ray, int& rear)
{
	rear = (rear + 1) % QUEUE_SIZE;
	rayQueue[rear] = ray;
}

__device__ void Dequeue(Ray* rayQueue, int& front)
{
	Ray ray = rayQueue[front];
	front = (front + 1) % QUEUE_SIZE;
}

__device__ Ray GetQueueFront(Ray* rayQueue, const int front)
{
	return rayQueue[(front + 1) % QUEUE_SIZE];
}

__device__ bool IsQueueEmpty(const int front, const int rear)
{
	return front == rear;
}

__device__ float Radiance(
	vec3 hitPoint,
	Light light,
	Material* materials,
	Triangle* triangles,
	const int triangleNum,
	const int nearestTriangleIdx,
	Sphere* spheres,
	const int sphereNum,
	const int nearestSphereIdx)
{
	float radiance = 3000.0f;

	// shadow ray 생성, origin은 hit point, 방향은 hit point부터 광원까지의 방향
	Ray shadowRay;
	shadowRay.origin = hitPoint;
	shadowRay.dir = normalize(light.pos - hitPoint);
	float distance = glm::distance(light.pos, hitPoint);

	float distToTriangle;

	for (int t_i = 0; t_i < triangleNum; ++t_i)
	{
		// 처음 hit한 triangle은 제외
		if (nearestTriangleIdx != t_i)
		{
			// shadow
			if (RayTriangleIntersect(shadowRay, triangles[t_i], distToTriangle))
			{
				// 앞쪽의 dir만 봄
				if (distToTriangle > 0.01f && distToTriangle < glm::distance(light.pos, hitPoint))
				{
					radiance *= glm::clamp(materials[triangles[t_i].materialId].refractivity, 0.0f, 1.0f);
				}
			}
		}
	}

	float distToSphere;

	for (int s_i = 0; s_i < sphereNum; ++s_i)
	{
		// 광원은 0임, 광원을 제외한 경우에만 그림자 생김
		if (nearestSphereIdx != s_i && s_i != 0)
		{
			if (RaySphereIntersect(shadowRay, spheres[s_i], distToSphere))
			{
				// 앞쪽의 dir만 봄
				if (distToSphere > 0.01f && distToSphere < glm::distance(light.pos, hitPoint))
				{
					radiance *= glm::clamp(materials[triangles[s_i].materialId].refractivity, 0.0f, 1.0f);
				}
			}
		}
	}

	radiance /= (distance*distance);

	return radiance;
}

// ray가 hit 했다면 true를 리턴하고 hit한 곳의 정보를 가져오는 함수
__device__ bool GetHitPointInfo(
	Ray nowRay,
	Triangle* triangles,
	int triangleNum,
	int& nearestTriangleIdx,
	Sphere* spheres,
	int sphereNum,
	int& nearestSphereIdx,
	vec3& hitPoint, 
	int& materialId, 
	vec3& N,
	vec2& uv)
{
	float distToTriangle, distToSphere, distToAreaLight = 0.0f;
	nearestTriangleIdx = FindNearestTriangleIdx(nowRay, triangles, triangleNum, distToTriangle);
	nearestSphereIdx = FindNearestSphereIdx(nowRay, spheres, sphereNum, distToSphere);

	// 아무곳도 intersect를 못했다거나 뒤쪽에 있다면
	if ((nearestTriangleIdx == -1 || distToTriangle < 0.0f) &&
		(nearestSphereIdx == -1 || distToSphere < 0.0f) &&
		(distToAreaLight <= 0.0f))
		return false;

	if (distToSphere > distToTriangle)
	{
		Triangle nearestTriangle = triangles[nearestTriangleIdx];
		hitPoint = nowRay.origin + nowRay.dir * distToTriangle;
		materialId = nearestTriangle.materialId;
		Interpolation(nearestTriangle, hitPoint, N, uv);
	}
	else
	{
		Sphere nearestSphere = spheres[nearestSphereIdx];
		hitPoint = nowRay.origin + nowRay.dir * distToSphere;
		materialId = nearestSphere.materialId;
		N = glm::normalize(hitPoint - nearestSphere.origin);
		// no uv...
	}

	return true;
}

__device__ vec4 RayTraceColor(
	Ray ray,
	Ray* rayQueue,
	AABB* objects,
	int objNum,
	Triangle* triangles,
	int triangleNum,
	Sphere* spheres,
	int sphereNum,
	Light* lights,
	int lightNum,
	Material* materials,
	int matNum,
	float* textures,
	int texSize,
	int depth)
{
	vec4 color = vec4(0.0f);
	int front = 0, rear = 0;

	Enqueue(rayQueue, ray, rear);

	int nowDepth = 1;
	vec3 V = -ray.dir;

	for (int i = 1; i < depth; ++i)
	{
		int target = rear;

		while (!IsQueueEmpty(target, front))
		{
			Ray nowRay;
			nowRay = GetQueueFront(rayQueue, front);
			Dequeue(rayQueue, front);

			if (!RayAABBsIntersect(nowRay, objects, objNum))
				continue;

			vec4 lightedColor = glm::vec4(0.0f);
			vec3 hitPoint = glm::vec3(0.0f);
			// hit한 object의 material id
			int materialId = 0;
			// normal vector
			vec3 N = glm::vec3(0.0f);
			vec2 uv = glm::vec2(0.0f);
			int nearestTriangleIdx = 0;
			int nearestSphereIdx = 0;

			// hit point의 정보를 가져옴
			if (GetHitPointInfo(
				nowRay, 
				triangles, 
				triangleNum, 
				nearestTriangleIdx, 
				spheres, 
				sphereNum, 
				nearestSphereIdx, 
				hitPoint, 
				materialId, 
				N,
				uv))
			{
				for (int k = 0; k < lightNum; k++)
				{
					// ∫Ω(kd c / π + ks DFG / 4(ωo⋅n)(ωi⋅n)) Li(p,ωi) n⋅ωi dωi

					// radiance * (1.0f * textureColor/pi + 0.0f) * lightcolor * NdotL

					float kd = 1.0f;
					vec3 L = glm::normalize(lights[k].pos - hitPoint);
					float NdotL = glm::clamp(glm::dot(N, L), 0.0f, 1.0f);

					float radiance = Radiance(hitPoint,
						lights[k],
						materials,
						triangles, triangleNum, nearestTriangleIdx,
						spheres, sphereNum, nearestSphereIdx);

					vec3 ambientColor = materials[materialId].ambient;
					vec3 diffuseColor;

					if (materials[materialId].texWidth == 0)
						diffuseColor = materials[materialId].diffuse;
					else
					{
						float u = uv.x * 2048.0f - 0.5f;
						float v = uv.y * 2048.0f - 0.5f;
						int uu = floor(u);
						int vv = floor(v);
						float uRatio = u - uu;
						float vRatio = v - vv;
						float uOpposite = 1 - uRatio;
						float vOpposite = 1 - vRatio;
						
						int uu0 = uu * 2048;
						int uu1 = glm::clamp(uu + 1, 0, 2047) * 2048;

						int vv0 = 2047 - vv;
						int vv1 = 2047 - glm::clamp(vv + 1, 0, 2047);

						float texR =
							(textures[(uu0 + vv0) * 4 + 0] * uOpposite +
								textures[(uu1 + vv0) * 4 + 0] * uRatio) * vOpposite +
								(textures[(uu0 + vv1) * 4 + 0] * uOpposite +
									textures[(uu1 + vv1) * 4 + 0] * uRatio)*vRatio;

						float texG =
							(textures[(uu0 + vv0) * 4 + 1] * uOpposite +
								textures[(uu1 + vv0) * 4 + 1] * uRatio) * vOpposite +
								(textures[(uu0 + vv1) * 4 + 1] * uOpposite +
									textures[(uu1 + vv1) * 4 + 1] * uRatio)*vRatio;

						float texB =
							(textures[(uu0 + vv0) * 4 + 2] * uOpposite +
								textures[(uu1 + vv0) * 4 + 2] * uRatio) * vOpposite +
								(textures[(uu0 + vv1) * 4 + 2] * uOpposite +
									textures[(uu1 + vv1) * 4 + 2] * uRatio)*vRatio;

						diffuseColor = glm::vec3(texR, texG, texB);
					}

					diffuseColor *= kd / glm::pi<float>();
					diffuseColor = vec3(
						diffuseColor.r * lights[k].color.r,
						diffuseColor.g * lights[k].color.g,
						diffuseColor.b * lights[k].color.b);
					lightedColor += glm::vec4(ambientColor + radiance * NdotL * diffuseColor, 1.0f);
				}

				color += lightedColor * nowRay.decay;

				Ray reflectRay;
				// reflect ray의 시작점은 hit point
				reflectRay.origin = hitPoint;
				reflectRay.dir = normalize(reflect(nowRay.dir, N));
				// reflect ray
				reflectRay.rayType = 1;
				// 현재 빛의 감쇠 정도와 물체의 재질에 따라 reflect ray의 감쇠 정도가 정해짐 
				reflectRay.decay = nowRay.decay * materials[materialId].reflectivity;

				Ray refractRay;
				// refract ray의 시작점은 hit point
				refractRay.origin = hitPoint;
				refractRay.dir = normalize(refract(nowRay.dir, N, 0.95f));
				// refract ray
				refractRay.rayType = 2;
				// 현재 빛의 감쇠 정도와 물체의 재질에 따라 refract ray의 감쇠 정도가 정해짐
				refractRay.decay = nowRay.decay * materials[materialId].refractivity;

				if (reflectRay.decay > 0)
				{
					Enqueue(rayQueue, reflectRay, rear);
				}

				if (refractRay.decay > 0)
				{
					Enqueue(rayQueue, refractRay, rear);
				}
			}
		}

		nowDepth++;
	}

	// 나오지 못한 queue들 나오게 하기
	while (!IsQueueEmpty(front, rear))
	{
		Ray nowRay;
		nowRay = GetQueueFront(rayQueue, front);
		Dequeue(rayQueue, front);

		if (!RayAABBsIntersect(nowRay, objects, objNum))
			continue;

		vec4 lightedColor = glm::vec4(0.0f);
		vec3 hitPoint = glm::vec3(0.0f);
		int materialId = 0;
		vec3 N = glm::vec3(0.0f);
		vec2 uv = glm::vec2(0.0f);
		int nearestTriangleIdx = 0;
		int nearestSphereIdx = 0;

		if (GetHitPointInfo(
			nowRay,
			triangles,
			triangleNum,
			nearestTriangleIdx,
			spheres,
			sphereNum,
			nearestSphereIdx,
			hitPoint,
			materialId,
			N,
			uv))
		{
			for (int k = 0; k < lightNum; k++)
			{
				float kd = 1.0f;
				vec3 L = glm::normalize(lights[k].pos - hitPoint);
				float NdotL = glm::clamp(glm::dot(N, L), 0.0f, 1.0f);

				float radiance = Radiance(hitPoint,
					lights[k],
					materials,
					triangles, triangleNum, nearestTriangleIdx,
					spheres, sphereNum, nearestSphereIdx);

				vec3 ambientColor = materials[materialId].ambient;
				vec3 diffuseColor;

				if (materials[materialId].texWidth == 0)
					diffuseColor = materials[materialId].diffuse;
				else
				{
					float u = uv.x * 2048.0f - 0.5f;
					float v = uv.y * 2048.0f - 0.5f;
					int uu = floor(u);
					int vv = floor(v);
					float uRatio = u - uu;
					float vRatio = v - vv;
					float uOpposite = 1 - uRatio;
					float vOpposite = 1 - vRatio;

					int uu0 = uu * 2048;
					int uu1 = glm::clamp(uu + 1, 0, 2047) * 2048;

					int vv0 = 2047 - vv;
					int vv1 = 2047 - glm::clamp(vv + 1, 0, 2047);

					float texR =
						(textures[(uu0 + vv0) * 4 + 0] * uOpposite +
							textures[(uu1 + vv0) * 4 + 0] * uRatio) * vOpposite +
							(textures[(uu0 + vv1) * 4 + 0] * uOpposite +
								textures[(uu1 + vv1) * 4 + 0] * uRatio)*vRatio;

					float texG =
						(textures[(uu0 + vv0) * 4 + 1] * uOpposite +
							textures[(uu1 + vv0) * 4 + 1] * uRatio) * vOpposite +
							(textures[(uu0 + vv1) * 4 + 1] * uOpposite +
								textures[(uu1 + vv1) * 4 + 1] * uRatio)*vRatio;

					float texB =
						(textures[(uu0 + vv0) * 4 + 2] * uOpposite +
							textures[(uu1 + vv0) * 4 + 2] * uRatio) * vOpposite +
							(textures[(uu0 + vv1) * 4 + 2] * uOpposite +
								textures[(uu1 + vv1) * 4 + 2] * uRatio)*vRatio;

					diffuseColor = glm::vec3(texR, texG, texB);
				}

				diffuseColor *= kd / glm::pi<float>();
				diffuseColor = vec3(
					diffuseColor.r * lights[k].color.r,
					diffuseColor.g * lights[k].color.g,
					diffuseColor.b * lights[k].color.b);
				lightedColor += glm::vec4(ambientColor + radiance * NdotL * diffuseColor, 1.0f);
			}

			color += lightedColor * nowRay.decay;
		}
	}

	return color;
}

__global__ void RayTraceD(
	glm::vec4* data,
	const int gridX,
	const int gridY,
	glm::mat4 view,
	OctreeNode* root,
	AABB* boundingboxes, int boxNum,
	Triangle* triangles, int triangleNum,
	Sphere* spheres, int sphereNum,
	Light* lights, int lightNum,
	Material* materials, int matNum,
	float* textures, int texSize)
{
	//unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int x = (blockIdx.x + gridY * RAY_Y_NUM) * WINDOW_HEIGHT + (threadIdx.x + gridX * RAY_X_NUM);

	Ray ray = GenerateCameraRay(blockIdx.x + gridY * RAY_Y_NUM, threadIdx.x + gridX * RAY_X_NUM, view);
	ray.rayType = 0;
	ray.decay = 1.0f;

	Ray rayQueue[QUEUE_SIZE];
	// NOTICE for문을 돌릴 때 iter를 변수로 하니까 검은 화면이 나옴
	// y, x로 들어가고
	// 0, 0 좌표는 좌하단
	data[x] = RayTraceColor(
		ray,
		rayQueue,
		boundingboxes,
		boxNum,
		triangles,
		triangleNum,
		spheres,
		sphereNum,
		lights,
		lightNum,
		materials,
		matNum, 
		textures,
		texSize,
		1);
}

void RayTrace(
	glm::vec4* data,
	const int gridX,
	const int gridY,
	glm::mat4 view,
	OctreeNode* root,
	const vector<AABB>& boundingboxes,
	const vector<Triangle>& triangles,
	const vector<Sphere>& spheres,
	const vector<Light>& lights,
	const vector<Material>& materials,
	const vector<float>& textures)
{
	thrust::device_vector<AABB> b = boundingboxes;
	thrust::device_vector<Triangle> t = triangles;
	thrust::device_vector<Sphere> s = spheres;
	thrust::device_vector<Light> l = lights;
	thrust::device_vector<Material> m = materials;
	thrust::device_vector<float> tex = textures;

	hipDeviceSetLimit(hipLimitMallocHeapSize, 5000000000 * sizeof(float));

	vector<Triangle> tss;
	OctreeNode* d_root = BuildOctree(tss);

	RayTraceD << <RAY_Y_NUM, RAY_X_NUM >> > (
		data,
		gridX,
		gridY,
		view,
		d_root,
		b.data().get(),
		b.size(),
		t.data().get(),
		t.size(),
		s.data().get(),
		s.size(),
		l.data().get(),
		l.size(),
		m.data().get(),
		m.size(),
		tex.data().get(),
		tex.size()
	);
}