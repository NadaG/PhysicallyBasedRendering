#include "hip/hip_runtime.h"
#include "RayTracer.cuh"
#include "Octree.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm\glm.hpp>
#include <stdio.h>
#include <glm\gtc\matrix_transform.hpp>
#include <hip/hip_math_constants.h>
#include <math.h>
#include <thrust\device_vector.h>
#include <queue>

// TODO LIST
// 1. per line draw to eliminate kernel time out problem
// 2. octree acceleration
// 3. mtl file load
// 4. texture mapping with interpolation
// 5. monte calro path tracer

struct Ray
{
	// Ray�� ����
	vec3 origin;
	// Ray�� ����
	vec3 dir;
};

struct Sphere
{
	glm::vec3 origin;
	float radius;
};

const int WINDOW_HEIGHT = 1024;
const int WINDOW_WIDTH = 1024;

const int QUEUE_SIZE = 3;

using std::cout;
using std::endl;

__device__ bool RaySphereIntersect(Ray ray, Sphere sphere, float& dist)
{
	glm::vec3 s = ray.origin - sphere.origin;

	float a = dot(ray.dir, ray.dir);
	float bPrime = dot(s, ray.dir);
	float c = dot(s, s) - sphere.radius * sphere.radius;

	float D = bPrime * bPrime - a * c;
	if (D >= 0 && bPrime <= 0)
	{
		float t1 = (-bPrime + sqrt(D)) / a;
		float t2 = (-bPrime - sqrt(D)) / a;
		dist = t1 > t2 ? t2 : t1;
		return true;
	}
	else
		return false;
}

// back face culling�� ����Ǿ� ����
__device__ bool RayTriangleIntersect(Ray ray, Triangle triangle, float& dist)
{
	glm::vec3 v0v1 = triangle.v1 - triangle.v0;
	glm::vec3 v0v2 = triangle.v2 - triangle.v0;
	glm::vec3 pvec = glm::cross(ray.dir, v0v2);

	float det = dot(v0v1, pvec);

	float epsilon = 0.0001f;

	if (det < epsilon)
		return false;

	if (fabs(det) < epsilon)
		return false;

	float invDet = 1 / det;

	glm::vec3 tvec = ray.origin - triangle.v0;
	float u = glm::dot(tvec, pvec) * invDet;
	if (u < 0 || u > 1)
		return false;

	glm::vec3 qvec = cross(tvec, v0v1);
	float v = dot(ray.dir, qvec) * invDet;
	if (v < 0 || u + v > 1)
		return false;

	dist = dot(v0v2, qvec) * invDet;

	return true;
}

__device__ bool RayAABBIntersect(Ray ray, AABB box)
{
	float tmin, tmax, tymin, tymax, tzmin, tzmax;

	glm::vec3 invdir = 1.0f / ray.dir;
	int sign[3];
	sign[0] = invdir.x < 0;
	sign[1] = invdir.y < 0;
	sign[2] = invdir.z < 0;

	tmin = (box.bounds[sign[0]].x - ray.origin.x) * invdir.x;
	tmax = (box.bounds[1 - sign[0]].x - ray.origin.x) * invdir.x;
	tymin = (box.bounds[sign[1]].y - ray.origin.y) * invdir.y;
	tymax = (box.bounds[1 - sign[1]].y - ray.origin.y) * invdir.y;

	if ((tmin > tymax) || (tymin > tmax))
		return false;

	if (tymin > tmin)
		tmin = tymin;
	if (tymax < tmax)
		tmax = tymax;

	tzmin = (box.bounds[sign[2]].z - ray.origin.z) * invdir.z;
	tzmax = (box.bounds[1 - sign[2]].z - ray.origin.z) * invdir.z;

	if ((tmin > tzmax) || (tzmin > tmax))
		return false;

	return true;
}

__device__ bool RayAABBsIntersect(Ray ray, AABB* boxes, int boxNum)
{
	bool isIntersect = false;

	for (int it = 0; it < boxNum; it++)
	{
		if (RayAABBIntersect(ray, boxes[it]))
			isIntersect = true;
	}
	return isIntersect;
}

// ���� ����� triangle�� id�� ��ȯ�ϰ� �ش� �������� dist�� �����´�
__device__ int FindNearestTriangleIdx(Ray ray, Triangle* triangles, int triangleNum, float& dist)
{
	const float rayThreshold = 0.001f;
	float minDist = 9999999.0f;
	int minIdx = -1;
	float tmpDist;

	// �״�� dist�� �����ͼ� ����ϴϱ� �̻�����
	for (int i = 0; i < triangleNum; ++i)
	{
		// intersect �� ���
		if (RayTriangleIntersect(ray, triangles[i], tmpDist))
		{
			// �� ã�� ���, �ٽ� ã�� �ʱ�
			if (tmpDist > rayThreshold && tmpDist < minDist)
			{
				minDist = tmpDist;
				minIdx = i;
			}
		}
	}

	dist = minDist;
	return minIdx;
}

__device__ Ray GenerateCameraRay(int y, int x, glm::mat4 view)
{
	Ray ray;

	// 0~1
	// world ��ǥ�� ray�� ��, ������ �� window�ϼ��� ������ ���� ray�� ��
	// ���� NDC ��ǥ�� ��ȯ��
	float NDCy = (y + 0.5f) / WINDOW_HEIGHT;
	float NDCx = (x + 0.5f) / WINDOW_WIDTH;

	float aspectRatio = WINDOW_WIDTH / WINDOW_HEIGHT;

	float fov = 45.0f;

	// NDC ��ǥ�� -1 ~ 1�� ��ȯ
	// tan(halfRadian)
	// world ��ǥ���� z�� ������ 1�̱� ������ ������ ����
	float xx = (NDCx * 2.0f - 1.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f) * aspectRatio;
	float yy = (NDCy * 2.0f - 1.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f);

	// ray���� world ������ ������

	ray.origin = glm::vec3(-view * glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
	ray.dir = normalize(vec3(view * vec4(glm::vec3(xx, yy, -1.0), 0.0f)));

	return ray;
}

__device__ void Enqueue(Ray* rayQueue, Ray ray, int& rear)
{
	rear = (rear + 1) % QUEUE_SIZE;
	rayQueue[rear] = ray;
}

__device__ void Dequeue(Ray* rayQueue, int& front)
{
	Ray ray = rayQueue[front];
	front = (front + 1) % QUEUE_SIZE;
}

__device__ Ray GetQueueFront(Ray* rayQueue, const int front)
{
	return rayQueue[(front + 1) % QUEUE_SIZE];
}

__device__ bool IsQueueFull(const int front, const int rear)
{
	return front == (rear + 1) % QUEUE_SIZE;
}

__device__ bool IsQueueEmpty(const int front, const int rear)
{
	return front == rear;
}

// hit point, selected light, all triangles, nearestTriangleIdx
__device__ vec3 RayCastColor(
	vec3 V,
	vec3 hitPoint, 
	Light light,
	Triangle* triangles, 
	const int triangleNum, 
	Material* materials,
	const int materialNum,
	const int nearestTriangleIdx)
{
	vec3 color = glm::vec3(0.0f, 0.0f, 0.0f);

	Ray shadowRay;
	shadowRay.origin = hitPoint;
	shadowRay.dir = normalize(light.pos - hitPoint);

	bool isLighted = true;
	float tmp;
	for (int k = 0; k < triangleNum; k++)
	{
		if (nearestTriangleIdx != k)
			if (RayTriangleIntersect(shadowRay, triangles[k], tmp))
				// ������ dir�� ��
				if (tmp > 0.0001f)
					isLighted = false;
	}

	if (isLighted)
	{
		Triangle nearestTriangle = triangles[nearestTriangleIdx];

		glm::vec3 N = nearestTriangle.normal;
		glm::vec3 L = glm::normalize(light.pos - hitPoint);

		glm::vec3 matAmbient = materials[nearestTriangle.materialId].ambient;
		glm::vec3 matDiffuse = materials[nearestTriangle.materialId].diffuse;
		glm::vec3 matSpecular = materials[nearestTriangle.materialId].specular;

		glm::vec3 ambient = glm::vec3(
			matAmbient.r * light.color.r,
			matAmbient.g * light.color.g,
			matAmbient.b * light.color.b);

		glm::vec3 diffuse = glm::vec3(
			matDiffuse.r * light.color.r,
			matDiffuse.g * light.color.g,
			matDiffuse.b * light.color.b) *
			glm::clamp(dot(N, L), 0.0f, 1.0f);

		glm::vec3 specular = glm::vec3(
			matSpecular.r * light.color.r,
			matSpecular.g * light.color.g,
			matSpecular.b * light.color.b) *
			glm::max(0.0f, pow(glm::max(dot(normalize(reflect(-L, N)), V), 0.0f), 16));

		color = glm::vec4(glm::vec3(ambient + diffuse + specular), 1.0f);
	}

	return color;
}

__device__ vec4 RayTraceColor(
	Ray ray,
	Ray* rayQueue,
	AABB* objects,
	int objNum,
	Triangle* triangles,
	int triangleNum,
	Light* lights,
	int lightNum,
	Material* materials,
	int matNum,
	int depth)
{
	vec4 color = vec4(0.25f);
	int front = 0, rear = 0;

	// ù ��° ray�� node�� �ϴ� queue ����
	Enqueue(rayQueue, ray, rear);

	int nowDepth = 1;

	// �� 7 (1 + 2 + 4)���� ray�� ����
	for (int i = 1; i < depth; ++i)
	{
		int target = rear;

		while (!IsQueueEmpty(target, front))
		{
			Ray nowRay;
			nowRay = GetQueueFront(rayQueue, front);
			Dequeue(rayQueue, front);

			if (!RayAABBsIntersect(nowRay, objects, objNum))
				continue;

			float distToTriangle;
			int nearestTriangleIdx = FindNearestTriangleIdx(nowRay, triangles, triangleNum, distToTriangle);
			
			// �� ã�Ұų� ���ʿ� �ִٸ�
			if (nearestTriangleIdx == -1 || distToTriangle < 0.0f)
				continue;

			Triangle nearestTriangle = triangles[nearestTriangleIdx];
			glm::vec3 N = glm::normalize(nearestTriangle.normal);

			glm::vec4 lightedColor = glm::vec4(0.0f);
			glm::vec3 hitPoint = nowRay.origin + nowRay.dir * distToTriangle;

			for (int k = 0; k < lightNum; k++)
			{
				lightedColor += glm::vec4(
					RayCastColor(-nowRay.dir, hitPoint, lights[k], 
						triangles, triangleNum, materials, matNum, nearestTriangleIdx)
					, 1.0f);

				lightedColor += glm::vec4(0.01f, 0.0f, 0.0f, 0.0f);
			}

			Ray reflectRay;
			reflectRay.origin = hitPoint;
			reflectRay.dir = normalize(reflect(nowRay.dir, N));

			Ray refractRay;
			refractRay.origin = hitPoint;
			refractRay.dir = normalize(refract(nowRay.dir, N, 1.2f));

			Enqueue(rayQueue, reflectRay, rear);
			Enqueue(rayQueue, refractRay, rear);

			float s = (float)glm::floor(glm::log((float)nowDepth) / glm::log(2.0f));
			color += lightedColor * pow(0.2f, s);
		}

		nowDepth++;
	}

	// ������ ���� queue�� ������ �ϱ�
	while (!IsQueueEmpty(front, rear))
	{
		Ray nowRay;
		nowRay = GetQueueFront(rayQueue, front);
		Dequeue(rayQueue, front);

		if (!RayAABBsIntersect(nowRay, objects, objNum))
			continue;

		float distToTriangle;
		int nearestTriangleIdx = FindNearestTriangleIdx(nowRay, triangles, triangleNum, distToTriangle);
		
		// �� ã�Ұų� ���ʿ� �ִٸ�
		if (nearestTriangleIdx == -1 || distToTriangle < 0.0f)
			continue;

		Triangle nearestTriangle = triangles[nearestTriangleIdx];
		glm::vec3 N = normalize(nearestTriangle.normal);

		glm::vec4 lightedColor = glm::vec4(0.0f);
		glm::vec3 hitPoint = nowRay.origin + nowRay.dir * distToTriangle;

		for (int k = 0; k < lightNum; k++)
		{
			lightedColor += glm::vec4(
				RayCastColor(-nowRay.dir, hitPoint, lights[k], 
					triangles, triangleNum, materials, matNum, nearestTriangleIdx)
				, 1.0f);
		}

		float s = (float)glm::floor(glm::log((float)nowDepth) / glm::log(2.0f));
		color += lightedColor * pow(0.2f, s);
	}

	return color;
}

__global__ void RayTraceD(
	glm::vec4* data,
	glm::mat4 view,
	OctreeNode* root,
	AABB* boundingboxes, int boxNum,
	Triangle* triangles, int triangleNum,
	Light* lights, int lightNum,
	Material* materials, int matNum)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	Ray ray = GenerateCameraRay(blockIdx.x, threadIdx.x, view);

	Ray rayQueue[QUEUE_SIZE];
	// NOTICE for���� ���� �� iter�� ������ �ϴϱ� ���� ȭ���� ����
	// y, x�� ����
	// 0, 0 ��ǥ�� ���ϴ�

	/*if (ray.dir.x < root->bndMin.x)
	{
		data[x] = glm::vec4(1.0f, 0.0f, 0.0f, 1.0f);
		return;
	}*/

	data[x] = RayTraceColor(
		ray,
		rayQueue,
		boundingboxes,
		boxNum,
		triangles,
		triangleNum,
		lights,
		lightNum,
		materials,
		matNum, 
		2);
}

void RayTrace(
	glm::vec4* data,
	glm::mat4 view,
	OctreeNode* root,
	const vector<AABB>& boundingboxes,
	const vector<Triangle>& triangles, 
	const vector<Light>& lights,
	const vector<Material>& materials)
{
	thrust::device_vector<AABB> b = boundingboxes;
	thrust::device_vector<Triangle> t = triangles;
	thrust::device_vector<Light> l = lights;
	thrust::device_vector<Material> m = materials;

	hipDeviceSetLimit(hipLimitMallocHeapSize, 500000000 * sizeof(float));

	vector<Triangle> tss;
	OctreeNode* d_root = BuildOctree(tss);

	RayTraceD << <WINDOW_HEIGHT, WINDOW_WIDTH >> > (
		data,
		view,
		d_root,
		b.data().get(),
		b.size(),
		t.data().get(),
		t.size(),
		l.data().get(),
		l.size(),
		m.data().get(),
		m.size()
	);
}