#include "hip/hip_runtime.h"
#include "RayTracer.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm\glm.hpp>
#include <stdio.h>
#include <glm\gtc\matrix_transform.hpp>
#include <hip/hip_math_constants.h>
#include <math.h>
#include <thrust\device_vector.h>
#include <queue>

struct Ray
{
	glm::vec3 origin;
	glm::vec3 dir;
};

struct RayNode
{
	Ray ray;
	int depth = 1;
};

struct Sphere
{
	glm::vec3 origin;
	float radius;
};

const int WINDOW_HEIGHT = 1024;
const int WINDOW_WIDTH = 1024;

using std::queue;

__device__ bool RaySphereIntersect(Ray ray, Sphere sphere, float& dist)
{
	glm::vec3 s = ray.origin - sphere.origin;

	float a = dot(ray.dir, ray.dir);
	float bPrime = dot(s, ray.dir);
	float c = dot(s, s) - sphere.radius * sphere.radius;

	float D = bPrime * bPrime - a * c;
	if (D >= 0 && bPrime <= 0)
	{
		float t1 = (-bPrime + sqrt(D)) / a;
		float t2 = (-bPrime - sqrt(D)) / a;
		dist = t1 > t2 ? t2 : t1;
		return true;
	}
	else
		return false;
}

// back face culling�� ����Ǿ� ����
__device__ bool RayTriangleIntersect(Ray ray, Triangle triangle, float& dist)
{
	glm::vec3 v0v1 = triangle.v1 - triangle.v0;
	glm::vec3 v0v2 = triangle.v2 - triangle.v0;
	glm::vec3 pvec = glm::cross(ray.dir, v0v2);

	float det = dot(v0v1, pvec);

	float epsilon = 0.0001f;

	if (det < epsilon)
		return false;

	if (fabs(det) < epsilon)
		return false;

	float invDet = 1 / det;

	glm::vec3 tvec = ray.origin - triangle.v0;
	float u = glm::dot(tvec, pvec) * invDet;
	if (u < 0 || u > 1)
		return false;

	glm::vec3 qvec = cross(tvec, v0v1);
	float v = dot(ray.dir, qvec) * invDet;
	if (v < 0 || u + v > 1)
		return false;

	dist = dot(v0v2, qvec) * invDet;

	return true;
}

__device__ Ray GenerateCameraRay(int y, int x, glm::mat4 view)
{
	Ray ray;

	float NDCy = (y + 0.5f) / WINDOW_HEIGHT;
	float NDCx = (x + 0.5f) / WINDOW_WIDTH;

	float aspectRatio = WINDOW_WIDTH / WINDOW_HEIGHT;

	float fov = 45.0f;

	float xx = ((NDCx) * 2.0f - 1.0f) * tan(fov *0.5f * 3.141592653f / 180.0f) * aspectRatio;
	float yy = (1.0f - NDCy * 2.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f);

	// -1 ~ 1
	ray.origin = glm::vec3(-view * glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
	ray.dir = normalize(vec3(view * vec4(glm::vec3(xx, yy, -1.0), 0.0f)));

	return ray;
}

__device__ vec3 GenerateRayQueue(
	Ray ray, 
	RayNode* rayQueue, 
	Triangle* triangles, 
	int triangleNum, 
	Light* lights, 
	int lightNum, 
	int depth)
{
	vec3 color = vec3(0.0f);
	int front = 0, rear = 0;

	// ù ��° ray�� node�� �ϴ� queue ����
	RayNode rayNode;
	rayNode.ray = ray;
	rayNode.depth = 1;

	rayQueue[front] = rayNode;
	front++;

	// �� 7 (1 + 2 + 4)���� ray�� ����
	for (int d = 0; d < depth; d++)
	{
		int target = front;

		while (rear < target)
		{
			Ray nowRay = rayQueue[rear].ray;
			const int nowDepth = rayQueue[rear].depth;
			rear++;

			float distToTriangle;
			float minDistToTriangle = 99999999.0f;
			glm::vec3 minColor = glm::vec3(0, 0, 0);

			for (int i = 0; i < triangleNum; i++)
			{
				// intersect �� ���
				// reflect ray�� refract ray ����
				if (RayTriangleIntersect(nowRay, triangles[i], distToTriangle))
				{
					if (distToTriangle < 1.0f) {
						continue;
					}

					// ���� �տ� �ִ� �ȼ��� �׸���
					if (distToTriangle < minDistToTriangle)
					{
						minDistToTriangle = distToTriangle;

						for (int j = 0; j < lightNum; j++)
						{
							glm::vec3 hitPoint = nowRay.origin + nowRay.dir * distToTriangle;

							Ray shadowRay;
							shadowRay.origin = hitPoint;
							shadowRay.dir = normalize(lights[j].pos - hitPoint);

							bool isLighted = true;
							float tmp;
							for (int k = 0; k < triangleNum; k++)
							{
								if (i != k)
									if (RayTriangleIntersect(shadowRay, triangles[k], tmp))
										// ������ dir�� ��
										if (tmp > 0.0f)
											isLighted = false;
							}

							if (!isLighted)
							{
								//color = glm::vec3(0.1f, 0.1f, 0.1f);
								continue;
							}

							glm::vec3 L = glm::normalize(lights[j].pos - hitPoint);
							glm::vec3 N = normalize(triangles[i].normal);
							glm::vec3 V = -nowRay.dir;

							glm::vec3 ambient = glm::vec3(0.2, 0.2, 0.2) * lights[j].color;
							glm::vec3 diffuse = glm::vec3(0.3, 0.3, 0.3) * lights[j].color * glm::max(0.0f, dot(N, L));
							glm::vec3 specular = glm::vec3(0.1, 0.8, 0.2) * lights[j].color * glm::max(0.0f, pow(glm::max(dot(normalize(reflect(-L, N)), V), 0.0f), 16));

							glm::vec3 col = ambient + diffuse + specular;

							//float s = (float)glm::floor(glm::log((float)nowDepth) / glm::log(2.0f));
							//color += col * pow(0.2f, s);
							minColor = col;

							Ray reflectRay;
							reflectRay.origin = hitPoint;
							reflectRay.dir = normalize(reflect(-L, N));

							rayQueue[front].depth = nowDepth + 1;
							rayQueue[front].ray = reflectRay;
							front++;

							Ray refractRay;
							refractRay.origin = hitPoint;
							refractRay.dir = normalize(refract(-L, N, 2.0f));

							rayQueue[front].depth = nowDepth + 1;
							rayQueue[front].ray = refractRay;
							front++;

							//data = i*scalingfactor*color;
						}
					}
				}
			}
			
			float s = (float)glm::floor(glm::log((float)nowDepth) / glm::log(2.0f));
			color += minColor * pow(0.2f, s);
		}
	}

	// ������ ���� queue�� ������ �ϱ�
	while (rear < front)
	{
		rear++;
		// color ���
	}

	return color;
}

//__device__ vec3 RayTraceColor(Ray ray, Triangle* triangles, int triangleNum, Light* lights, int lightNum, int depth)
//{
//	if (depth <= 0)
//		return vec3(0.0f);
//
//	vec3 color = vec3(0.0f);
//
//	float distToTriangle;
//	float minDistToTriangle = 99999999.0f;
//
//	for (int i = 0; i < triangleNum; i++)
//	{
//		if (RayTriangleIntersect(ray, triangles[i], distToTriangle))
//		{
//			// ���� �տ� �ִ� �ȼ��� �׸���
//			if (distToTriangle < minDistToTriangle)
//			{
//				minDistToTriangle = distToTriangle;
//
//				for (int j = 0; j < lightNum; j++)
//				{
//					glm::vec3 hitPoint = ray.origin + ray.dir * distToTriangle;
//
//					Ray shadowRay;
//					shadowRay.origin = hitPoint;
//					shadowRay.dir = normalize(lights[j].pos - hitPoint);
//
//					bool isLighted = true;
//					float tmp;
//					for (int k = 0; k < triangleNum; k++)
//					{
//						if (i != k)
//							if (RayTriangleIntersect(shadowRay, triangles[k], tmp))
//								// ������ dir�� ��
//								if (tmp > 0.0f)
//									isLighted = false;
//					}
//
//					if (!isLighted)
//					{
//						color = glm::vec3(0.1f, 0.1f, 0.1f);
//						continue;
//					}
//
//					glm::vec3 L = glm::normalize(lights[j].pos - hitPoint);
//					glm::vec3 N = normalize(triangles[i].normal);
//					glm::vec3 V = -ray.dir;
//
//					glm::vec3 ambient = glm::vec3(0.2, 0.2, 0.2) * lights[j].color;
//					glm::vec3 diffuse = glm::vec3(0.3, 0.3, 0.3) * lights[j].color * glm::max(0.0f, dot(N, L));
//					glm::vec3 specular = glm::vec3(0.1, 0.8, 0.2) * lights[j].color * glm::max(0.0f, pow(glm::max(dot(normalize(reflect(-L, N)), V), 0.0f), 16));
//
//					glm::vec3 col = ambient + diffuse + specular;
//
//					color = col;
//
//					if (depth > 1)
//					{
//						Ray reflectRay;
//						reflectRay.origin = hitPoint;
//						reflectRay.dir = normalize(reflect(-L, N));
//
//						//glm::vec3 C1 = RayTraceColor(reflectRay, triangles, triangleNum, lights, lightNum, depth - 1) * 0.0002f;
//						////////////////////////////////////////////////////////////////////////////////////
//
//						// �������� ���ٴ� ���� �� �־����� ���ٴ� ��, (normal �������� �־���)
//						Ray refractRay;
//						refractRay.origin = hitPoint;
//						refractRay.dir = normalize(refract(-L, N, 2.0f));
//
//						///////////////// Naive Algorithm without Recursion ////////////////////////////////
//						//
//
//						//color += RayTraceColor(refractRay, triangles, triangleNum, lights, lightNum, depth - 1) * 0.0002f;
//						//glm::vec3 C2 = RayTraceColor(refractRay, triangles, triangleNum, lights, lightNum, depth - 1) * 0.0002f;
//
//						////////////////////////////////////////////////////////////////////////////////////
//					}
//				}
//			}
//		}
//	}
//
//	return color;
//}

// TODO view matrix�� ������ ��
__global__ void RayTraceD(
	glm::vec4* data,
	glm::mat4 view,
	Triangle* triangles, int triangleNum,
	Light* lights, int lightNum,
	RayNode* rayQueue, int queueSize)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	Ray ray = GenerateCameraRay(blockIdx.x, threadIdx.x, view);

	rayQueue = new RayNode[1+2+4];

	// ray���� �����ϰ� queue�� �ִ� ����
	vec3 color = GenerateRayQueue(ray, rayQueue, triangles, triangleNum, lights, lightNum, 2);

	delete[] rayQueue;

	// ray���� queue���� �����鼭 color�� ���ϴ� ����

	//vec3 color = RayTraceColor(ray, triangles, triangleNum, lights, lightNum, 1);

	data[x] = glm::vec4(color, 1.0f);
}

//void RayTrace(glm::vec4* data, glm::mat4 view, Triangle* triangles, int triangleNum)
void RayTrace(glm::vec4* data, glm::mat4 view, const vector<Triangle> &triangles, const vector<Light>& lights)
{
	thrust::device_vector<Triangle> t = triangles;
	thrust::device_vector<Light> l = lights;

	std::vector<RayNode> hQueue;
	hQueue.resize(7);

	thrust::device_vector<RayNode> dQueue = hQueue;

	size_t size;
	hipDeviceSetLimit(hipLimitMallocHeapSize, 100000000 * sizeof(float));
	hipDeviceGetLimit(&size, hipLimitMallocHeapSize);


	RayTraceD << <WINDOW_HEIGHT, WINDOW_WIDTH >> > (
		data,
		view,
		t.data().get(),
		t.size(),
		l.data().get(),
		l.size(),
		dQueue.data().get(),
		dQueue.size());
}