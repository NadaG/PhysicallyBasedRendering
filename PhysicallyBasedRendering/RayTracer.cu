#include "hip/hip_runtime.h"
#include "RayTracer.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm\glm.hpp>
#include <stdio.h>
#include <glm\gtc\matrix_transform.hpp>
#include <hip/hip_math_constants.h>
#include <math.h>

struct Ray
{
	glm::vec3 origin;
	glm::vec3 dir;
};

struct Sphere
{
	glm::vec3 origin;
	float radius;
};

const int WINDOW_HEIGHT = 1024;
const int WINDOW_WIDTH = 1024;

__device__ bool RaySphereIntersect(Ray ray, Sphere sphere, float& dist)
{
	glm::vec3 s = ray.origin - sphere.origin;

	float a = dot(ray.dir, ray.dir);
	float bPrime = dot(s, ray.dir);
	float c = dot(s, s) - sphere.radius * sphere.radius;

	float D = bPrime * bPrime - a * c;
	if (D >= 0 && bPrime <= 0)
	{
		float t1 = (-bPrime + sqrt(D)) / a;
		float t2 = (-bPrime - sqrt(D)) / a;
		dist = t1 > t2 ? t2 : t1;
		return true;
	}
	else
		return false;
}

__device__ bool RayTriangleIntersect(Ray ray, Triangle triangle, float& dist)
{
	glm::vec3 v0v1 = triangle.v1 - triangle.v0;
	glm::vec3 v0v2 = triangle.v2 - triangle.v0;
	glm::vec3 pvec = glm::cross(ray.dir, v0v2);

	float det = dot(v0v1, pvec);

	float epsilon = 0.0001f;

	if (det < epsilon) 
		return false;
	
	if (fabs(det) < epsilon)
		return false;

	float invDet = 1 / det;

	glm::vec3 tvec = ray.origin - triangle.v0;
	float u = glm::dot(tvec, pvec) * invDet;
	if (u < 0 || u > 1)
		return false;

	glm::vec3 qvec = cross(tvec, v0v1);
	float v = dot(ray.dir, qvec) * invDet;
	if (v < 0 || u + v > 1)
		return false;

	float t = dot(v0v2, qvec) * invDet;

	return true;
}

// Camera functions
///////////////////
__device__ Ray GenerateCameraRay(int y, int x, glm::mat4 view)
{
	Ray ray;

	float NDCy = (y + 0.5f) / WINDOW_HEIGHT;
	float NDCx = (x + 0.5f) / WINDOW_WIDTH;

	float aspectRatio = WINDOW_WIDTH / WINDOW_HEIGHT;

	float fov = 45.0f;

	float xx = (((float)(x + 0.5f) / (float)WINDOW_WIDTH) * 2.0f - 1.0f) * tan(fov *0.5f * 3.141592653f / 180.0f) * aspectRatio;
	float yy = (1.0f - ((float)(y + 0.5f) / (float)WINDOW_HEIGHT) * 2.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f);

	//// -1 ~ 1
	ray.origin = glm::vec3(-view * glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
	ray.dir = normalize(vec3(view * vec4(glm::vec3(xx, yy, -1.0), 0.0f)));

	//// view matrix�� translate ���и� ������
	//ray.origin = (-view*vec4(vec3(0.0), 1)).xyz;
	//// view matrix�� rotate ������ ������
	//ray.dir = normalize((view*vec4(ray.dir, 0)).xyz);
	//// view matrix�� camera ������ x, y, z���� column���� �α� ������(normalize��) scale ������ �ǹ̰� ���� 

	return ray;
}

__global__ void RayTraceD(glm::vec4* data, glm::mat4 view, Triangle* triangles)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	Ray ray = GenerateCameraRay(blockIdx.x, threadIdx.x, view);

	Sphere sphere;
	sphere.origin = glm::vec3(0.0f, 0.0f, -20.0f);
	sphere.radius = 2.0f;

	Sphere sphere2;
	sphere2.origin = glm::vec3(3.0f, 0.0f, -20.0f);
	sphere2.radius = 2.0f;

	Triangle triangle;
	triangle.v0 = glm::vec3(0.0f, 0.0f, -5.0f);
	triangle.v1 = glm::vec3(1.0f, 0.0f, -5.0f);
	triangle.v2 = glm::vec3(1.0f, 1.0f, -5.0f);

	float distToSphere, distToTriangle;

	glm::vec3 lightPos = glm::vec3(10.0f, 0.0f, 0.0f);

	if (RaySphereIntersect(ray, sphere, distToSphere))
	{
		glm::vec3 hitPoint = ray.origin + ray.dir * distToSphere;
		glm::vec3 L = glm::normalize(lightPos - hitPoint);
		glm::vec3 N = normalize(hitPoint - sphere.origin);

		glm::vec3 ambient = glm::vec3(0.2, 0.2, 0.2);

		glm::vec3 diffuse = glm::vec3(0.1, 0.4, 0.2) * glm::max(0.0f, dot(N, L));

		glm::vec3 V = -ray.dir;

		glm::vec3 specular = glm::vec3(0.1, 0.4, 0.2) * glm::max(0.0f, pow(glm::max(dot(normalize(reflect(-L, N)), V), 0.0f), 16));

		glm::vec3 col = ambient + diffuse + specular;

		data[x] = glm::vec4(col.x, col.y, col.z, 1.0f);
	}
	else if (RaySphereIntersect(ray, sphere2, distToSphere))
	{
		glm::vec3 hitPoint = ray.origin + ray.dir * distToSphere;
		glm::vec3 L = normalize(lightPos - hitPoint);
		glm::vec3 N = normalize(hitPoint - sphere2.origin);

		glm::vec3 ambient = glm::vec3(0.2, 0.2, 0.2);

		glm::vec3 diffuse = glm::vec3(0.1, 0.4, 0.2) * glm::max(0.0f, dot(N, L));

		glm::vec3 V = -ray.dir;

		glm::vec3 specular = glm::vec3(0.1, 0.4, 0.2) * glm::max(0.0f, pow(glm::max(dot(normalize(reflect(-L, N)), V), 0.0f), 16));

		glm::vec3 col = ambient + diffuse + specular;

		data[x] = glm::vec4(col.x, col.y, col.z, 1.0f);
	}
	else if (RayTriangleIntersect(ray, triangle, distToTriangle))
	{
		data[x] = glm::vec4(1.0f, 0.0f, 0.0f, 1.0f);
	}
	else
	{
		data[x] = glm::vec4(0.0f, 0.0f, 0.0f, 1.0f);
	}
}

void RayTrace(glm::vec4* data, glm::mat4 view, Triangle* triangles)
{
	RayTraceD << <WINDOW_HEIGHT, WINDOW_WIDTH >> > (data, view, triangles);
}