#include "hip/hip_runtime.h"
﻿#include "KDTree.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm\glm.hpp>
#include <glm\gtx\component_wise.hpp>
#include <stdio.h>
#include <glm\gtc\matrix_transform.hpp>
#include <hip/hip_math_constants.h>
#include <math.h>
#include <thrust\device_vector.h>
#include <queue>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <ctime>
#include <stdio.h>

//#pragma region old version
//__global__ void ComputeAABB(Triangle* T, int triangleNum)
//{
//	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
//	if (idx >= triangleNum)
//		return;
//
//	T[idx].tbb.bounds[0].x = glm::min(T[idx].v2.x, glm::min(T[idx].v0.x, T[idx].v1.x));
//	T[idx].tbb.bounds[1].x = glm::max(T[idx].v2.x, glm::max(T[idx].v0.x, T[idx].v1.x));
//
//	T[idx].tbb.bounds[0].y = glm::min(T[idx].v2.y, glm::min(T[idx].v0.y, T[idx].v1.y));
//	T[idx].tbb.bounds[1].y = glm::max(T[idx].v2.y, glm::max(T[idx].v0.y, T[idx].v1.y));
//
//	T[idx].tbb.bounds[0].z = glm::min(T[idx].v2.z, glm::min(T[idx].v0.z, T[idx].v1.z));
//	T[idx].tbb.bounds[1].z = glm::max(T[idx].v2.z, glm::max(T[idx].v0.z, T[idx].v1.z));
//}
//
//KDTreeNode* BuildKDTree(const vector<Triangle>& T)
//{
//	thrust::device_vector<Triangle> t = T;
//	int block = t.size() / 1024 + 1;
//
//	ComputeAABB << < block, 1024 >> > (t.data().get(), t.size());
//	cout << "Compute Tri AABB" << endl;
//
//	//// node list
//	//Nvector* nodeList = new Nvector();
//	//hipMalloc((void**)&(nodeList->nodeTriangleList), sizeof(int)*t.size());
//
//	//Nvector* deviceNodeList;
//	//hipMalloc((void**)&deviceNodeList, sizeof(Nvector));
//	//hipMemcpy(deviceNodeList, nodeList, sizeof(Nvector), hipMemcpyHostToDevice);
//
//	//// active list
//	//Nvector* activeList = new Nvector();
//	//hipMalloc((void**)&(activeList->nodeTriangleList), sizeof(int)*t.size());
//
//	//Nvector* deviceActiveList;
//	//hipMalloc((void**)&deviceActiveList, sizeof(Nvector));
//	//hipMemcpy(deviceActiveList, activeList, sizeof(Nvector), hipMemcpyHostToDevice);
//
//	//// small list
//	//Nvector* smallList = new Nvector();
//	//hipMalloc((void**)&(smallList->nodeTriangleList), sizeof(int)*t.size());
//
//	//Nvector* deviceSmallList;
//	//hipMalloc((void**)&deviceSmallList, sizeof(Nvector));
//	//hipMemcpy(deviceSmallList, smallList, sizeof(Nvector), hipMemcpyHostToDevice);
//
//	//// next list
//	//Nvector* nextList = new Nvector();
//	//hipMalloc((void**)&(nextList->nodeTriangleList), sizeof(int)*t.size());
//
//	//Nvector* deviceNextList;
//	//hipMalloc((void**)&deviceNextList, sizeof(Nvector));
//	//hipMemcpy(deviceNextList, nextList, sizeof(Nvector), hipMemcpyHostToDevice);
//
//
//	//KDTreeNode* root;
//	//hipMalloc((void**)&root, sizeof(KDTreeNode));
//	
//	Nvector* nodeList = new Nvector();
//	nodeList->nodeTriangleList = new int[t.size()];
//
//	Nvector* activeList = new Nvector();
//	activeList->nodeTriangleList = new int[t.size()];
//
//	Nvector* smallList = new Nvector();
//	smallList->nodeTriangleList = new int[t.size()];
//
//	Nvector* nextList = new Nvector();
//	nextList->nodeTriangleList = new int[t.size()];
//
//	KDTreeNode* root = new KDTreeNode();
//	root->firstTriangle = 0;
//	root->triangleNum = t.size();
//	root->bnd.bounds[0] = vec3(-60, -60, -60);
//	root->bnd.bounds[1] = vec3(60, 60, 60);
//	root->chunkSize = t.size() / CHUNKSIZE + 1;
//
//	activeList->push_back(*root);
//	for (int i = 0; i < t.size(); i++)
//	{
//		activeList->nodeTriangleList[i] = i;
//	}
//
//	// Large node stage
//	int aa = 1;
//	while (!activeList->empty() && aa == 1)
//	{
//		nodeList->append(activeList);
//		delete nextList;
//		nextList = new Nvector();
//
//		ProcessLargeNodes(activeList, smallList, nextList, t.data().get(), t.size());
//
//		/*Nvector* tmp = activeList;
//		activeList = nextList;
//		nextList = tmp;*/
//
//		aa = 0;
//	}
//	
//
//	cout << "KDend\n" << endl;
//	
//	delete nodeList;
//	delete activeList;
//	delete smallList;
//	delete nextList;
//	delete root;
//
//
//	return nullptr;
//}
//
//__global__ void ChunkingTriangle(Nvector* activeList, ChunkNode* chunkList)
//{
//	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
//
//	if (idx < activeList->size())
//	{
//		int size = activeList->operator[](idx).chunkSize;
//
//		for (int i = 0; i < size; i++)
//		{
//			ChunkNode newChunk;
//			newChunk.node = &activeList->operator[](idx);
//			newChunk.firstTriangle = activeList->operator[](idx).firstTriangle + i*CHUNKSIZE;
//			//newChunk.firstTriangle = 100;
//
//			if (i == size - 1)
//				newChunk.triangleNum = activeList->operator[](idx).triangleNum - CHUNKSIZE*i;
//			else
//				newChunk.triangleNum = CHUNKSIZE;
//
//			// 자신보다 index가 앞인 노드들의 chunk 수를 모두 더해서 chunk list에서 현재 노드의 시작 위치를 알아낸다.
//			int startIdx = 0;
//			for (int j = 0; j < idx; j++)
//			{
//				startIdx += activeList->operator[](j).chunkSize;
//			}
//
//			chunkList[startIdx + i] = newChunk;
//	
//			//chunkList[0].triangleNum = size;
//		}
//	}
//}
//
//
//__global__ void ComputeChunkAABB(ChunkNode* chunkList, int chunkNum, Triangle* T)
//{
//	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
//
//	if (idx < chunkNum)
//	{
//		vec3 max = { -99999,-99999,-99999 };
//		vec3 min = { 99999,99999,99999 };
//
//		for (int i = 0; i < CHUNKSIZE; i++)
//		{
//			chunkList[idx].cbb.bounds[0].x = thrust::min(min.x, T[chunkList[idx].firstTriangle + i].tbb.bounds[0].x);
//			chunkList[idx].cbb.bounds[0].y = thrust::min(min.y, T[chunkList[idx].firstTriangle + i].tbb.bounds[0].y);
//			chunkList[idx].cbb.bounds[0].z = thrust::min(min.z, T[chunkList[idx].firstTriangle + i].tbb.bounds[0].z);
//
//			chunkList[idx].cbb.bounds[1].x = thrust::max(max.x, T[chunkList[idx].firstTriangle + i].tbb.bounds[1].x);
//			chunkList[idx].cbb.bounds[1].y = thrust::max(max.y, T[chunkList[idx].firstTriangle + i].tbb.bounds[1].y);
//			chunkList[idx].cbb.bounds[1].z = thrust::max(max.z, T[chunkList[idx].firstTriangle + i].tbb.bounds[1].z);
//
//			min.x = chunkList[idx].cbb.bounds[0].x;
//			min.y = chunkList[idx].cbb.bounds[0].y;
//			min.z = chunkList[idx].cbb.bounds[0].z;
//
//			max.x = chunkList[idx].cbb.bounds[1].x;
//			max.y = chunkList[idx].cbb.bounds[1].y;
//			max.z = chunkList[idx].cbb.bounds[1].z;
//
//		}	
//	}
//}
//
//
//__global__ void SegmentedReduction(ChunkNode* chunkList, int gap, int cnum)
//{
//	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
//
//	if (idx*gap * 2 > cnum - 1 || idx*gap * 2 + gap > cnum-1)
//		return;
//
//	if (chunkList[idx*gap * 2].node == chunkList[idx*gap * 2 + gap].node)
//	{
//		chunkList[idx*gap * 2].cbb.bounds[0].x = thrust::min(chunkList[idx*gap * 2].cbb.bounds[0].x, chunkList[idx*gap * 2 + gap].cbb.bounds[0].x);
//		chunkList[idx*gap * 2].cbb.bounds[0].y = thrust::min(chunkList[idx*gap * 2].cbb.bounds[0].y, chunkList[idx*gap * 2 + gap].cbb.bounds[0].y);
//		chunkList[idx*gap * 2].cbb.bounds[0].z = thrust::min(chunkList[idx*gap * 2].cbb.bounds[0].z, chunkList[idx*gap * 2 + gap].cbb.bounds[0].z);
//
//		chunkList[idx*gap * 2].cbb.bounds[1].x = thrust::max(chunkList[idx*gap * 2].cbb.bounds[1].x, chunkList[idx*gap * 2 + gap].cbb.bounds[1].x);
//		chunkList[idx*gap * 2].cbb.bounds[1].y = thrust::max(chunkList[idx*gap * 2].cbb.bounds[1].y, chunkList[idx*gap * 2 + gap].cbb.bounds[1].y);
//		chunkList[idx*gap * 2].cbb.bounds[1].z = thrust::max(chunkList[idx*gap * 2].cbb.bounds[1].z, chunkList[idx*gap * 2 + gap].cbb.bounds[1].z);
//	}
//	else
//	{	
//		if (chunkList[idx*gap * 2 + gap].node == chunkList[0].node)
//		{
//			chunkList[idx*gap * 2].node->tbb.bounds[0].x = thrust::min(chunkList[idx*gap * 2].node->tbb.bounds[0].x, chunkList[idx*gap * 2].cbb.bounds[0].x);
//			chunkList[idx*gap * 2].node->tbb.bounds[0].y = thrust::min(chunkList[idx*gap * 2].node->tbb.bounds[0].y, chunkList[idx*gap * 2].cbb.bounds[0].y);
//			chunkList[idx*gap * 2].node->tbb.bounds[0].z = thrust::min(chunkList[idx*gap * 2].node->tbb.bounds[0].z, chunkList[idx*gap * 2].cbb.bounds[0].z);
//
//			chunkList[idx*gap * 2].node->tbb.bounds[1].x = thrust::max(chunkList[idx*gap * 2].node->tbb.bounds[1].x, chunkList[idx*gap * 2].cbb.bounds[1].x);
//			chunkList[idx*gap * 2].node->tbb.bounds[1].y = thrust::max(chunkList[idx*gap * 2].node->tbb.bounds[1].y, chunkList[idx*gap * 2].cbb.bounds[1].y);
//			chunkList[idx*gap * 2].node->tbb.bounds[1].z = thrust::max(chunkList[idx*gap * 2].node->tbb.bounds[1].z, chunkList[idx*gap * 2].cbb.bounds[1].z);
//
//			chunkList[idx*gap * 2] = chunkList[idx*gap * 2 + gap];
//		}
//		else
//		{
//			chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].x = thrust::min(chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].x, chunkList[idx*gap * 2 + gap].cbb.bounds[0].x);
//			chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].y = thrust::min(chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].y, chunkList[idx*gap * 2 + gap].cbb.bounds[0].y);
//			chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].z = thrust::min(chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].z, chunkList[idx*gap * 2 + gap].cbb.bounds[0].z);
//
//			chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].x = thrust::max(chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].x, chunkList[idx*gap * 2 + gap].cbb.bounds[1].x);
//			chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].y = thrust::max(chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].y, chunkList[idx*gap * 2 + gap].cbb.bounds[1].y);
//			chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].z = thrust::max(chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].z, chunkList[idx*gap * 2 + gap].cbb.bounds[1].z);
//		}
//	}
//
//	//	기준이 되는 노드의 정보 수정
//	chunkList[0].node->tbb = chunkList[0].cbb;
//}
//
//
//__global__ void SplitLargeNode(Nvector* activeList, KDTreeNode* devNextData)
//{
//	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
//
//	if (idx >= activeList->size())
//		return;
//
//
//	////////////////////////////////////////////////////
//	//	cut off empty space
//	//	min
//	if (abs(activeList->operator[](idx).bnd.bounds[0].x-activeList->operator[](idx).tbb.bounds[0].x) / 
//		abs(activeList->operator[](idx).bnd.bounds[0].x - activeList->operator[](idx).bnd.bounds[1].x) > CUTOFF)
//	{
//		activeList->operator[](idx).bnd.bounds[0].x = activeList->operator[](idx).tbb.bounds[0].x;
//	}
//
//	if (abs(activeList->operator[](idx).bnd.bounds[0].y - activeList->operator[](idx).tbb.bounds[0].y) / 
//		abs(activeList->operator[](idx).bnd.bounds[0].y- activeList->operator[](idx).bnd.bounds[1].y) > CUTOFF)
//	{
//		activeList->operator[](idx).bnd.bounds[0].y = activeList->operator[](idx).tbb.bounds[0].y;
//	}
//
//	if (abs(activeList->operator[](idx).bnd.bounds[0].z - activeList->operator[](idx).tbb.bounds[0].z) / 
//		abs(activeList->operator[](idx).bnd.bounds[0].z - activeList->operator[](idx).bnd.bounds[1].z) > CUTOFF)
//	{
//		activeList->operator[](idx).bnd.bounds[0].z = activeList->operator[](idx).tbb.bounds[0].z;
//	}
//
//
//	// max
//	if (abs(activeList->operator[](idx).bnd.bounds[1].x - activeList->operator[](idx).tbb.bounds[1].x) / 
//		abs(activeList->operator[](idx).bnd.bounds[0].x - activeList->operator[](idx).bnd.bounds[1].x)> CUTOFF)
//	{
//		activeList->operator[](idx).bnd.bounds[1].x = activeList->operator[](idx).tbb.bounds[1].x;
//	}
//
//	if (abs((*activeList)[idx].bnd.bounds[1].y - (*activeList)[idx].tbb.bounds[1].y) /
//		abs((*activeList)[idx].bnd.bounds[0].y - (*activeList)[idx].bnd.bounds[1].y) > CUTOFF)
//	{
//		(*activeList)[idx].bnd.bounds[1].y = (*activeList)[idx].tbb.bounds[1].y;
//	}
//
//	if (abs(activeList->operator[](idx).bnd.bounds[1].z - activeList->operator[](idx).tbb.bounds[1].z) / 
//		abs(activeList->operator[](idx).bnd.bounds[0].z - activeList->operator[](idx).bnd.bounds[1].z) > CUTOFF)
//	{
//		activeList->operator[](idx).bnd.bounds[1].z = activeList->operator[](idx).tbb.bounds[1].z;
//	}
//	////////////////////////////////////////////////////
//
//
//	////////////////////////////////////////////////////
//	//	split node at spatial median of the longest axis
//	float xAxis = abs(activeList->operator[](idx).tbb.bounds[0].x - activeList->operator[](idx).tbb.bounds[1].x);
//	float yAxis = abs(activeList->operator[](idx).tbb.bounds[0].y - activeList->operator[](idx).tbb.bounds[1].y);
//	float zAxis = abs(activeList->operator[](idx).tbb.bounds[0].z - activeList->operator[](idx).tbb.bounds[1].z);
//
//	float maxAxis = thrust::max(zAxis, thrust::max(xAxis, yAxis));
//	
//
//	if (xAxis == maxAxis)
//	{
//		float median = (activeList->operator[](idx).tbb.bounds[0].x + activeList->operator[](idx).tbb.bounds[1].x) / 2;
//
//		KDTreeNode* leftChild = &devNextData[idx * 2];
//		KDTreeNode* rightChild = &devNextData[idx * 2 + 1];
//
//		leftChild->bnd = activeList->operator[](idx).tbb;
//		leftChild->bnd.bounds[1].x = median;
//
//		rightChild->bnd = activeList->operator[](idx).tbb;
//		rightChild->bnd.bounds[0].x = median;
//
//		activeList->operator[](idx).leftChild = leftChild;
//		activeList->operator[](idx).rightChild = rightChild;
//
//	}
//	else if (yAxis == maxAxis)
//	{
//		float median = (activeList->operator[](idx).bnd.bounds[0].y + activeList->operator[](idx).bnd.bounds[1].y) / 2;
//
//		KDTreeNode* leftChild = &devNextData[idx * 2];
//		KDTreeNode* rightChild = &devNextData[idx * 2 + 1];
//
//		leftChild->bnd = activeList->operator[](idx).tbb;
//		leftChild->bnd.bounds[1].y = median;
//
//		rightChild->bnd = activeList->operator[](idx).tbb;
//		rightChild->bnd.bounds[0].y = median;
//
//		activeList->operator[](idx).leftChild = leftChild;
//		activeList->operator[](idx).rightChild = rightChild;
//	}
//	else
//	{
//		float median = (activeList->operator[](idx).bnd.bounds[0].z + activeList->operator[](idx).bnd.bounds[1].z) / 2;
//
//		KDTreeNode* leftChild = &devNextData[idx * 2];
//		KDTreeNode* rightChild = &devNextData[idx * 2 + 1];
//
//		leftChild->bnd = activeList->operator[](idx).tbb;
//		leftChild->bnd.bounds[1].z = median;
//
//		rightChild->bnd = activeList->operator[](idx).tbb;
//		rightChild->bnd.bounds[0].z = median;
//
//		activeList->operator[](idx).leftChild = leftChild;
//		activeList->operator[](idx).rightChild = rightChild;
//	}
//	////////////////////////////////////////////////////
//
//	//if (activeList->operator[](0).leftChild->triangleNum == 100)
//	//	activeList->operator[](idx).bnd.bounds[0].x = 11111.0f;
//}
//
//
//__global__ void SortAndClip(ChunkNode* chunkList)
//{
//
//}
//
//
//__global__ void Add(int* a)
//{
//	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
//
//	a++;
//}
//
//void ProcessLargeNodes(Nvector* activeList, Nvector* smallList, Nvector* nextList, Triangle* T, int triangleNum)
//{
//	//	copy active list to GPU memory
//	int* devNTL;
//	hipMalloc((void**)&devNTL, sizeof(int)*triangleNum);
//	hipMemcpy(devNTL, activeList->nodeTriangleList, sizeof(int)*triangleNum, hipMemcpyHostToDevice);
//
//	KDTreeNode* devData;
//	hipMalloc((void**)&devData, sizeof(KDTreeNode)*activeList->size());
//	hipMemcpy(devData, activeList->data, sizeof(KDTreeNode)*activeList->size(), hipMemcpyHostToDevice);
//
//	Nvector* tmp = new Nvector();
//	tmp->nodeTriangleList = devNTL;
//	tmp->data = devData;
//	tmp->capacity = activeList->capacity;
//	tmp->sz = activeList->sz;
//
//	Nvector* devActiveList;
//	hipMalloc((void**)&devActiveList, sizeof(Nvector));
//	hipMemcpy(devActiveList, tmp, sizeof(Nvector), hipMemcpyHostToDevice);
//
//	
//
//	///////////////////////////////////////////////
//	//	1st step, group triangles into chunks
//	ChunkNode* chunkList;
//
//	//	active list에 존재하는 모든 chunk의 개수를 구한다.
//	int cnum = 0;
//	for (int i = 0; i < activeList->size(); i++)
//	{
//		cnum += activeList->operator[](i).chunkSize;
//	}
//	hipMalloc((void**)&chunkList, sizeof(ChunkNode)*cnum);
//	
//	int block = activeList->size();
//
//	ChunkingTriangle << < block, 1 >> > (devActiveList, chunkList);	
//	///////////////////////////////////////////////
//
//
//	///////////////////////////////////////////////
//	//	2nd step, compute per-node bounding box
//	ComputeChunkAABB << < cnum, 1 >> > (chunkList, cnum, T);
//	
//	int a = 0;
//	if (cnum % 2 == 0)
//		a = cnum / 2;
//	else
//		a = cnum / 2 + 1;
//
//	for (int gap = 0; gap < a; gap++)
//	{
//		SegmentedReduction << < 1, cnum >> > (chunkList, pow(2, gap), cnum);
//	}
//	///////////////////////////////////////////////
//
//
//	///////////////////////////////////////////////
//	//	3rd step, split large node
//	block = activeList->size();
//
//	KDTreeNode* devNextData;
//	hipMalloc((void**)&devNextData, 2 * sizeof(KDTreeNode) * block);
//
//	block = activeList->size();
//	SplitLargeNode << < block, 1 >> > (devActiveList, devNextData);
//
//	KDTreeNode* nextData = new KDTreeNode();
//	hipMemcpy(nextData, devNextData, 2 * sizeof(KDTreeNode) * block, hipMemcpyDeviceToHost);
//
//	for (int i = 0; i < 2 * sizeof(KDTreeNode) * block; i++)
//	{
//		nextList->push_back(nextData[i]);	
//	}
//	///////////////////////////////////////////////
//
//
//	///////////////////////////////////////////////
//	//	4th step, sort and clip triangles to child nodes
//
//	//SortAndClip << < block, thread >> > ();
//
//	///////////////////////////////////////////////
//
//
//	ChunkNode* hostList = new ChunkNode[cnum];
//	hipMemcpy(hostList, chunkList, sizeof(ChunkNode)*cnum, hipMemcpyDeviceToHost);
//
//	cout << "idx " <<hostList[33].triangleNum << endl;
//
//	for (int i = 0; i < 34; i++)
//	{
//		//cout << "AABB " << hostList[i].cbb.bounds[1].y << endl;
//	}
//
//	
//	Nvector *aa = new Nvector();
//	hipMemcpy(aa, devActiveList, sizeof(Nvector), hipMemcpyDeviceToHost);
//
//	//hipMemcpy(hostList, chunkList, sizeof(ChunkNode)*cnum, hipMemcpyDeviceToHost);
//	KDTreeNode* node11 = new KDTreeNode();
//	hipMemcpy(node11, &aa->operator[](0), sizeof(KDTreeNode), hipMemcpyDeviceToHost);
//
//	cout << "=============node tri=============" << endl;
//	cout << "AABB " << node11->tbb.bounds[0].x << endl;
//	cout << "AABB " << node11->tbb.bounds[0].y << endl;
//	cout << "AABB " << node11->tbb.bounds[0].z << endl;
//
//	cout << "AABB " << node11->tbb.bounds[1].x << endl;
//	cout << "AABB " << node11->tbb.bounds[1].y << endl;
//	cout << "AABB " << node11->tbb.bounds[1].z << endl;
//
//	cout << "=============node =============" << endl;
//	cout << "AABB " << node11->bnd.bounds[0].x << endl;
//	cout << "AABB " << node11->bnd.bounds[0].y << endl;
//	cout << "AABB " << node11->bnd.bounds[0].z << endl;
//
//	cout << "AABB " << node11->bnd.bounds[1].x << endl;
//	cout << "AABB " << node11->bnd.bounds[1].y << endl;
//	cout << "AABB " << node11->bnd.bounds[1].z << endl;
//
//
//	KDTreeNode* leftnode = new KDTreeNode();
//	hipMemcpy(leftnode, node11->leftChild, sizeof(KDTreeNode), hipMemcpyDeviceToHost);
//
//	cout << "============left==============" << endl;
//	cout << "AABB " << leftnode->bnd.bounds[0].x << endl;
//	cout << "AABB " << leftnode->bnd.bounds[0].y << endl;
//	cout << "AABB " << leftnode->bnd.bounds[0].z << endl;
//
//	cout << "AABB " << leftnode->bnd.bounds[1].x << endl;
//	cout << "AABB " << leftnode->bnd.bounds[1].y << endl;
//	cout << "AABB " << leftnode->bnd.bounds[1].z << endl;
//
//	KDTreeNode* rightnode = new KDTreeNode();
//	hipMemcpy(rightnode, node11->rightChild, sizeof(KDTreeNode), hipMemcpyDeviceToHost);
//
//	cout << "============right==============" << endl;
//	cout << "AABB " << rightnode->bnd.bounds[0].x << endl;
//	cout << "AABB " << rightnode->bnd.bounds[0].y << endl;
//	cout << "AABB " << rightnode->bnd.bounds[0].z << endl;
//
//	cout << "AABB " << rightnode->bnd.bounds[1].x << endl;
//	cout << "AABB " << rightnode->bnd.bounds[1].y << endl;
//	cout << "AABB " << rightnode->bnd.bounds[1].z << endl;
//
//
//	hipFree(chunkList);
//	delete hostList;
//	hipFree(devNTL);
//	hipFree(devData);
//	hipFree(devActiveList);
//
//	tmp->nodeTriangleList = nullptr;
//	tmp->data = nullptr;
//	delete tmp;
//} 
//#pragma endregion


__global__ void ComputeAABB(Triangle* T, int triangleNum, AABB* tbbs)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= triangleNum)
		return;

	tbbs[idx].bounds[0].x = glm::min(T[idx].v2.x, glm::min(T[idx].v0.x, T[idx].v1.x));
	tbbs[idx].bounds[1].x = glm::max(T[idx].v2.x, glm::max(T[idx].v0.x, T[idx].v1.x));

	tbbs[idx].bounds[0].y = glm::min(T[idx].v2.y, glm::min(T[idx].v0.y, T[idx].v1.y));
	tbbs[idx].bounds[1].y = glm::max(T[idx].v2.y, glm::max(T[idx].v0.y, T[idx].v1.y));

	tbbs[idx].bounds[0].z = glm::min(T[idx].v2.z, glm::min(T[idx].v0.z, T[idx].v1.z));
	tbbs[idx].bounds[1].z = glm::max(T[idx].v2.z, glm::max(T[idx].v0.z, T[idx].v1.z));
}

KDTreeNode* BuildKDTree(const vector<Triangle>& T)
{
	thrust::device_vector<Triangle> t = T;
	int block = t.size() / 1024 + 1;

	AABB* tbbs;
	hipMalloc((void**)&tbbs, sizeof(AABB) * t.size());

	ComputeAABB << < block, 1024 >> > (t.data().get(), t.size(), tbbs);
	cout << "Compute Tri AABB" << endl;
	
	Nvector* nodeList = new Nvector();

	Nvector* activeList = new Nvector();

	Nvector* smallList = new Nvector();

	Nvector* nextList = new Nvector();

	KDTreeNode* root = new KDTreeNode();

	//////////////////////////////////////////////
	//	initialize root node

	root->bnd.bounds[0] = vec3(-60, -60, -60);
	root->bnd.bounds[1] = vec3(60, 60, 60);
	root->triangleNum = t.size();
	root->chunkSize = t.size() / CHUNKSIZE + 1;

	int* triIdx = new int[t.size()];
	for (int i = 0; i < t.size(); i++)
	{
		triIdx[i] = i;
	}
	int* triIdxDev;
	hipMalloc((void**)&triIdxDev, sizeof(int) * t.size());
	hipMemcpy(triIdxDev, triIdx, sizeof(int) * t.size(), hipMemcpyHostToDevice);
	root->triIdx = triIdxDev;
	
	root->triAABB = tbbs;

	int* tag;
	hipMalloc((void**)&tag, sizeof(int) * t.size());
	hipMemcpy(tag, triIdx, sizeof(int) * t.size(), hipMemcpyHostToDevice);
	root->tag = tag;

	activeList->push_back(*root);

	//////////////////////////////////////////////



	//////////////////////////////////////////////
	// Large node stage
	int aa = 1;
	while (!activeList->empty() && aa == 1)
	{
		nodeList->append(activeList);
		delete nextList;
		nextList = new Nvector();

		ProcessLargeNodes(activeList, smallList, nextList, t.data().get(), t.size());

		/*Nvector* tmp = activeList;
		activeList = nextList;
		nextList = tmp;*/

		aa = 0;
	}
	

	cout << "KDend\n" << endl;
	
	delete nodeList;
	delete activeList;
	delete smallList;
	delete nextList;
	delete root;


	return nullptr;
}

__global__ void ChunkingTriangle(Nvector* activeList, ChunkNode* chunkList)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < activeList->size())
	{
		int size = activeList->operator[](idx).chunkSize;

		for (int i = 0; i < size; i++)
		{
			ChunkNode newChunk;
			newChunk.node = &activeList->operator[](idx);
			newChunk.firstTriangle = i*CHUNKSIZE;
			//newChunk.firstTriangle = 100;

			if (i == size - 1)
				newChunk.triangleNum = activeList->operator[](idx).triangleNum - CHUNKSIZE*i;
			else
				newChunk.triangleNum = CHUNKSIZE;

			// 자신보다 index가 앞인 노드들의 chunk 수를 모두 더해서 chunk list에서 현재 노드의 시작 위치를 알아낸다.
			int startIdx = 0;
			for (int j = 0; j < idx; j++)
			{
				startIdx += activeList->operator[](j).chunkSize;
			}

			chunkList[startIdx + i] = newChunk;
	
			//chunkList[0].triangleNum = size;
		}
	}
}


__global__ void ComputeChunkAABB(ChunkNode* chunkList, int chunkNum, Triangle* T)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < chunkNum)
	{
		vec3 max = { -99999,-99999,-99999 };
		vec3 min = { 99999,99999,99999 };

		for (int i = 0; i < CHUNKSIZE; i++)
		{
			chunkList[idx].cbb.bounds[0].x = thrust::min(min.x, chunkList[idx].node->triAABB[chunkList[idx].firstTriangle + i].bounds[0].x);
			chunkList[idx].cbb.bounds[0].y = thrust::min(min.y, chunkList[idx].node->triAABB[chunkList[idx].firstTriangle + i].bounds[0].y);
			chunkList[idx].cbb.bounds[0].z = thrust::min(min.z, chunkList[idx].node->triAABB[chunkList[idx].firstTriangle + i].bounds[0].z);

			chunkList[idx].cbb.bounds[1].x = thrust::max(max.x, chunkList[idx].node->triAABB[chunkList[idx].firstTriangle + i].bounds[1].x);
			chunkList[idx].cbb.bounds[1].y = thrust::max(max.y, chunkList[idx].node->triAABB[chunkList[idx].firstTriangle + i].bounds[1].y);
			chunkList[idx].cbb.bounds[1].z = thrust::max(max.z, chunkList[idx].node->triAABB[chunkList[idx].firstTriangle + i].bounds[1].z);
			//	???chunkList[idx].cbb.bounds[1].z = thrust::max(max.z, chunkList[idx].node->triAABB[chunkList[idx].node->triIdx[chunkList[idx].firstTriangle + i]].bounds[1].z);

			min.x = chunkList[idx].cbb.bounds[0].x;
			min.y = chunkList[idx].cbb.bounds[0].y;
			min.z = chunkList[idx].cbb.bounds[0].z;

			max.x = chunkList[idx].cbb.bounds[1].x;
			max.y = chunkList[idx].cbb.bounds[1].y;
			max.z = chunkList[idx].cbb.bounds[1].z;

		}	
	}
}


__global__ void SegmentedReduction(ChunkNode* chunkList, int gap, int cnum)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx*gap * 2 > cnum - 1 || idx*gap * 2 + gap > cnum-1)
		return;

	if (chunkList[idx*gap * 2].node == chunkList[idx*gap * 2 + gap].node)
	{
		chunkList[idx*gap * 2].cbb.bounds[0].x = thrust::min(chunkList[idx*gap * 2].cbb.bounds[0].x, chunkList[idx*gap * 2 + gap].cbb.bounds[0].x);
		chunkList[idx*gap * 2].cbb.bounds[0].y = thrust::min(chunkList[idx*gap * 2].cbb.bounds[0].y, chunkList[idx*gap * 2 + gap].cbb.bounds[0].y);
		chunkList[idx*gap * 2].cbb.bounds[0].z = thrust::min(chunkList[idx*gap * 2].cbb.bounds[0].z, chunkList[idx*gap * 2 + gap].cbb.bounds[0].z);

		chunkList[idx*gap * 2].cbb.bounds[1].x = thrust::max(chunkList[idx*gap * 2].cbb.bounds[1].x, chunkList[idx*gap * 2 + gap].cbb.bounds[1].x);
		chunkList[idx*gap * 2].cbb.bounds[1].y = thrust::max(chunkList[idx*gap * 2].cbb.bounds[1].y, chunkList[idx*gap * 2 + gap].cbb.bounds[1].y);
		chunkList[idx*gap * 2].cbb.bounds[1].z = thrust::max(chunkList[idx*gap * 2].cbb.bounds[1].z, chunkList[idx*gap * 2 + gap].cbb.bounds[1].z);
	}
	else
	{	
		if (chunkList[idx*gap * 2 + gap].node == chunkList[0].node)
		{
			chunkList[idx*gap * 2].node->tbb.bounds[0].x = thrust::min(chunkList[idx*gap * 2].node->tbb.bounds[0].x, chunkList[idx*gap * 2].cbb.bounds[0].x);
			chunkList[idx*gap * 2].node->tbb.bounds[0].y = thrust::min(chunkList[idx*gap * 2].node->tbb.bounds[0].y, chunkList[idx*gap * 2].cbb.bounds[0].y);
			chunkList[idx*gap * 2].node->tbb.bounds[0].z = thrust::min(chunkList[idx*gap * 2].node->tbb.bounds[0].z, chunkList[idx*gap * 2].cbb.bounds[0].z);

			chunkList[idx*gap * 2].node->tbb.bounds[1].x = thrust::max(chunkList[idx*gap * 2].node->tbb.bounds[1].x, chunkList[idx*gap * 2].cbb.bounds[1].x);
			chunkList[idx*gap * 2].node->tbb.bounds[1].y = thrust::max(chunkList[idx*gap * 2].node->tbb.bounds[1].y, chunkList[idx*gap * 2].cbb.bounds[1].y);
			chunkList[idx*gap * 2].node->tbb.bounds[1].z = thrust::max(chunkList[idx*gap * 2].node->tbb.bounds[1].z, chunkList[idx*gap * 2].cbb.bounds[1].z);

			chunkList[idx*gap * 2] = chunkList[idx*gap * 2 + gap];
		}
		else
		{
			chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].x = thrust::min(chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].x, chunkList[idx*gap * 2 + gap].cbb.bounds[0].x);
			chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].y = thrust::min(chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].y, chunkList[idx*gap * 2 + gap].cbb.bounds[0].y);
			chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].z = thrust::min(chunkList[idx*gap * 2 + gap].node->tbb.bounds[0].z, chunkList[idx*gap * 2 + gap].cbb.bounds[0].z);

			chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].x = thrust::max(chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].x, chunkList[idx*gap * 2 + gap].cbb.bounds[1].x);
			chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].y = thrust::max(chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].y, chunkList[idx*gap * 2 + gap].cbb.bounds[1].y);
			chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].z = thrust::max(chunkList[idx*gap * 2 + gap].node->tbb.bounds[1].z, chunkList[idx*gap * 2 + gap].cbb.bounds[1].z);
		}
	}

	//	기준이 되는 노드의 정보 수정
	chunkList[0].node->tbb = chunkList[0].cbb;
}


__global__ void SplitLargeNode(Nvector* activeList, KDTreeNode* devNextData)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx >= activeList->size())
		return;


	////////////////////////////////////////////////////
	//	cut off empty space
	//	min
	if (abs(activeList->operator[](idx).bnd.bounds[0].x-activeList->operator[](idx).tbb.bounds[0].x) / 
		abs(activeList->operator[](idx).bnd.bounds[0].x - activeList->operator[](idx).bnd.bounds[1].x) > CUTOFF)
	{
		activeList->operator[](idx).bnd.bounds[0].x = activeList->operator[](idx).tbb.bounds[0].x;
	}

	if (abs(activeList->operator[](idx).bnd.bounds[0].y - activeList->operator[](idx).tbb.bounds[0].y) / 
		abs(activeList->operator[](idx).bnd.bounds[0].y- activeList->operator[](idx).bnd.bounds[1].y) > CUTOFF)
	{
		activeList->operator[](idx).bnd.bounds[0].y = activeList->operator[](idx).tbb.bounds[0].y;
	}

	if (abs(activeList->operator[](idx).bnd.bounds[0].z - activeList->operator[](idx).tbb.bounds[0].z) / 
		abs(activeList->operator[](idx).bnd.bounds[0].z - activeList->operator[](idx).bnd.bounds[1].z) > CUTOFF)
	{
		activeList->operator[](idx).bnd.bounds[0].z = activeList->operator[](idx).tbb.bounds[0].z;
	}


	// max
	if (abs(activeList->operator[](idx).bnd.bounds[1].x - activeList->operator[](idx).tbb.bounds[1].x) / 
		abs(activeList->operator[](idx).bnd.bounds[0].x - activeList->operator[](idx).bnd.bounds[1].x)> CUTOFF)
	{
		activeList->operator[](idx).bnd.bounds[1].x = activeList->operator[](idx).tbb.bounds[1].x;
	}

	if (abs((*activeList)[idx].bnd.bounds[1].y - (*activeList)[idx].tbb.bounds[1].y) /
		abs((*activeList)[idx].bnd.bounds[0].y - (*activeList)[idx].bnd.bounds[1].y) > CUTOFF)
	{
		(*activeList)[idx].bnd.bounds[1].y = (*activeList)[idx].tbb.bounds[1].y;
	}

	if (abs(activeList->operator[](idx).bnd.bounds[1].z - activeList->operator[](idx).tbb.bounds[1].z) / 
		abs(activeList->operator[](idx).bnd.bounds[0].z - activeList->operator[](idx).bnd.bounds[1].z) > CUTOFF)
	{
		activeList->operator[](idx).bnd.bounds[1].z = activeList->operator[](idx).tbb.bounds[1].z;
	}
	////////////////////////////////////////////////////


	////////////////////////////////////////////////////
	//	split node at spatial median of the longest axis
	float xAxis = abs(activeList->operator[](idx).tbb.bounds[0].x - activeList->operator[](idx).tbb.bounds[1].x);
	float yAxis = abs(activeList->operator[](idx).tbb.bounds[0].y - activeList->operator[](idx).tbb.bounds[1].y);
	float zAxis = abs(activeList->operator[](idx).tbb.bounds[0].z - activeList->operator[](idx).tbb.bounds[1].z);

	float maxAxis = thrust::max(zAxis, thrust::max(xAxis, yAxis));
	

	if (xAxis == maxAxis)
	{
		float median = (activeList->operator[](idx).tbb.bounds[0].x + activeList->operator[](idx).tbb.bounds[1].x) / 2;

		KDTreeNode* leftChild = &devNextData[idx * 2];
		KDTreeNode* rightChild = &devNextData[idx * 2 + 1];

		leftChild->bnd = activeList->operator[](idx).tbb;
		leftChild->bnd.bounds[1].x = median;

		rightChild->bnd = activeList->operator[](idx).tbb;
		rightChild->bnd.bounds[0].x = median;

		activeList->operator[](idx).leftChild = leftChild;
		activeList->operator[](idx).rightChild = rightChild;

	}
	else if (yAxis == maxAxis)
	{
		float median = (activeList->operator[](idx).bnd.bounds[0].y + activeList->operator[](idx).bnd.bounds[1].y) / 2;

		KDTreeNode* leftChild = &devNextData[idx * 2];
		KDTreeNode* rightChild = &devNextData[idx * 2 + 1];

		leftChild->bnd = activeList->operator[](idx).tbb;
		leftChild->bnd.bounds[1].y = median;

		rightChild->bnd = activeList->operator[](idx).tbb;
		rightChild->bnd.bounds[0].y = median;

		activeList->operator[](idx).leftChild = leftChild;
		activeList->operator[](idx).rightChild = rightChild;
	}
	else
	{
		float median = (activeList->operator[](idx).bnd.bounds[0].z + activeList->operator[](idx).bnd.bounds[1].z) / 2;

		KDTreeNode* leftChild = &devNextData[idx * 2];
		KDTreeNode* rightChild = &devNextData[idx * 2 + 1];

		leftChild->bnd = activeList->operator[](idx).tbb;
		leftChild->bnd.bounds[1].z = median;

		rightChild->bnd = activeList->operator[](idx).tbb;
		rightChild->bnd.bounds[0].z = median;

		activeList->operator[](idx).leftChild = leftChild;
		activeList->operator[](idx).rightChild = rightChild;
	}
	////////////////////////////////////////////////////

	//if (activeList->operator[](0).leftChild->triangleNum == 100)
	//	activeList->operator[](idx).bnd.bounds[0].x = 11111.0f;
}


__global__ void SortAndClip(ChunkNode* chunkList, Triangle* T, int& cnum)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	unsigned int chunkIdx = blockIdx.x;
	unsigned int triIdx = threadIdx.x;


	if (triIdx >= chunkList[chunkIdx].triangleNum || chunkIdx >= cnum)
		return;

	if (chunkList[chunkIdx].node->triAABB[chunkList[chunkIdx].node->triIdx[chunkList[chunkIdx].firstTriangle + triIdx]].bounds[0].x == 123445)
	{
		//T[idx].tag = 1;
	}
	
}



void ProcessLargeNodes(Nvector* activeList, Nvector* smallList, Nvector* nextList, Triangle* T, int triangleNum)
{
	//	copy active list to GPU memory

	KDTreeNode* devData;
	hipMalloc((void**)&devData, sizeof(KDTreeNode)*activeList->size());
	hipMemcpy(devData, activeList->data, sizeof(KDTreeNode)*activeList->size(), hipMemcpyHostToDevice);

	Nvector* tmp = new Nvector();
	tmp->data = devData;
	tmp->capacity = activeList->capacity;
	tmp->sz = activeList->sz;

	Nvector* devActiveList;
	hipMalloc((void**)&devActiveList, sizeof(Nvector));
	hipMemcpy(devActiveList, tmp, sizeof(Nvector), hipMemcpyHostToDevice);

	

	///////////////////////////////////////////////
	//	1st step, group triangles into chunks
	ChunkNode* chunkList;

	//	active list에 존재하는 모든 chunk의 개수를 구한다.
	int cnum = 0;
	for (int i = 0; i < activeList->size(); i++)
	{
		cnum += activeList->operator[](i).chunkSize;
	}
	hipMalloc((void**)&chunkList, sizeof(ChunkNode)*cnum);
	
	int block = activeList->size();

	ChunkingTriangle << < block, 1 >> > (devActiveList, chunkList);	
	///////////////////////////////////////////////


	///////////////////////////////////////////////
	//	2nd step, compute per-node bounding box
	ComputeChunkAABB << < cnum, 1 >> > (chunkList, cnum, T);
	
	int a = 0;
	if (cnum % 2 == 0)
		a = cnum / 2;
	else
		a = cnum / 2 + 1;

	for (int gap = 0; gap < a; gap++)
	{
		SegmentedReduction << < 1, cnum >> > (chunkList, pow(2, gap), cnum);
	}
	///////////////////////////////////////////////


	///////////////////////////////////////////////
	//	3rd step, split large node
	block = activeList->size();

	KDTreeNode* devNextData;
	hipMalloc((void**)&devNextData, 2 * sizeof(KDTreeNode) * block);

	block = activeList->size();
	SplitLargeNode << < block, 1 >> > (devActiveList, devNextData);

	KDTreeNode* nextData = new KDTreeNode();
	hipMemcpy(nextData, devNextData, 2 * sizeof(KDTreeNode) * block, hipMemcpyDeviceToHost);

	for (int i = 0; i < 2 * sizeof(KDTreeNode) * block; i++)
	{
		nextList->push_back(nextData[i]);	
	}
	///////////////////////////////////////////////


	///////////////////////////////////////////////
	//	4th step, sort and clip triangles to child nodes

	block = cnum;
	SortAndClip << < block, CHUNKSIZE >> > (chunkList, T, cnum);

	///////////////////////////////////////////////


	ChunkNode* hostList = new ChunkNode[cnum];
	hipMemcpy(hostList, chunkList, sizeof(ChunkNode)*cnum, hipMemcpyDeviceToHost);

	cout << "idx " <<hostList[33].triangleNum << endl;

	for (int i = 0; i < 34; i++)
	{
		//cout << "AABB " << hostList[i].cbb.bounds[1].y << endl;
	}

	
	Nvector *aa = new Nvector();
	hipMemcpy(aa, devActiveList, sizeof(Nvector), hipMemcpyDeviceToHost);

	//hipMemcpy(hostList, chunkList, sizeof(ChunkNode)*cnum, hipMemcpyDeviceToHost);
	KDTreeNode* node11 = new KDTreeNode();
	hipMemcpy(node11, &aa->operator[](0), sizeof(KDTreeNode), hipMemcpyDeviceToHost);

	cout << "=============node tri=============" << endl;
	cout << "AABB " << node11->tbb.bounds[0].x << endl;
	cout << "AABB " << node11->tbb.bounds[0].y << endl;
	cout << "AABB " << node11->tbb.bounds[0].z << endl;

	cout << "AABB " << node11->tbb.bounds[1].x << endl;
	cout << "AABB " << node11->tbb.bounds[1].y << endl;
	cout << "AABB " << node11->tbb.bounds[1].z << endl;

	cout << "=============node =============" << endl;
	cout << "AABB " << node11->bnd.bounds[0].x << endl;
	cout << "AABB " << node11->bnd.bounds[0].y << endl;
	cout << "AABB " << node11->bnd.bounds[0].z << endl;

	cout << "AABB " << node11->bnd.bounds[1].x << endl;
	cout << "AABB " << node11->bnd.bounds[1].y << endl;
	cout << "AABB " << node11->bnd.bounds[1].z << endl;


	KDTreeNode* leftnode = new KDTreeNode();
	hipMemcpy(leftnode, node11->leftChild, sizeof(KDTreeNode), hipMemcpyDeviceToHost);

	cout << "============left==============" << endl;
	cout << "AABB " << leftnode->bnd.bounds[0].x << endl;
	cout << "AABB " << leftnode->bnd.bounds[0].y << endl;
	cout << "AABB " << leftnode->bnd.bounds[0].z << endl;

	cout << "AABB " << leftnode->bnd.bounds[1].x << endl;
	cout << "AABB " << leftnode->bnd.bounds[1].y << endl;
	cout << "AABB " << leftnode->bnd.bounds[1].z << endl;

	KDTreeNode* rightnode = new KDTreeNode();
	hipMemcpy(rightnode, node11->rightChild, sizeof(KDTreeNode), hipMemcpyDeviceToHost);

	cout << "============right==============" << endl;
	cout << "AABB " << rightnode->bnd.bounds[0].x << endl;
	cout << "AABB " << rightnode->bnd.bounds[0].y << endl;
	cout << "AABB " << rightnode->bnd.bounds[0].z << endl;

	cout << "AABB " << rightnode->bnd.bounds[1].x << endl;
	cout << "AABB " << rightnode->bnd.bounds[1].y << endl;
	cout << "AABB " << rightnode->bnd.bounds[1].z << endl;


	hipFree(chunkList);
	delete hostList;

	hipFree(devData);
	hipFree(devActiveList);

	tmp->data = nullptr;
	delete tmp;

} 
