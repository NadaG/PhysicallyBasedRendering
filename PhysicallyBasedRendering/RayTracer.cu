#include "hip/hip_runtime.h"
#include "RayTracer.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm\glm.hpp>
#include <stdio.h>
#include <glm\gtc\matrix_transform.hpp>
#include <hip/hip_math_constants.h>
#include <math.h>
#include <thrust\device_vector.h>
#include <queue>

// TODO LIST
// 1. per line draw to eliminate kernel time out problem
// 2. octree acceleration
// 3. mtl file load
// 4. texture mapping with interpolation
// 5. monte calro path tracer

struct Ray
{
	glm::vec3 origin;
	glm::vec3 dir;
};

struct Sphere
{
	glm::vec3 origin;
	float radius;
};

const int WINDOW_HEIGHT = 1024;
const int WINDOW_WIDTH = 1024;

const int QUEUE_SIZE = 3;

__device__ bool RaySphereIntersect(Ray ray, Sphere sphere, float& dist)
{
	glm::vec3 s = ray.origin - sphere.origin;

	float a = dot(ray.dir, ray.dir);
	float bPrime = dot(s, ray.dir);
	float c = dot(s, s) - sphere.radius * sphere.radius;

	float D = bPrime * bPrime - a * c;
	if (D >= 0 && bPrime <= 0)
	{
		float t1 = (-bPrime + sqrt(D)) / a;
		float t2 = (-bPrime - sqrt(D)) / a;
		dist = t1 > t2 ? t2 : t1;
		return true;
	}
	else
		return false;
}

// back face culling�� ����Ǿ� ����
__device__ bool RayTriangleIntersect(Ray ray, Triangle triangle, float& dist)
{
	glm::vec3 v0v1 = triangle.v1 - triangle.v0;
	glm::vec3 v0v2 = triangle.v2 - triangle.v0;
	glm::vec3 pvec = glm::cross(ray.dir, v0v2);

	float det = dot(v0v1, pvec);

	float epsilon = 0.0001f;

	if (det < epsilon)
		return false;

	if (fabs(det) < epsilon)
		return false;

	float invDet = 1 / det;

	glm::vec3 tvec = ray.origin - triangle.v0;
	float u = glm::dot(tvec, pvec) * invDet;
	if (u < 0 || u > 1)
		return false;

	glm::vec3 qvec = cross(tvec, v0v1);
	float v = dot(ray.dir, qvec) * invDet;
	if (v < 0 || u + v > 1)
		return false;

	dist = dot(v0v2, qvec) * invDet;

	return true;
}

// ���� ����� triangle�� id�� ��ȯ�ϰ� �ش� �������� dist�� �����´�
__device__ int FindNearestTriangleIdx(Ray ray, Triangle* triangles, int triangleNum, float& dist)
{
	const float rayThreshold = 0.001f;
	float minDist = 9999999.0f;
	int minIdx = -1;
	float tmpDist;

	// �״�� dist�� �����ͼ� ����ϴϱ� �̻�����
	for (int i = 0; i < triangleNum; ++i)
	{
		// intersect �� ���
		if (RayTriangleIntersect(ray, triangles[i], tmpDist))
		{
			// �� ã�� ���, �ٽ� ã�� �ʱ�
			if (tmpDist > rayThreshold && tmpDist < minDist)
			{
				minDist = tmpDist;
				minIdx = i;
			}
		}
	}

	dist = minDist;
	return minIdx;
}

__device__ Ray GenerateCameraRay(int y, int x, glm::mat4 view)
{
	Ray ray;

	// 0~1
	float NDCy = (y + 0.5f) / WINDOW_HEIGHT;
	float NDCx = (x + 0.5f) / WINDOW_WIDTH;

	float aspectRatio = WINDOW_WIDTH / WINDOW_HEIGHT;

	float fov = 45.0f;

	// -1 ~ 1
	// tan(halfRadian)
	float xx = (NDCx * 2.0f - 1.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f) * aspectRatio;
	float yy = (NDCy * 2.0f - 1.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f);

	ray.origin = glm::vec3(-view * glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
	ray.dir = normalize(vec3(view * vec4(glm::vec3(xx, yy, -1.0), 0.0f)));

	return ray;
}

__device__ void Enqueue(Ray* rayQueue, Ray ray, int& rear)
{
	rear = (rear + 1) % QUEUE_SIZE;
	rayQueue[rear] = ray;
}

__device__ void Dequeue(Ray* rayQueue, int& front)
{
	Ray ray = rayQueue[front];
	front = (front + 1) % QUEUE_SIZE;
}

__device__ Ray GetQueueFront(Ray* rayQueue, const int front)
{
	return rayQueue[(front + 1) % QUEUE_SIZE];
}

__device__ bool IsQueueFull(const int front, const int rear)
{
	return front == (rear + 1) % QUEUE_SIZE;
}

__device__ bool IsQueueEmpty(const int front, const int rear)
{
	return front == rear;
}

// hit point, selected light, all triangles, nearestTriangleIdx
__device__ vec3 RayCastColor(
	vec3 V,
	vec3 hitPoint, 
	Light light, 
	Triangle* triangles, 
	const int triangleNum, 
	Material* materials,
	const int nearestTriangleIdx)
{
	vec3 color = glm::vec3(0.0f, 0.0f, 0.0f);

	Ray shadowRay;
	shadowRay.origin = hitPoint;
	shadowRay.dir = normalize(light.pos - hitPoint);

	bool isLighted = true;
	float tmp;
	for (int k = 0; k < triangleNum; k++)
	{
		if (nearestTriangleIdx != k)
			if (RayTriangleIntersect(shadowRay, triangles[k], tmp))
				// ������ dir�� ��
				if (tmp > 0.0001f)
					isLighted = false;
	}

	if (isLighted)
	{
		Triangle nearestTriangle = triangles[nearestTriangleIdx];

		glm::vec3 N = nearestTriangle.normal;
		glm::vec3 L = glm::normalize(light.pos - hitPoint);

		glm::vec3 matAmbient = materials[nearestTriangle.matrialId].ambient;
		glm::vec3 matDiffuse = materials[nearestTriangle.matrialId].diffuse;
		glm::vec3 matSpecular = materials[nearestTriangle.matrialId].specular;

		glm::vec3 ambient = glm::vec3(
			matAmbient.r * light.color.r,
			matAmbient.g * light.color.g,
			matAmbient.b * light.color.b);

		glm::vec3 diffuse = glm::vec3(
			matDiffuse.r * light.color.r,
			matDiffuse.g * light.color.g,
			matDiffuse.b * light.color.b) *
			glm::clamp(dot(N, L), 0.0f, 1.0f);

		glm::vec3 specular = glm::vec3(
			matSpecular.r * light.color.r,
			matSpecular.g * light.color.g,
			matSpecular.b * light.color.b) *
			glm::max(0.0f, pow(glm::max(dot(normalize(reflect(-L, N)), V), 0.0f), 16));

		color = glm::vec4(glm::vec3(ambient + diffuse + specular), 1.0f);
	}

	return color;
}

__device__ vec4 RayTraceColor(
	Ray ray,
	Ray* rayQueue,
	Triangle* triangles,
	int triangleNum,
	Light* lights,
	int lightNum,
	Material* materials,
	int matNum,
	int depth)
{
	vec4 color = vec4(0.0f);
	int front = 0, rear = 0;

	// ù ��° ray�� node�� �ϴ� queue ����
	Enqueue(rayQueue, ray, rear);

	int nowDepth = 1;

	// �� 7 (1 + 2 + 4)���� ray�� ����
	for (int i = 1; i < depth; ++i)
	{
		int target = rear;

		while (!IsQueueEmpty(target, front))
		{
			Ray nowRay;
			nowRay = GetQueueFront(rayQueue, front);
			Dequeue(rayQueue, front);

			float distToTriangle;
			int nearestTriangleIdx = FindNearestTriangleIdx(nowRay, triangles, triangleNum, distToTriangle);
			
			// �� ã�Ұų� ���ʿ� �ִٸ�
			if (nearestTriangleIdx == -1 || distToTriangle < 0.0f)
				continue;

			Triangle nearestTriangle = triangles[nearestTriangleIdx];
			glm::vec3 N = glm::normalize(nearestTriangle.normal);

			glm::vec4 lightedColor = glm::vec4(0.0f);
			glm::vec3 hitPoint = nowRay.origin + nowRay.dir * distToTriangle;

			for (int k = 0; k < lightNum; k++)
			{
				lightedColor += glm::vec4(
					RayCastColor(-nowRay.dir, hitPoint, lights[k], triangles, triangleNum, materials, nearestTriangleIdx)
					, 1.0f);
			}

			Ray reflectRay;
			reflectRay.origin = hitPoint;
			reflectRay.dir = normalize(reflect(nowRay.dir, N));

			Ray refractRay;
			refractRay.origin = hitPoint;
			refractRay.dir = normalize(refract(nowRay.dir, N, 1.2f));

			Enqueue(rayQueue, reflectRay, rear);
			Enqueue(rayQueue, refractRay, rear);

			float s = (float)glm::floor(glm::log((float)nowDepth) / glm::log(2.0f));
			color += lightedColor * pow(0.2f, s);
		}

		nowDepth++;
	}

	// ������ ���� queue�� ������ �ϱ�
	while (!IsQueueEmpty(front, rear))
	{
		Ray nowRay;
		nowRay = GetQueueFront(rayQueue, front);
		Dequeue(rayQueue, front);

		float distToTriangle;
		int nearestTriangleIdx = FindNearestTriangleIdx(nowRay, triangles, triangleNum, distToTriangle);
		
		// �� ã�Ұų� ���ʿ� �ִٸ�
		if (nearestTriangleIdx == -1 || distToTriangle < 0.0f)
			continue;

		Triangle nearestTriangle = triangles[nearestTriangleIdx];
		glm::vec3 N = normalize(nearestTriangle.normal);

		glm::vec4 lightedColor = glm::vec4(0.0f);
		glm::vec3 hitPoint = nowRay.origin + nowRay.dir * distToTriangle;

		for (int k = 0; k < lightNum; k++)
		{
			lightedColor += glm::vec4(
				RayCastColor(-nowRay.dir, hitPoint, lights[k], triangles, triangleNum, materials, nearestTriangleIdx)
				, 1.0f);
		}

		float s = (float)glm::floor(glm::log((float)nowDepth) / glm::log(2.0f));
		color += lightedColor * pow(0.2f, s);
	}

	return color;
}

// TODO view matrix�� ������ ��
__global__ void RayTraceD(
	glm::vec4* data,
	glm::mat4 view,
	Triangle* triangles, int triangleNum,
	Light* lights, int lightNum,
	Material* materials, int matNum)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	Ray ray = GenerateCameraRay(blockIdx.x, threadIdx.x, view);

	Ray rayQueue[QUEUE_SIZE];

	// y, x�� ����
	// 0, 0 ��ǥ�� ���ϴ�
	vec4 color = RayTraceColor(ray, rayQueue, triangles, triangleNum, lights, lightNum, materials, matNum, 2);

	data[x] = color;
}

void RayTrace(
	glm::vec4* data, 
	glm::mat4 view, 
	const vector<Triangle> &triangles, 
	const vector<Light>& lights,
	const vector<Material>& materials)
{
	thrust::device_vector<Triangle> t = triangles;
	thrust::device_vector<Light> l = lights;
	thrust::device_vector<Material> m = materials;

	size_t size;
	hipDeviceSetLimit(hipLimitMallocHeapSize, 10000000 * sizeof(float));
	hipDeviceGetLimit(&size, hipLimitMallocHeapSize);

	RayTraceD << <WINDOW_HEIGHT, WINDOW_WIDTH >> > (
		data,
		view,
		t.data().get(),
		t.size(),
		l.data().get(),
		l.size(),
		m.data().get(),
		m.size()
		);
}