#include "hip/hip_runtime.h"
#include "RayTracer.cuh"
#include "Octree.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm\glm.hpp>
#include <stdio.h>
#include <glm\gtc\matrix_transform.hpp>
#include <hip/hip_math_constants.h>
#include <math.h>
#include <thrust\device_vector.h>
#include <queue>

// TODO LIST
// 1. per line draw to eliminate kernel time out problem
// 2. octree acceleration
// 3. mtl file load
// 4. texture mapping with interpolation
// 5. monte calro path tracer

struct Ray
{
	// Ray�� ����
	vec3 origin;
	// Ray�� ����
	vec3 dir;
	// 0: primary, 1: reflect, 2: refract
	int rayType;

	float decay;
};

const int WINDOW_HEIGHT = 1024;
const int WINDOW_WIDTH = 1024;

const int QUEUE_SIZE = 3;

using std::cout;
using std::endl;

__device__ bool RaySphereIntersect(Ray ray, Sphere sphere, float& dist)
{
	glm::vec3 s = ray.origin - sphere.origin;

	float a = dot(ray.dir, ray.dir);
	float bPrime = dot(s, ray.dir);
	float c = dot(s, s) - sphere.radius * sphere.radius;

	float D = bPrime * bPrime - a * c;
	if (D >= 0 && bPrime <= 0)
	{
		float t1 = (-bPrime + sqrt(D)) / a;
		float t2 = (-bPrime - sqrt(D)) / a;
		dist = t1 > t2 ? t2 : t1;
		return true;
	}
	else
		return false;
}

// back face culling�� ����Ǿ� ����
__device__ bool RayTriangleIntersect(Ray ray, Triangle triangle, float& dist)
{
	glm::vec3 v0v1 = triangle.v1 - triangle.v0;
	glm::vec3 v0v2 = triangle.v2 - triangle.v0;
	glm::vec3 pvec = glm::cross(ray.dir, v0v2);

	float det = dot(v0v1, pvec);

	float epsilon = 0.0001f;

	if (det < epsilon)
		return false;

	if (fabs(det) < epsilon)
		return false;

	float invDet = 1 / det;

	glm::vec3 tvec = ray.origin - triangle.v0;
	float u = glm::dot(tvec, pvec) * invDet;
	if (u < 0 || u > 1)
		return false;

	glm::vec3 qvec = cross(tvec, v0v1);
	float v = dot(ray.dir, qvec) * invDet;
	if (v < 0 || u + v > 1)
		return false;

	dist = dot(v0v2, qvec) * invDet;

	return true;
}

__device__ bool RayAABBIntersect(Ray ray, AABB box)
{
	float tmin, tmax, tymin, tymax, tzmin, tzmax;

	glm::vec3 invdir = 1.0f / ray.dir;
	int sign[3];
	sign[0] = invdir.x < 0;
	sign[1] = invdir.y < 0;
	sign[2] = invdir.z < 0;

	tmin = (box.bounds[sign[0]].x - ray.origin.x) * invdir.x;
	tmax = (box.bounds[1 - sign[0]].x - ray.origin.x) * invdir.x;
	tymin = (box.bounds[sign[1]].y - ray.origin.y) * invdir.y;
	tymax = (box.bounds[1 - sign[1]].y - ray.origin.y) * invdir.y;

	if ((tmin > tymax) || (tymin > tmax))
		return false;

	if (tymin > tmin)
		tmin = tymin;
	if (tymax < tmax)
		tmax = tymax;

	tzmin = (box.bounds[sign[2]].z - ray.origin.z) * invdir.z;
	tzmax = (box.bounds[1 - sign[2]].z - ray.origin.z) * invdir.z;

	if ((tmin > tzmax) || (tzmin > tmax))
		return false;

	return true;
}

__device__ bool RayAABBsIntersect(Ray ray, AABB* boxes, int boxNum)
{
	bool isIntersect = false;

	for (int it = 0; it < boxNum; it++)
	{
		if (RayAABBIntersect(ray, boxes[it]))
			isIntersect = true;
	}
	return isIntersect;
}

// ���� ����� triangle�� id�� ��ȯ�ϰ� �ش� �������� dist�� �����´�
__device__ int FindNearestTriangleIdx(Ray ray, Triangle* triangles, int triangleNum, float& dist)
{
	const float rayThreshold = 0.001f;
	float minDist = 9999999.0f;
	int minIdx = -1;
	float tmpDist;

	// �״�� dist�� �����ͼ� ����ϴϱ� �̻�����
	for (int i = 0; i < triangleNum; ++i)
	{
		if (dot(triangles[i].normal, ray.dir) > 0.0f)
			continue;
		// intersect �� ���
		if (RayTriangleIntersect(ray, triangles[i], tmpDist))
		{
			// �� ã�� ���, �ٽ� ã�� �ʱ�
			if (tmpDist > rayThreshold && tmpDist < minDist)
			{
				minDist = tmpDist;
				minIdx = i;
			}
		}
	}

	dist = minDist;
	return minIdx;
}

__device__ int FindNearestSphereIdx(Ray ray, Sphere* spheres, int sphereNum, float& dist)
{
	const float rayThreshold = 0.001f;
	float minDist = 9999999.0f;
	int minIdx = -1;
	float tmpDist;

	// �״�� dist�� �����ͼ� ����ϴϱ� �̻�����
	for (int i = 0; i < sphereNum; ++i)
	{
		// intersect �� ���
		if (RaySphereIntersect(ray, spheres[i], tmpDist))
		{
			if (dot(ray.dir, ray.origin + ray.dir * tmpDist - spheres[i].origin) > 0.0f)
				continue;

			// �� ã�� ���, �ٽ� ã�� �ʱ�
			if (tmpDist > rayThreshold && tmpDist < minDist)
			{
				minDist = tmpDist;
				minIdx = i;
			}
		}
	}

	dist = minDist;
	return minIdx;
}

__device__ Ray GenerateCameraRay(int y, int x, glm::mat4 view)
{
	Ray ray;

	// 0~1
	// world ��ǥ�� ray�� ��, ������ �� window�ϼ��� ������ ���� ray�� ��
	// ���� NDC ��ǥ�� ��ȯ��
	float NDCy = (y + 0.5f) / WINDOW_HEIGHT;
	float NDCx = (x + 0.5f) / WINDOW_WIDTH;

	float aspectRatio = WINDOW_WIDTH / WINDOW_HEIGHT;

	float fov = 45.0f;

	// NDC ��ǥ�� -1 ~ 1�� ��ȯ
	// tan(halfRadian)
	// world ��ǥ���� z�� ������ 1�̱� ������ ������ ����
	float xx = (NDCx * 2.0f - 1.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f) * aspectRatio;
	float yy = (NDCy * 2.0f - 1.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f);

	// ray���� world ������ ������

	ray.origin = glm::vec3(-view * glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
	ray.dir = normalize(vec3(view * vec4(glm::vec3(xx, yy, -1.0), 0.0f)));

	return ray;
}

__device__ void Enqueue(Ray* rayQueue, Ray ray, int& rear)
{
	rear = (rear + 1) % QUEUE_SIZE;
	rayQueue[rear] = ray;
}

__device__ void Dequeue(Ray* rayQueue, int& front)
{
	Ray ray = rayQueue[front];
	front = (front + 1) % QUEUE_SIZE;
}

__device__ Ray GetQueueFront(Ray* rayQueue, const int front)
{
	return rayQueue[(front + 1) % QUEUE_SIZE];
}

__device__ bool IsQueueFull(const int front, const int rear)
{
	return front == (rear + 1) % QUEUE_SIZE;
}

__device__ bool IsQueueEmpty(const int front, const int rear)
{
	return front == rear;
}

__device__ bool IsLighted(
	vec3 hitPoint,
	Light light,
	Material *materials,
	Triangle* triangles,
	const int triangleNum,
	const int nearestTriangleIdx,
	Sphere* spheres,
	const int sphereNum,
	const int nearestSphereIdx)
{
	Ray shadowRay;
	shadowRay.origin = hitPoint;
	shadowRay.dir = normalize(light.pos - hitPoint);

	float tmp;
	for (int k = 0; k < triangleNum; ++k)
	{
		if (nearestTriangleIdx != k)
			if (RayTriangleIntersect(shadowRay, triangles[k], tmp)) {
				// ������ dir�� ��
				if (materials[triangles[k].materialId].refractivity == 0) {
					if (tmp > 0.0001f)
						return false;
				}
			}
	}

	for (int k = 0; k < sphereNum; ++k)
	{
		if (nearestSphereIdx != k)
			if (RaySphereIntersect(shadowRay, spheres[k], tmp))
				// ������ dir�� ��
				if (tmp > 0.0001f)
					return false;
	}

	return true;
}

__device__ vec3 RayCastColor(
	vec3 N,
	vec3 L,
	vec3 V,
	int rayType,
	Material material,
	Light light)
{
	vec3 color = glm::vec3(0.0f, 0.0f, 0.0f);
	
	glm::vec3 matAmbient = material.ambient;
	glm::vec3 matDiffuse = material.diffuse;
	glm::vec3 matSpecular = material.specular;

	glm::vec3 ambient = glm::vec3(
		matAmbient.r * light.color.r,
		matAmbient.g * light.color.g,
		matAmbient.b * light.color.b);

	glm::vec3 diffuse = glm::vec3(
		matDiffuse.r * light.color.r,
		matDiffuse.g * light.color.g,
		matDiffuse.b * light.color.b) *
		glm::clamp(dot(N, L), 0.0f, 1.0f);

	glm::vec3 specular = glm::vec3(
		matSpecular.r * light.color.r,
		matSpecular.g * light.color.g,
		matSpecular.b * light.color.b) *
		glm::max(0.0f, pow(glm::max(dot(normalize(reflect(-L, N)), V), 0.0f), 16));

	//switch (rayType)
	//{
	//case 0:
	//	color = glm::vec4(glm::vec3(ambient + diffuse + specular), 1.0f);
	//	break;
	//case 1: // reflect
	//	color = glm::vec4(glm::vec3(ambient + diffuse + specular), 1.0f) * material.reflectivity;
	//	break;
	//case 2: //refract
	//	color = glm::vec4(glm::vec3(ambient + diffuse + specular), 1.0f) * material.refractivity;
	//	break;
	//default:
	//	break;
	//}
	color = glm::vec4(glm::vec3(ambient + diffuse + specular), 1.0f);

	return color;
}

__device__ vec4 RayTraceColor(
	Ray ray,
	Ray* rayQueue,
	AABB* objects,
	int objNum,
	Triangle* triangles,
	int triangleNum,
	Sphere* spheres,
	int sphereNum,
	Light* lights,
	int lightNum,
	Material* materials,
	int matNum,
	int depth)
{
	vec4 color = vec4(0.0f);
	int front = 0, rear = 0;

	// ù ��° ray�� node�� �ϴ� queue ����
	Enqueue(rayQueue, ray, rear);

	int nowDepth = 1;

	// �� 7 (1 + 2 + 4)���� ray�� ����
	for (int i = 1; i < depth; ++i)
	{
		int target = rear;

		while (!IsQueueEmpty(target, front))
		{
			Ray nowRay;
			nowRay = GetQueueFront(rayQueue, front);
			Dequeue(rayQueue, front);

			/*if (!RayAABBsIntersect(nowRay, objects, objNum))
				continue;*/

			float distToTriangle, distToSphere;
			int nearestTriangleIdx = FindNearestTriangleIdx(nowRay, triangles, triangleNum, distToTriangle);
			int nearestSphereIdx = FindNearestSphereIdx(nowRay, spheres, sphereNum, distToSphere);

			// �ƹ����� intersect�� ���ߴٰų� ���ʿ� �ִٸ�
			if ((nearestTriangleIdx == -1 || distToTriangle < 0.0f) && 
				(nearestSphereIdx == -1 || distToSphere < 0.0f))
				continue;
			// ��� �ϳ��� intersect �ߴٸ�
			else
			{
				vec4 lightedColor = glm::vec4(0.0f);
				vec3 hitPoint = glm::vec3(0.0f);
				int materialId = 0;
				vec3 N = glm::vec4(0.0f);
				vec3 V = -ray.dir;

				if (distToSphere > distToTriangle)
				{
					Triangle nearestTriangle = triangles[nearestTriangleIdx];
					hitPoint = nowRay.origin + nowRay.dir * distToTriangle;
					materialId = nearestTriangle.materialId;
					N = glm::normalize(nearestTriangle.normal);
				}
				else
				{
					Sphere nearestSphere = spheres[nearestSphereIdx];
					hitPoint = nowRay.origin + nowRay.dir * distToSphere;
					materialId = nearestSphere.materialId;
					N = glm::normalize(hitPoint - nearestSphere.origin);
				}

				for (int k = 0; k < lightNum; k++)
				{
					vec3 L = glm::normalize(lights[k].pos - hitPoint);

					if (IsLighted(
						hitPoint,
						lights[k],
						materials,
						triangles, triangleNum, nearestTriangleIdx,
						spheres, sphereNum, nearestSphereIdx))
					{
						lightedColor += glm::vec4(RayCastColor(
							N, L, V, nowRay.rayType, materials[materialId], lights[k]),
							1.0f);
					}
				}

				Ray reflectRay;
				reflectRay.origin = hitPoint;
				reflectRay.dir = normalize(reflect(nowRay.dir, N));
				reflectRay.rayType = 1;
				reflectRay.decay = nowRay.decay * materials[materialId].reflectivity;

				Ray refractRay;
				refractRay.origin = hitPoint;
				refractRay.dir = normalize(refract(nowRay.dir, N, 1.2f));
				refractRay.rayType = 2;
				refractRay.decay = nowRay.decay * materials[materialId].refractivity;

				if (reflectRay.decay > 0) {
					Enqueue(rayQueue, reflectRay, rear);
				}

				if (refractRay.decay > 0) {
					Enqueue(rayQueue, refractRay, rear);
				}

				color += lightedColor * nowRay.decay;
			}
		}

		nowDepth++;
	}

	// ������ ���� queue�� ������ �ϱ�
	while (!IsQueueEmpty(front, rear))
	{
		Ray nowRay;
		nowRay = GetQueueFront(rayQueue, front);
		Dequeue(rayQueue, front);

		if (!RayAABBsIntersect(nowRay, objects, objNum))
			continue;

		float distToTriangle, distToSphere;
		int nearestTriangleIdx = FindNearestTriangleIdx(nowRay, triangles, triangleNum, distToTriangle);
		int nearestSphereIdx = FindNearestSphereIdx(nowRay, spheres, sphereNum, distToSphere);
		
		// �ƹ����� intersect�� ���ߴٰų� ���ʿ� �ִٸ�
		if ((nearestTriangleIdx == -1 || distToTriangle < 0.0f) &&
			(nearestSphereIdx == -1 || distToSphere < 0.0f))
			continue;
		// ��� �ϳ��� intersect �ߴٸ�
		else
		{
			vec4 lightedColor = glm::vec4(0.0f);
			vec3 hitPoint = glm::vec3(0.0f);
			int materialId = 0;
			vec3 N = glm::vec4(0.0f);
			vec3 V = -ray.dir;

			if (distToSphere > distToTriangle)
			{
				Triangle nearestTriangle = triangles[nearestTriangleIdx];
				hitPoint = nowRay.origin + nowRay.dir * distToTriangle;
				materialId = nearestTriangle.materialId;
				N = glm::normalize(nearestTriangle.normal);
			}
			else
			{
				Sphere nearestSphere = spheres[nearestSphereIdx];
				hitPoint = nowRay.origin + nowRay.dir * distToSphere;
				materialId = nearestSphere.materialId;
				N = glm::normalize(hitPoint - nearestSphere.origin);
			}

			for (int k = 0; k < lightNum; k++)
			{
				vec3 L = glm::normalize(lights[k].pos - hitPoint);

				if (IsLighted(
					hitPoint,
					lights[k],
					materials,
					triangles, triangleNum, nearestTriangleIdx,
					spheres, sphereNum, nearestSphereIdx))
				{
					lightedColor += glm::vec4(RayCastColor(
						N, L, V, nowRay.rayType, materials[materialId], lights[k]),
						1.0f);
				}
			}

			/*Ray reflectRay;
			reflectRay.origin = hitPoint;
			reflectRay.dir = normalize(reflect(nowRay.dir, N));
			reflectRay.rayType = 1;
			reflectRay.decay = nowRay.decay * materials[materialId].reflectivity;

			Ray refractRay;
			refractRay.origin = hitPoint;
			refractRay.dir = normalize(refract(nowRay.dir, N, 1.2f));
			refractRay.rayType = 2;
			refractRay.decay = nowRay.decay * materials[materialId].refractivity;*/

			//if (reflectRay.decay > 0) {
			//	Enqueue(rayQueue, reflectRay, rear);
			//}

			//if (refractRay.decay > 0) {
			//	Enqueue(rayQueue, refractRay, rear);
			//}

			color += lightedColor * nowRay.decay;
		}
	}

	return color;
}

__global__ void RayTraceD(
	glm::vec4* data,
	glm::mat4 view,
	OctreeNode* root,
	AABB* boundingboxes, int boxNum,
	Triangle* triangles, int triangleNum,
	Sphere* spheres, int sphereNum,
	Light* lights, int lightNum,
	Material* materials, int matNum)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	Ray ray = GenerateCameraRay(blockIdx.x, threadIdx.x, view);
	ray.rayType = 0;
	ray.decay = 1.0f;

	Ray rayQueue[QUEUE_SIZE];
	// NOTICE for���� ���� �� iter�� ������ �ϴϱ� ���� ȭ���� ����
	// y, x�� ����
	// 0, 0 ��ǥ�� ���ϴ�

	/*if (ray.dir.x < root->bndMin.x)
	{
		data[x] = glm::vec4(1.0f, 0.0f, 0.0f, 1.0f);
		return;
	}*/

	data[x] = RayTraceColor(
		ray,
		rayQueue,
		boundingboxes,
		boxNum,
		triangles,
		triangleNum,
		spheres,
		sphereNum,
		lights,
		lightNum,
		materials,
		matNum, 
		2);
}

void RayTrace(
	glm::vec4* data,
	glm::mat4 view,
	OctreeNode* root,
	const vector<AABB>& boundingboxes,
	const vector<Triangle>& triangles,
	const vector<Sphere>& spheres,
	const vector<Light>& lights,
	const vector<Material>& materials)
{
	thrust::device_vector<AABB> b = boundingboxes;
	thrust::device_vector<Triangle> t = triangles;
	thrust::device_vector<Sphere> s = spheres;
	thrust::device_vector<Light> l = lights;
	thrust::device_vector<Material> m = materials;

	hipDeviceSetLimit(hipLimitMallocHeapSize, 500000000 * sizeof(float));

	vector<Triangle> tss;
	OctreeNode* d_root = BuildOctree(tss);

	RayTraceD << <WINDOW_HEIGHT, WINDOW_WIDTH >> > (
		data,
		view,
		d_root,
		b.data().get(),
		b.size(),
		t.data().get(),
		t.size(),
		s.data().get(),
		s.size(),
		l.data().get(),
		l.size(),
		m.data().get(),
		m.size()
	);
}