#include "hip/hip_runtime.h"
#include "Test.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void makernel()
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	printf("%d", i);
	printf("Hello from mykernel\n");
}

int hello()
{
	makernel << <1, 10 >> > ();
	hipDeviceSynchronize();
	return 0;
}