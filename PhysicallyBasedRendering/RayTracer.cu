#include "hip/hip_runtime.h"
#include "RayTracer.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

struct Ray
{
	float3 origin;
	float3 dir;
};

struct Sphere
{
	float3 origin;
	float radius;
};

__device__ bool RaySphereIntersect(Ray ray, Sphere sphere, float* t)
{
	//float3 s = ray.origin - sphere.origin;
	return true;

	/*float a = dot(ray.dir, ray.dir);
	float bPrime = dot(s, ray.dir);
	float c = dot(s, s) - sphere.radius * sphere.radius;

	float D = bPrime * bPrime - a * c;
	if (D >= 0 && bPrime <= 0)
	{
		float t1 = (-bPrime + sqrt(D)) / a;
		float t2 = (-bPrime - sqrt(D)) / a;
		t = t1 > t2 ? t2 : t1;
		return true;
	}
	else
		return false;*/
}

// Camera functions
///////////////////
__device__ Ray GenerateCameraRay()
{
	Ray ray;
	//// -1 ~ 1
	//float x = (outUV.x - 0.5) * 2.0;
	//float y = (outUV.y - 0.5) * 2.0;
	//ray.dir = normalize(vec3(x, y, -1.0));

	//ray.origin = vec3(0.0);

	//// view matrix�� translate ���и� ������
	//ray.origin = (-view*vec4(vec3(0.0), 1)).xyz;
	//// view matrix�� rotate ������ ������
	//ray.dir = normalize((view*vec4(ray.dir, 0)).xyz);
	//// view matrix�� camera ������ x, y, z���� column���� �α� ������(normalize��) scale ������ �ǹ̰� ���� 

	return ray;
}

__global__ void RayTraceD(float4* data)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	float yy = ((float)blockIdx.x / (float)blockDim.x - 0.5f) * 2.0f;
	float xx = ((float)threadIdx.x / (float)blockDim.x - 0.5f) * 2.0f;

	if (xx > 0.0f)
	{
		data[x].x = 1.0f;
		data[x].y = 0.0f;
		data[x].z = 0.0f;
		data[x].w = 1.0f;
	}
	else if (xx < 0.0f && yy > 0.0f)
	{
		data[x].x = 1.0f;
		data[x].y = 1.0f;
		data[x].z = 1.0f;
		data[x].w = 1.0f;
	}
	else
	{
		data[x].x = 0.0f;
		data[x].y = 1.0f;
		data[x].z = 0.0f;
		data[x].w = 1.0f;
	}
}

void RayTrace(float4* data)
{
	RayTraceD << <1024, 1024 >> > (data);
}