#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <thrust/transform_reduce.h>
#include "RayTracer.cuh"

#define INF 999999.0f
__global__
void dkdtree::cu_create_AABB(int n, Triangle* tri, AABB* aabb)
{
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= n)
		return;
	dkdtree::AABBMax(&(tri[tid].v0), &(tri[tid].v1), &(tri[tid].v2), &(aabb[tid].bounds[1]));
	dkdtree::AABBMin(&(tri[tid].v0), &(tri[tid].v1), &(tri[tid].v2), &(aabb[tid].bounds[0]));
}

__inline__ __device__ void dkdtree::AABBMax(vec3* x, vec3* y, vec3* z, vec3* dist)
{
	float xmax = x->x>y->x ? x->x : y->x;
	xmax = xmax>z->x ? xmax : z->x;
	float ymax = x->y>y->y ? x->y : y->y;
	ymax = ymax>z->y ? ymax : z->y;
	float zmax = x->z>y->z ? x->z : y->z;
	zmax = zmax>z->z ? zmax : z->z;
	dist->x = xmax;
	dist->y = ymax;
	dist->z = zmax;
}
__inline__ __device__ void dkdtree::AABBMin(vec3* x, vec3* y, vec3* z, vec3* dist)
{
	float xmax = x->x<y->x ? x->x : y->x;
	xmax = xmax<z->x ? xmax : z->x;
	float ymax = x->y<y->y ? x->y : y->y;
	ymax = ymax<z->y ? ymax : z->y;
	float zmax = x->z<y->z ? x->z : y->z;
	zmax = zmax<z->z ? zmax : z->z;
	dist->x = xmax;
	dist->y = ymax;
	dist->z = zmax;
}

gpukdtree::gpukdtree(Triangle* tri, int n, AABB rootaabb)
{
	h_Triangles = tri;
	nTriangle = n;
	rootAABB = rootaabb;
}
gpukdtree::~gpukdtree()
{
	freeMemory();
}
void gpukdtree::allocateMemory()
{
	hipMalloc((void**)&d_Triangles, sizeof(Triangle)*nTriangle);
	hipMalloc((void**)&d_AABB, sizeof(AABB)*nTriangle);
	hipMemcpy(d_Triangles, h_Triangles, sizeof(Triangle)*nTriangle, hipMemcpyHostToDevice);

	

	nodes.allocateMemory(nTriangle / 3);
	triangleNodeAssociation.allocateMemory(nTriangle * 30);
	triangleNodeAssociationHelper.allocateMemory(nTriangle * 10);
	activeList.allocateMemory(nTriangle / 3);
	nextList.allocateMemory(nTriangle / 3);
	smallList.allocateMemory(nTriangle / 3);
}

void gpukdtree::freeMemory()
{
	hipFree(d_Triangles);
	hipFree(d_AABB);
}



void gpukdtree::create()
{
	hipError_t err = hipSuccess;
	int blocksize = (nTriangle + 255) / 256;

	allocateMemory();

	//cout << "memcpy on gpu" << endl;
	// calculate AABB
	dkdtree::cu_create_AABB << <blocksize, 256 >> >(nTriangle, d_Triangles, d_AABB);
	hipDeviceSynchronize();

	//create kd tree
	MidSplit();
	SAHSplit();
	//cout<<"gpu kdtree debug info:"<<endl;
	//cout<<nodes.size()<<endl;
	//cout<<triangleNodeAssociation.size()<<endl;

	err = hipGetLastError();
	if (err != hipSuccess)cout << hipGetErrorString(err) << endl;
}

void gpukdtree::MidSplit()
{
	//dkdtree::InitRoot<<<1,1>>>(nTriangle, nodes.data, nodes.d_ptr, activeList.data, activeList.d_ptr, nextList.d_ptr, triangleNodeAssociation.d_ptr, CalculateRootAABB());
	dkdtree::InitRoot << <1, 1 >> >(nTriangle, nodes.data, nodes.d_ptr, activeList.data, activeList.d_ptr, nextList.d_ptr, smallList.d_ptr, triangleNodeAssociation.d_ptr, rootAABB);
	hipDeviceSynchronize();

	dkdtree::CopyTriangle << <(nTriangle + 255) / 256, 256 >> >(triangleNodeAssociation.data, nTriangle);
	hipDeviceSynchronize();


	while (!activeList.h_empty())
	{
		int base = nodes.size() - 1;
		int startnode = nodes.size();
		int start = triangleNodeAssociation.size();
		triangleNodeAssociationHelper.h_clear();
		dkdtree::MidSplitNode << <(activeList.size() + 255) / 256, 256 >> >(d_Triangles, d_AABB, nTriangle,
			nodes.data,
			nodes.d_ptr,
			activeList.data,
			activeList.d_ptr,
			nextList.data,
			nextList.d_ptr,
			smallList.data,
			smallList.d_ptr,
			triangleNodeAssociation.data,
			triangleNodeAssociation.d_ptr,
			triangleNodeAssociationHelper.data,
			triangleNodeAssociationHelper.d_ptr,
			start);
		hipDeviceSynchronize();
		int end = triangleNodeAssociation.size();
		int endnode = nodes.size() - 1;
		int noftna = end - start;
		thrust::sort_by_key(triangleNodeAssociationHelper.thrustPtr, triangleNodeAssociationHelper.thrustPtr + noftna, triangleNodeAssociation.thrustPtr + start);
		hipDeviceSynchronize();
		// calculate triangleIndex
		dkdtree::CalculateTriangleIndex << <1, 1 >> >(startnode, endnode, base, nodes.data);
		hipDeviceSynchronize();
		// switch aciveList and nextList
		//cout<<"nextlist size:"<<nextList.size()<<" tnasize="<<noftna<<endl;
		hipMemcpy(activeList.data, nextList.data, sizeof(int)*nextList.size(), hipMemcpyDeviceToDevice);
		hipMemcpy(activeList.d_ptr, nextList.d_ptr, sizeof(unsigned int), hipMemcpyDeviceToDevice);

		nextList.h_clear();
		triangleNodeAssociationHelper.h_clear();
		hipDeviceSynchronize();
	}
}

void gpukdtree::SAHSplit()
{
	while (!smallList.h_empty())
	{
		int base = nodes.size() - 1;
		int startnode = nodes.size();
		int start = triangleNodeAssociation.size();
		triangleNodeAssociationHelper.h_clear();
		dkdtree::SAHSplitNode << <(smallList.size() + 255) / 256, 256 >> >(d_Triangles, d_AABB, nTriangle,
			nodes.data,
			nodes.d_ptr,
			smallList.data,
			smallList.d_ptr,
			nextList.data,
			nextList.d_ptr,
			triangleNodeAssociation.data,
			triangleNodeAssociation.d_ptr,
			triangleNodeAssociationHelper.data,
			triangleNodeAssociationHelper.d_ptr,
			start);
		hipDeviceSynchronize();
		int end = triangleNodeAssociation.size();
		int endnode = nodes.size() - 1;
		int noftna = end - start;
		thrust::sort_by_key(triangleNodeAssociationHelper.thrustPtr, triangleNodeAssociationHelper.thrustPtr + noftna, triangleNodeAssociation.thrustPtr + start);
		hipDeviceSynchronize();
		// calculate triangleIndex
		dkdtree::CalculateTriangleIndex << <1, 1 >> >(startnode, endnode, base, nodes.data);
		hipDeviceSynchronize();
		// switch aciveList and nextList
		//cout<<"nextlist size:"<<nextList.size()<<" tnasize="<<noftna<<endl;
		hipMemcpy(smallList.data, nextList.data, sizeof(int)*nextList.size(), hipMemcpyDeviceToDevice);
		hipMemcpy(smallList.d_ptr, nextList.d_ptr, sizeof(unsigned int), hipMemcpyDeviceToDevice);

		nextList.h_clear();
		triangleNodeAssociationHelper.h_clear();
		hipDeviceSynchronize();
	}
}

__global__ void dkdtree::SAHSplitNode(Triangle* tri,
	AABB* aabb,
	int nTri,
	gpukdtreeNode* nodes,
	unsigned int* nodesPtr,
	int* smallList,
	unsigned int* smallListPtr,
	int* nextList,
	unsigned int* nextListPtr,
	int* tna,
	unsigned int* tnaPtr,
	int* tnahelper,
	unsigned int* tnahelperPtr,
	unsigned int tnaStartPtr)
{
	unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid >= *smallListPtr)
		return;
	//printf("tid=%d\n",tid);
	int id = smallList[tid];
	//printf("node triangle number=%d\n",nodes[id].triangleNumber);
	int leftid;
	int rightid;
	float tpos;
	//gpukdtreeNode currentNode(nodes[id]);
	vec3 volume = nodes[id].nodeAABB.bounds[1] - nodes[id].nodeAABB.bounds[0];
	if (volume.x >= volume.y && volume.x >= volume.z)// split x
	{
		nodes[id].splitAxis = 0;
		// looking for best candidate
		float minsah = 999999.0f;
		float minpos;

		for (float p = 0.1f; p<1.0f; p += 0.1f) {
			tpos = nodes[id].nodeAABB.bounds[0].x + volume.x*p;
			int ct1, ct2;
			ct1 = ct2 = 0;
			for (int i = nodes[id].triangleIndex, j = 0; j<nodes[id].triangleNumber; i++, j++) {
				if ((aabb[tnaPtr[i]].bounds[0].x + aabb[tnaPtr[i]].bounds[1].x) / 2<tpos)
					ct1++;
				else
					ct2++;
			}
			float sah = ct1*p + ct2*(1 - p);
			if (sah<minsah) {
				minsah = sah;
				minpos = tpos;
			}
		}
		nodes[id].splitPos = tpos;

		gpukdtreeNode atarashiiNode;
		atarashiiNode.nodeAABB = nodes[id].nodeAABB;
		atarashiiNode.nodeAABB.bounds[1].x = tpos;
		leftid = DeviceVector<gpukdtreeNode>::push_back(nodes, nodesPtr, atarashiiNode);
		nodes[id].leftChild = leftid;

		atarashiiNode.nodeAABB.bounds[1].x = nodes[id].nodeAABB.bounds[1].x;
		atarashiiNode.nodeAABB.bounds[0].x = tpos;
		rightid = DeviceVector<gpukdtreeNode>::push_back(nodes, nodesPtr, atarashiiNode);
		nodes[id].rightChild = rightid;
	}
	else if (volume.y >= volume.x && volume.y >= volume.z)// split y
	{
		nodes[id].splitAxis = 1;
		// looking for best candidate
		float minsah = 999999.0f;
		float minpos;

		for (float p = 0.1f; p<1.0f; p += 0.1f) {
			tpos = nodes[id].nodeAABB.bounds[0].y + volume.y*p;
			int ct1, ct2;
			ct1 = ct2 = 0;
			for (int i = nodes[id].triangleIndex, j = 0; j<nodes[id].triangleNumber; i++, j++) {
				if ((aabb[tnaPtr[i]].bounds[0].y + aabb[tnaPtr[i]].bounds[1].y) / 2<tpos)
					ct1++;
				else
					ct2++;
			}
			float sah = ct1*p + ct2*(1 - p);
			if (sah<minsah) {
				minsah = sah;
				minpos = tpos;
			}
		}
		nodes[id].splitPos = tpos;

		gpukdtreeNode atarashiiNode;
		atarashiiNode.nodeAABB = nodes[id].nodeAABB;
		atarashiiNode.nodeAABB.bounds[1].y = tpos;
		leftid = DeviceVector<gpukdtreeNode>::push_back(nodes, nodesPtr, atarashiiNode);
		nodes[id].leftChild = leftid;

		atarashiiNode.nodeAABB.bounds[1].y = nodes[id].nodeAABB.bounds[1].y;
		atarashiiNode.nodeAABB.bounds[0].y = tpos;
		rightid = DeviceVector<gpukdtreeNode>::push_back(nodes, nodesPtr, atarashiiNode);
		nodes[id].rightChild = rightid;
	}
	else // split z
	{
		nodes[id].splitAxis = 2;
		// looking for best candidate
		float minsah = 999999.0f;
		float minpos;

		for (float p = 0.1f; p<1.0f; p += 0.1f) {
			tpos = nodes[id].nodeAABB.bounds[0].z + volume.z*p;
			int ct1, ct2;
			ct1 = ct2 = 0;
			for (int i = nodes[id].triangleIndex, j = 0; j<nodes[id].triangleNumber; i++, j++) {
				if ((aabb[tnaPtr[i]].bounds[0].z + aabb[tnaPtr[i]].bounds[1].z) / 2<tpos)
					ct1++;
				else
					ct2++;
			}
			float sah = ct1*p + ct2*(1 - p);
			if (sah<minsah) {
				minsah = sah;
				minpos = tpos;
			}
		}
		nodes[id].splitPos = tpos;

		gpukdtreeNode atarashiiNode;
		atarashiiNode.nodeAABB = nodes[id].nodeAABB;
		atarashiiNode.nodeAABB.bounds[1].z = tpos;
		leftid = DeviceVector<gpukdtreeNode>::push_back(nodes, nodesPtr, atarashiiNode);
		nodes[id].leftChild = leftid;

		atarashiiNode.nodeAABB.bounds[1].z = nodes[id].nodeAABB.bounds[1].z;
		atarashiiNode.nodeAABB.bounds[0].z = tpos;
		rightid = DeviceVector<gpukdtreeNode>::push_back(nodes, nodesPtr, atarashiiNode);
		nodes[id].rightChild = rightid;
	}
	//printf("sp=%.3f\n",sp);
	// split triangles
	int leftcount = 0;
	int rightcount = 0;
	unsigned int tnapos;
	int endPtr = nodes[id].triangleIndex + nodes[id].triangleNumber - 1;
	/*printf("triangleIndex=%d\n", currentNode.triangleIndex);
	printf("triangleNumber=%d\n", currentNode.triangleNumber);
	printf("endPtr=%d\n", endPtr);*/
	for (int i = nodes[id].triangleIndex; i <= endPtr; i++)
	{
		int triid = tna[i];

		switch (nodes[id].splitAxis)
		{
		case 0:
			if (aabb[triid].bounds[0].x <= tpos) {
				tnapos = DeviceVector<int>::push_back(tna, tnaPtr, triid);
				//DeviceVector<int>::push_back(tnahelper, tnahelperPtr, leftid);
				tnahelper[tnapos - tnaStartPtr] = leftid;
				leftcount++;
			}
			if (aabb[triid].bounds[1].x >= tpos) {
				tnapos = DeviceVector<int>::push_back(tna, tnaPtr, triid);
				tnahelper[tnapos - tnaStartPtr] = rightid;
				rightcount++;
			}
			break;
		case 1:
			if (aabb[triid].bounds[0].y <= tpos) {
				tnapos = DeviceVector<int>::push_back(tna, tnaPtr, triid);
				tnahelper[tnapos - tnaStartPtr] = leftid;
				leftcount++;
			}
			if (aabb[triid].bounds[1].y >= tpos) {
				tnapos = DeviceVector<int>::push_back(tna, tnaPtr, triid);
				tnahelper[tnapos - tnaStartPtr] = rightid;
				rightcount++;
			}
			break;
		case 2:
			if (aabb[triid].bounds[0].z <= tpos) {
				tnapos = DeviceVector<int>::push_back(tna, tnaPtr, triid);
				tnahelper[tnapos - tnaStartPtr] = leftid;
				leftcount++;
			}
			if (aabb[triid].bounds[1].z >= tpos) {
				tnapos = DeviceVector<int>::push_back(tna, tnaPtr, triid);
				tnahelper[tnapos - tnaStartPtr] = rightid;
				rightcount++;
			}
			break;
		}
	}
	//printf("leftcount=%d\nrightcount=%d\n", leftcount, rightcount);
	nodes[leftid].triangleNumber = leftcount;
	nodes[rightid].triangleNumber = rightcount;
	//printf("node %d was splited with left = %d and right = %d with sp=%.5f tna=%d\n", id, leftcount, rightcount, sp, *tnaPtr);
	// add to nextList

	if (leftcount>GPUKDTREETHRESHOLD)
		DeviceVector<int>::push_back(smallList, smallListPtr, leftid);

	if (rightcount>GPUKDTREETHRESHOLD)
		DeviceVector<int>::push_back(smallList, smallListPtr, rightid);
}

void gpukdtree::IntersectRay(const Ray* r, int n, float* dist, int* iid)
{
	Ray* d_r;
	float* d_dist;
	int* d_iid;
	hipMalloc((void**)&d_r, sizeof(Ray)*n);
	hipMalloc((void**)&d_dist, sizeof(float)*n);
	hipMalloc((void**)&d_iid, sizeof(int)*n);
	hipMemcpy(d_r, r, sizeof(Ray)*n, hipMemcpyHostToDevice);
	hipMemcpy(d_dist, dist, sizeof(float)*n, hipMemcpyHostToDevice);
	hipMemcpy(d_iid, iid, sizeof(int)*n, hipMemcpyHostToDevice);
	dkdtree::IntersectRay << <(n + 255) / 256, 256 >> >(d_r, n, d_dist, d_iid, nodes.data, d_Triangles, triangleNodeAssociation.data);
	hipDeviceSynchronize();
	hipMemcpy(dist, d_dist, sizeof(float)*n, hipMemcpyDeviceToHost);
	hipMemcpy(iid, d_iid, sizeof(int)*n, hipMemcpyDeviceToHost);
	hipFree(d_r);
	hipFree(d_dist);
	hipFree(d_iid);
}

AABB gpukdtree::CalculateRootAABB()
{
	thrust::device_ptr<AABB> thrustPtr(d_AABB);
	float maxx = thrust::transform_reduce(thrustPtr, thrustPtr + nTriangle, dkdtree::MaxX(), 0, thrust::maximum<float>());
	float maxy = thrust::transform_reduce(thrustPtr, thrustPtr + nTriangle, dkdtree::MaxY(), 0, thrust::maximum<float>());
	float maxz = thrust::transform_reduce(thrustPtr, thrustPtr + nTriangle, dkdtree::MaxZ(), 0, thrust::maximum<float>());
	float minx = thrust::transform_reduce(thrustPtr, thrustPtr + nTriangle, dkdtree::MinX(), 0, thrust::minimum<float>());
	float miny = thrust::transform_reduce(thrustPtr, thrustPtr + nTriangle, dkdtree::MinY(), 0, thrust::minimum<float>());
	float minz = thrust::transform_reduce(thrustPtr, thrustPtr + nTriangle, dkdtree::MinZ(), 0, thrust::minimum<float>());
	hipDeviceSynchronize();

	AABB tmp;

	tmp.bounds[0] = vec3(minx, miny, minz);
	tmp.bounds[1] = vec3(maxx, maxy, maxz);

	return tmp;
}

__global__ void dkdtree::InitRoot(int nTri,
	gpukdtreeNode* nodes,
	unsigned int* nodesPtr,
	int* activeList,
	unsigned int* activeListPtr,
	unsigned int* nextListPtr,
	unsigned int* smallListPtr,
	unsigned int* tnaPtr,
	AABB aabb)
{
	DeviceVector<int>::clear(activeListPtr);
	DeviceVector<int>::clear(nextListPtr);
	DeviceVector<int>::clear(smallListPtr);
	DeviceVector<int>::clear(tnaPtr);
	DeviceVector<gpukdtreeNode>::clear(nodesPtr);

	

	gpukdtreeNode n;
	n.triangleIndex = 0;
	n.triangleNumber = nTri;
	n.nodeAABB = aabb;
	DeviceVector<gpukdtreeNode>::push_back(nodes, nodesPtr, n);
	*(tnaPtr) = nTri;

	int i = 0;
	DeviceVector<int>::push_back(activeList, activeListPtr, i);
}

__global__ void dkdtree::MidSplitNode(Triangle* tri,
	AABB* aabb,
	int nTri,
	gpukdtreeNode* nodes,
	unsigned int* nodesPtr,
	int* activeList,
	unsigned int* activeListPtr,
	int* nextList,
	unsigned int* nextListPtr,
	int* smallList,
	unsigned int* smallListPtr,
	int* tna,
	unsigned int* tnaPtr,
	int* tnahelper,
	unsigned int* tnahelperPtr,
	unsigned int tnaStartPtr)
{
	unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid >= *activeListPtr)
		return;
	//printf("tid=%d\n",tid);
	int id = activeList[tid];
	//printf("node triangle number=%d\n",nodes[id].triangleNumber);
	int leftid;
	int rightid;
	float sp;
	//gpukdtreeNode currentNode(nodes[id]);
	vec3 volume = nodes[id].nodeAABB.bounds[1] - nodes[id].nodeAABB.bounds[0];
	if (volume.x >= volume.y && volume.x >= volume.z)// split x
	{
		nodes[id].splitAxis = 0;
		sp = nodes[id].nodeAABB.bounds[0].x + volume.x / 2;
		nodes[id].splitPos = sp;

		gpukdtreeNode atarashiiNode;
		atarashiiNode.nodeAABB = nodes[id].nodeAABB;
		atarashiiNode.nodeAABB.bounds[1].x = sp;
		leftid = DeviceVector<gpukdtreeNode>::push_back(nodes, nodesPtr, atarashiiNode);
		nodes[id].leftChild = leftid;

		atarashiiNode.nodeAABB.bounds[1].x = nodes[id].nodeAABB.bounds[1].x;
		atarashiiNode.nodeAABB.bounds[0].x = sp;
		rightid = DeviceVector<gpukdtreeNode>::push_back(nodes, nodesPtr, atarashiiNode);
		nodes[id].rightChild = rightid;
	}
	else if (volume.y >= volume.x && volume.y >= volume.z)// split y
	{
		nodes[id].splitAxis = 1;
		sp = nodes[id].nodeAABB.bounds[0].y + volume.y / 2;
		nodes[id].splitPos = sp;

		gpukdtreeNode atarashiiNode;
		atarashiiNode.nodeAABB = nodes[id].nodeAABB;
		atarashiiNode.nodeAABB.bounds[1].y = sp;
		leftid = DeviceVector<gpukdtreeNode>::push_back(nodes, nodesPtr, atarashiiNode);
		nodes[id].leftChild = leftid;

		atarashiiNode.nodeAABB.bounds[1].y = nodes[id].nodeAABB.bounds[1].y;
		atarashiiNode.nodeAABB.bounds[0].y = sp;
		rightid = DeviceVector<gpukdtreeNode>::push_back(nodes, nodesPtr, atarashiiNode);
		nodes[id].rightChild = rightid;
	}
	else // split z
	{
		nodes[id].splitAxis = 2;
		sp = nodes[id].nodeAABB.bounds[0].z + volume.z / 2;
		nodes[id].splitPos = sp;

		gpukdtreeNode atarashiiNode;
		atarashiiNode.nodeAABB = nodes[id].nodeAABB;
		atarashiiNode.nodeAABB.bounds[1].z = sp;
		leftid = DeviceVector<gpukdtreeNode>::push_back(nodes, nodesPtr, atarashiiNode);
		nodes[id].leftChild = leftid;

		atarashiiNode.nodeAABB.bounds[1].z = nodes[id].nodeAABB.bounds[1].z;
		atarashiiNode.nodeAABB.bounds[0].z = sp;
		rightid = DeviceVector<gpukdtreeNode>::push_back(nodes, nodesPtr, atarashiiNode);
		nodes[id].rightChild = rightid;
	}
	//printf("sp=%.3f\n",sp);
	// split triangles
	int leftcount = 0;
	int rightcount = 0;
	unsigned int tnapos;
	int endPtr = nodes[id].triangleIndex + nodes[id].triangleNumber - 1;
	/*printf("triangleIndex=%d\n", currentNode.triangleIndex);
	printf("triangleNumber=%d\n", currentNode.triangleNumber);
	printf("endPtr=%d\n", endPtr);*/
	for (int i = nodes[id].triangleIndex; i <= endPtr; i++)
	{
		int triid = tna[i];

		switch (nodes[id].splitAxis)
		{
		case 0:
			if (aabb[triid].bounds[0].x <= sp) {
				tnapos = DeviceVector<int>::push_back(tna, tnaPtr, triid);
				//DeviceVector<int>::push_back(tnahelper, tnahelperPtr, leftid);
				tnahelper[tnapos - tnaStartPtr] = leftid;
				leftcount++;
			}
			if (aabb[triid].bounds[1].x >= sp) {
				tnapos = DeviceVector<int>::push_back(tna, tnaPtr, triid);
				tnahelper[tnapos - tnaStartPtr] = rightid;
				rightcount++;
			}
			break;
		case 1:
			if (aabb[triid].bounds[0].y <= sp) {
				tnapos = DeviceVector<int>::push_back(tna, tnaPtr, triid);
				tnahelper[tnapos - tnaStartPtr] = leftid;
				leftcount++;
			}
			if (aabb[triid].bounds[1].y >= sp) {
				tnapos = DeviceVector<int>::push_back(tna, tnaPtr, triid);
				tnahelper[tnapos - tnaStartPtr] = rightid;
				rightcount++;
			}
			break;
		case 2:
			if (aabb[triid].bounds[0].z <= sp) {
				tnapos = DeviceVector<int>::push_back(tna, tnaPtr, triid);
				tnahelper[tnapos - tnaStartPtr] = leftid;
				leftcount++;
			}
			if (aabb[triid].bounds[1].z >= sp) {
				tnapos = DeviceVector<int>::push_back(tna, tnaPtr, triid);
				tnahelper[tnapos - tnaStartPtr] = rightid;
				rightcount++;
			}
			break;
		}
	}
	//printf("leftcount=%d\nrightcount=%d\n", leftcount, rightcount);
	nodes[leftid].triangleNumber = leftcount;
	nodes[rightid].triangleNumber = rightcount;
	//printf("node %d was splited with left = %d and right = %d with sp=%.5f tna=%d\n", id, leftcount, rightcount, sp, *tnaPtr);
	// add to nextList
	if (leftcount>GPUKDTREETHRESHOLD * 2)
		DeviceVector<int>::push_back(nextList, nextListPtr, leftid);
	else if (leftcount>GPUKDTREETHRESHOLD)
		DeviceVector<int>::push_back(smallList, smallListPtr, leftid);
	if (rightcount>GPUKDTREETHRESHOLD * 2)
		DeviceVector<int>::push_back(nextList, nextListPtr, rightid);
	else if (rightcount>GPUKDTREETHRESHOLD)
		DeviceVector<int>::push_back(smallList, smallListPtr, rightid);
}

__global__ void dkdtree::CalculateTriangleIndex(int start, int end, int base, gpukdtreeNode* nodes)
{
	int count = 0;
	int basecount = nodes[base].triangleIndex + nodes[base].triangleNumber;
	for (int i = start; i <= end; i++)
	{
		nodes[i].triangleIndex = basecount + count;
		count += nodes[i].triangleNumber;
	}
}

__global__ void dkdtree::CopyTriangle(int* tna, int n)
{
	unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid >= n)
		return;
	tna[tid] = tid;
}

__global__ void dkdtree::IntersectRay(const Ray* ray, int n, float* dist, int* iid, gpukdtreeNode* nodes, Triangle* tri, int* tna)
{
	unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid >= n)
		return;
	float mindist = INF;
	float cdist;
	int currentid, leftid, rightid, cid;
	Ray r = ray[tid];
	iid[tid] = -1;

	DeviceStack<int> treestack;
	treestack.push(0);
	while (!treestack.empty())
	{
		currentid = treestack.pop();

		//test node intersection
		if (Intersect_nodeAABB_Ray(r, currentid, nodes)) {
			leftid = nodes[currentid].leftChild;
			rightid = nodes[currentid].rightChild;
			// leaf node
			if (leftid == -1 && rightid == -1) {
				if (Intersect_nodeTriangles_Ray(r, currentid, cdist, cid, nodes, tri, tna)) {
					if (cdist<mindist) {
						mindist = cdist;
						iid[tid] = cid;
					}
				}
				continue;
			}
			// middle node
			if (leftid != -1)
				treestack.push(leftid);
			if (rightid != -1)
				treestack.push(rightid);
		}
	}
	dist[tid] = mindist;
}

__device__ bool Intersect_nodeAABB_Ray(const Ray& r, int id, gpukdtreeNode* nodes)
{
	bool intersection = true;
	float p_near_result = -FLT_MAX;
	float p_far_result = FLT_MAX;
	float p_near_comp, p_far_comp;
	AABB aabb(nodes[id].nodeAABB);

	vec3 inv_dir(1.0 / r.dir.x, 1.0 / r.dir.y, 1.0 / r.dir.z);

	for (int i = 0; i<3; i++)
	{
		switch (i)
		{
		case 0:
			p_near_comp = (aabb.bounds[0].x - r.origin.x) * inv_dir.x;
			p_far_comp = (aabb.bounds[1].x - r.origin.x) * inv_dir.x;
			break;
		case 1:
			p_near_comp = (aabb.bounds[0].y - r.origin.y) * inv_dir.y;
			p_far_comp = (aabb.bounds[1].y - r.origin.y) * inv_dir.y;
			break;
		case 2:
			p_near_comp = (aabb.bounds[0].z - r.origin.z) * inv_dir.z;
			p_far_comp = (aabb.bounds[1].z - r.origin.z) * inv_dir.z;
			break;
		}

		if (p_near_comp > p_far_comp) {
			float temp = p_near_comp;
			p_near_comp = p_far_comp;
			p_far_comp = temp;
		}

		p_near_result = ((p_near_comp > p_near_result) ? p_near_comp : p_near_result);
		p_far_result = ((p_far_comp < p_far_result) ? p_far_comp : p_far_result);

		if (p_near_result > p_far_result)
			intersection = false;
	}

	return intersection;
}
__device__ __host__ bool intersect_triangle(Ray ray, Triangle triangle, float& dist)
{
	glm::vec3 v0v1 = triangle.v1 - triangle.v0;
	glm::vec3 v0v2 = triangle.v2 - triangle.v0;
	glm::vec3 pvec = glm::cross(ray.dir, v0v2);

	float det = dot(v0v1, pvec);

	// back face culling
	if (det < 0.001f)
		return false;

	/*if (fabsf(det) < 0.01f)
	return false;*/

	float invDet = 1 / det;

	glm::vec3 tvec = ray.origin - triangle.v0;
	float u = glm::dot(tvec, pvec) * invDet;
	if (u < 0 || u > 1)
		return false;

	glm::vec3 qvec = cross(tvec, v0v1);
	float v = dot(ray.dir, qvec) * invDet;
	if (v < 0 || u + v > 1)
		return false;

	dist = dot(v0v2, qvec) * invDet;

	return dist > 0.001f;
}

__device__ bool Intersect_nodeTriangles_Ray(const Ray& r, int id, float& dist, int& iid, gpukdtreeNode* nodes, Triangle* tri, int* tna)
{
	bool intersection = false;
	float cdist;
	float mindist = INF;
	int n = nodes[id].triangleIndex + nodes[id].triangleNumber - 1;

	for (int i = nodes[id].triangleIndex; i <= n; i++)
	{
		if (intersect_triangle(r, tri[tna[i]], cdist)) {
			if (cdist<mindist) {
				mindist = cdist;
				iid = tna[i];
				intersection = true;
			}
		}
	}
	dist = mindist;
	return intersection;
}