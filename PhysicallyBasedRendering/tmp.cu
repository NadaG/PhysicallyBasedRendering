
#include <hip/hip_runtime.h>
//#include "RayTracingRenderer.h"
//#include "LectureSceneRenderer.h"
//#include "PBRRenderer.h"
//#include "LTCRenderer.h"
//#include "FluidRenderer.h"
//
//#include "WindowManager.h"
//#include "Debug.h"
//#include "RayTracer.cuh"
//
//#include "Octree.cuh"
//
//#define STB_IMAGE_IMPLEMENTATION
//#include "stb_image.h"
//
//using namespace std;
//
//enum Scene
//{
//	PBR_SCENE = 0,
//	FLUID_SCENE = 1,
//	LTC_SCENE = 2,
//	SMOKE_SCENE = 3,
//	STARBURST_SCENE = 4,
//	TEMPORALGLARE_SCENE = 5,
//	RAYTRACING_SCENE = 6
//};
//
//// TODO movement �����ϰ� ����ϴ� �κ� �ʹ� ������, �Լ��� �������� �ؾ���
//// TODO scene object update�ϰ� render �ϴ� �κ� �̻���, renderer���� ���� object�� �� �����;��ϴ� ���� �ذ�
//// TODO shader �ҷ����� ����ϰ� �׸��� �κ� �ߺ��Ǵ� �κ��� �ʹ� ������ �� �κе� �����ϱ�
//
//struct BTNode
//{
//	int data = NULL;
//
//	BTNode *left = nullptr;
//	BTNode *right = nullptr;
//};
//
//
//BTNode* CreateNode(int data)
//{
//	BTNode *node = new BTNode;
//	node->data = data;
//
//	return node;
//}
//
//
//void InsertLeft(BTNode *parent, BTNode *left)
//{
//	if (parent->left != nullptr)
//		return;
//
//	parent->left = left;
//}
//
//void InsertRight(BTNode *parent, BTNode *right)
//{
//	if (parent->right != nullptr)
//		return;
//
//	parent->right = right;
//}
//
//void Traversal(BTNode *node)
//{
//	cout << "data : " << node->data << endl;
//
//	if (node->left != nullptr)
//		Traversal(node->left);
//	if (node->right != nullptr)
//		Traversal(node->right);
//
//}
//
//void DeleteTree(BTNode *node)
//{
//	if (node->left != nullptr)
//		DeleteTree(node->left);
//
//	if (node->right != nullptr)
//		DeleteTree(node->right);
//
//	delete node;
//}
//
//
//const int BTSize = sizeof(BTNode);
//
//BTNode* BTHostToDevice(BTNode *node)
//{
//	if (node == nullptr)
//		return nullptr;
//
//	node->left = BTHostToDevice(node->left);
//	node->right = BTHostToDevice(node->right);
//
//	BTNode* gnode;
//	cudaMalloc((void**)&gnode, BTSize);
//	cudaMemcpy(gnode, node, BTSize, cudaMemcpyHostToDevice);
//
//	return gnode;
//}
//
//BTNode* BTDeviceToHost(BTNode *node)
//{
//	if (node == nullptr)
//		return nullptr;
//
//	BTNode* cnode = new BTNode;
//	cudaMemcpy(cnode, node, BTSize, cudaMemcpyDeviceToHost);
//
//	cnode->left = BTDeviceToHost(cnode->left);
//	cnode->right = BTDeviceToHost(cnode->right);
//
//	return cnode;
//}
//
//__global__
//void Mull(BTNode *node)
//{
//	if (threadIdx.x == 0)
//		node->data = node->data * 0;
//	else if (threadIdx.x == 1)
//		node->left->data = node->left->data * 1;
//	else if (threadIdx.x == 2)
//		node->right->data = node->right->data * 2;
//}
//
//
//
//int main(int argc, char **argv)
//{
//	BTNode *node = CreateNode(1);
//
//	InsertLeft(node, CreateNode(2));
//	InsertRight(node, CreateNode(3));
//
//	Traversal(node);
//
//	BTNode *gnode = BTHostToDevice(node);
//
//	dim3 dimBlock(3, 1);
//	dim3 dimGrid(1, 1);
//	
//
//	Mull << < dimGrid, dimBlock >> >(gnode);
//	//Traversal(node);
//
//	//cudaMemcpy(node, gnode, BTSize, cudaMemcpyDeviceToHost);
//	/*cudaMemcpy(node->hostRight, gnode->deviceRight, BTSize, cudaMemcpyDeviceToHost);
//	if (gnode->deviceLeft == nullptr)
//		cout << "dd" << endl;*/
//	//cout << gnode->data << endl;
//	//cout << node->hostLeft << endl;
//
//	node = BTDeviceToHost(gnode);
//	Traversal(node);
//
//
//	tmpfunc();
//
//	/*Scene scene = RAYTRACING_SCENE;
//
//	WindowManager::GetInstance()->Initialize();
//
//	WindowManager::GetInstance()->WindowHint(GLFW_SAMPLES, 4);
//	WindowManager::GetInstance()->WindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
//	WindowManager::GetInstance()->WindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
//	WindowManager::GetInstance()->WindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
//	WindowManager::GetInstance()->WindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
//	GLFWwindow* window = WindowManager::GetInstance()->CreateMyWindow(1024, 1024, "OpenGL");
//
//	InputManager::GetInstance()->Initialize(window);
//
//	TO Refacto ��򰡷� ����������
//	glewExperimental = true;
//
//	if (glewInit() != GLEW_OK)
//	{
//	fprintf(stderr, "Failed to initialize GLEW\n");
//	return 0;
//	}
//
//	Renderer* renderer;
//	SceneManager* sceneManager;
//
//	switch (scene)
//	{
//	case RAYTRACING_SCENE:
//	{
//	sceneManager = new RayTracingSceneManager();
//	renderer = new RayTracingRenderer(sceneManager);
//	break;
//	}
//	case FLUID_SCENE:
//	{
//	sceneManager = new FluidSceneManager();
//	renderer = new FluidRenderer(sceneManager);
//	break;
//	}
//	case LTC_SCENE:
//	{
//	sceneManager = new LTCSceneManager();
//	renderer = new LTCRenderer(sceneManager);
//	break;
//	}
//	case PBR_SCENE:
//	{
//	sceneManager = new PBRSceneManager();
//	renderer = new PBRRenderer(sceneManager);
//	break;
//	}
//	default:
//	sceneManager = new RayTracingSceneManager();
//	renderer = new RayTracingRenderer(sceneManager);
//	break;
//	}
//
//	sceneManager->InitializeObjects();
//
//	renderer->Initialize(window);
//	renderer->InitializeRender();
//
//	do
//	{
//	sceneManager->Update();
//	renderer->Render();
//	InputManager::GetInstance()->PollEvents();
//	glfwSwapBuffers(window);
//	}
//	while (InputManager::GetInstance()->IsKey(GLFW_KEY_ESCAPE) != GLFW_PRESS &&
//	!WindowManager::GetInstance()->WindowShouldClose());
//
//	renderer->TerminateRender();
//
//	delete renderer;
//	delete sceneManager;
//
//	WindowManager::GetInstance()->Terminate();
//
//
//
//	return 0;*/
//}
//
////3 channel
////albedo, microsurface, metallic
////refracted light(diffuse)
////depth and color would be very
////
////metallic(conductvie) reflect many lights and energy less lost
////dielectric(insulation) refract many lights lost many energy and wide
////
////refracted light�� ��ü�� ������ ���� ���̴�.
////���� albedo���� ��������.
////�ȿ� ���ڰ� ���� ��� refracted light�� ������ �پ���(metallic)
////�ȿ� ���ڰ� ���� ��� refracted light�� ������ ��������.
////
////reflected light�� �� ��ü�� ������ ������,
////�ֳĸ� refracted light�� ��ü���� ������ ��������� reflected light�� �ƴϱ� �����̴�.
////roughness channel�̳� glossiness channel�� ǥ���Ѵ�.
////roughness channel������ �Ͼ���ϼ��� ��ĥ�ٴ� ���̴�.
////smoothness