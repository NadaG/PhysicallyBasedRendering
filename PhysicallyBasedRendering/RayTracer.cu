#include "hip/hip_runtime.h"
#include "RayTracer.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm\glm.hpp>
#include <stdio.h>
#include <glm\gtc\matrix_transform.hpp>
#include <hip/hip_math_constants.h>
#include <math.h>
#include <thrust\device_vector.h>
#include <queue>

struct Ray
{
	glm::vec3 origin;
	glm::vec3 dir;
};

struct Sphere
{
	glm::vec3 origin;
	float radius;
};

const int WINDOW_HEIGHT = 1024;
const int WINDOW_WIDTH = 1024;

const int QUEUE_SIZE = 3;

__device__ bool RaySphereIntersect(Ray ray, Sphere sphere, float& dist)
{
	glm::vec3 s = ray.origin - sphere.origin;

	float a = dot(ray.dir, ray.dir);
	float bPrime = dot(s, ray.dir);
	float c = dot(s, s) - sphere.radius * sphere.radius;

	float D = bPrime * bPrime - a * c;
	if (D >= 0 && bPrime <= 0)
	{
		float t1 = (-bPrime + sqrt(D)) / a;
		float t2 = (-bPrime - sqrt(D)) / a;
		dist = t1 > t2 ? t2 : t1;
		return true;
	}
	else
		return false;
}

// back face culling�� ����Ǿ� ����
__device__ bool RayTriangleIntersect(Ray ray, Triangle triangle, float& dist)
{
	glm::vec3 v0v1 = triangle.v1 - triangle.v0;
	glm::vec3 v0v2 = triangle.v2 - triangle.v0;
	glm::vec3 pvec = glm::cross(ray.dir, v0v2);

	float det = dot(v0v1, pvec);

	float epsilon = 0.0001f;

	if (det < epsilon)
		return false;

	if (fabs(det) < epsilon)
		return false;

	float invDet = 1 / det;

	glm::vec3 tvec = ray.origin - triangle.v0;
	float u = glm::dot(tvec, pvec) * invDet;
	if (u < 0 || u > 1)
		return false;

	glm::vec3 qvec = cross(tvec, v0v1);
	float v = dot(ray.dir, qvec) * invDet;
	if (v < 0 || u + v > 1)
		return false;

	dist = dot(v0v2, qvec) * invDet;

	return true;
}

// ���� ����� triangle�� id�� ��ȯ�ϰ� �ش� �������� dist�� �����´�
__device__ int FindNearestTriangleIdx(Ray ray, Triangle* triangles, int triangleNum, float& dist)
{
	const float rayThreshold = 0.001f;
	float minDist = 9999999.0f;
	int minIdx = -1;
	float tmpDist;

	// �״�� dist�� �����ͼ� ����ϴϱ� �̻�����
	for (int i = 0; i < triangleNum; ++i)
	{
		// intersect �� ���
		if (RayTriangleIntersect(ray, triangles[i], tmpDist))
		{
			// �� ã�� ���, �ٽ� ã�� �ʱ�
			if (tmpDist > rayThreshold && tmpDist < minDist)
			{
				minDist = tmpDist;
				minIdx = i;
			}
		}
	}

	dist = minDist;
	return minIdx;
}

__device__ Ray GenerateCameraRay(int y, int x, glm::mat4 view)
{
	Ray ray;

	float NDCy = (y + 0.5f) / WINDOW_HEIGHT;
	float NDCx = (x + 0.5f) / WINDOW_WIDTH;

	float aspectRatio = WINDOW_WIDTH / WINDOW_HEIGHT;

	float fov = 45.0f;

	float xx = ((NDCx) * 2.0f - 1.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f) * aspectRatio;
	float yy = (1.0f - NDCy * 2.0f) * tan(fov * 0.5f * 3.141592653f / 180.0f);

	// -1 ~ 1
	ray.origin = glm::vec3(-view * glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
	ray.dir = normalize(vec3(view * vec4(glm::vec3(xx, yy, -1.0), 0.0f)));

	return ray;
}

__device__ void Enqueue(Ray* rayQueue, Ray ray, int& rear)
{
	rear = (rear + 1) % QUEUE_SIZE;
	rayQueue[rear] = ray;
}

__device__ void Dequeue(Ray* rayQueue, int& front)
{
	Ray ray = rayQueue[front];
	front = (front + 1) % QUEUE_SIZE;
}

__device__ Ray GetQueueFront(Ray* rayQueue, const int front)
{
	return rayQueue[(front + 1) % QUEUE_SIZE];
}

__device__ bool IsQueueFull(const int front, const int rear)
{
	return front == (rear + 1) % QUEUE_SIZE;
}

__device__ bool IsQueueEmpty(const int front, const int rear)
{
	return front == rear;
}

__device__ vec3 GenerateRayQueue(
	Ray ray,
	Ray* rayQueue,
	Triangle* triangles,
	int triangleNum,
	Light* lights,
	int lightNum,
	int depth)
{
	vec3 color = vec3(0.0f);
	int front = 0, rear = 0;

	// ù ��° ray�� node�� �ϴ� queue ����
	Enqueue(rayQueue, ray, rear);

	int nowDepth = 1;

	// �� 7 (1 + 2 + 4)���� ray�� ����
	for (int i = 1; i < depth; ++i)
	{
		int target = rear;

		while (!IsQueueEmpty(target, front))
		{
			Ray nowRay;
			nowRay = GetQueueFront(rayQueue, front);
			Dequeue(rayQueue, front);

			float distToTriangle;
			int nearestTriangleIdx = FindNearestTriangleIdx(nowRay, triangles, triangleNum, distToTriangle);
			if (nearestTriangleIdx == -1 || distToTriangle < 0.0f)
			{
				color += vec3(0.0f, 0.0f, 0.0f);
				continue;
			}

			Triangle nearestTriangle = triangles[nearestTriangleIdx];
			glm::vec3 N = normalize(nearestTriangle.normal);

			glm::vec3 col = glm::vec3(0, 0, 0);
			glm::vec3 hitPoint = nowRay.origin + nowRay.dir * distToTriangle;

			for (int k = 0; k < lightNum; k++)
			{
				Ray shadowRay;
				shadowRay.origin = hitPoint;
				shadowRay.dir = normalize(lights[k].pos - hitPoint);

				bool isLighted = true;
				float tmp;
				for (int k = 0; k < triangleNum; k++)
				{
					if (nearestTriangleIdx != k)
						if (RayTriangleIntersect(shadowRay, triangles[k], tmp))
							// ������ dir�� ��
							if (tmp > 0.0001f)
								isLighted = false;
				}

				if (!isLighted)
					continue;

				glm::vec3 L = glm::normalize(lights[k].pos - hitPoint);
				glm::vec3 V = -nowRay.dir;

				glm::vec3 ambient = glm::vec3(0.2, 0.2, 0.2) * lights[k].color;
				glm::vec3 diffuse = glm::vec3(0.3, 0.3, 0.3) * lights[k].color * glm::max(0.0f, dot(N, L));
				glm::vec3 specular = glm::vec3(0.1, 0.8, 0.2) * lights[k].color * glm::max(0.0f, pow(glm::max(dot(normalize(reflect(-L, N)), V), 0.0f), 16));

				col = ambient + diffuse + specular;
			}

			Ray reflectRay;
			reflectRay.origin = hitPoint;
			reflectRay.dir = normalize(reflect(nowRay.dir, N));

			Ray refractRay;
			refractRay.origin = hitPoint;
			refractRay.dir = normalize(refract(nowRay.dir, N, 1.2f));

			Enqueue(rayQueue, reflectRay, rear);
			Enqueue(rayQueue, refractRay, rear);

			float s = (float)glm::floor(glm::log((float)nowDepth) / glm::log(2.0f));
			color += col * pow(0.2f, s);

			color = col;
		}

		nowDepth++;
	}

	//int num = 1;
	// ������ ���� queue�� ������ �ϱ�
	while (!IsQueueEmpty(front, rear))
	{
		Ray nowRay;
		nowRay = GetQueueFront(rayQueue, front);
		Dequeue(rayQueue, front);

		float distToTriangle;
		int nearestTriangleIdx = FindNearestTriangleIdx(nowRay, triangles, triangleNum, distToTriangle);
		if (nearestTriangleIdx == -1 || distToTriangle < 0.0f)
		{
			continue;
		}

		Triangle nearestTriangle = triangles[nearestTriangleIdx];
		glm::vec3 N = normalize(nearestTriangle.normal);

		glm::vec3 col = glm::vec3(0, 0, 0);
		glm::vec3 hitPoint = nowRay.origin + nowRay.dir * distToTriangle;

		for (int k = 0; k < lightNum; k++)
		{
			Ray shadowRay;
			shadowRay.origin = hitPoint;
			shadowRay.dir = normalize(lights[k].pos - hitPoint);

			bool isLighted = true;
			float tmp;
			for (int k = 0; k < triangleNum; k++)
			{
				if (nearestTriangleIdx != k)
					if (RayTriangleIntersect(shadowRay, triangles[k], tmp))
						// ������ dir�� ��
						if (tmp > 0.0001f)
							isLighted = false;
			}

			if (!isLighted)
				continue;

			glm::vec3 L = glm::normalize(lights[k].pos - hitPoint);
			glm::vec3 V = -nowRay.dir;

			glm::vec3 ambient = glm::vec3(0.2, 0.2, 0.2) * lights[k].color;
			glm::vec3 diffuse = glm::vec3(0.3, 0.3, 0.3) * lights[k].color * glm::max(0.0f, dot(N, L));
			glm::vec3 specular = glm::vec3(0.1, 0.8, 0.2) * lights[k].color * glm::max(0.0f, pow(glm::max(dot(normalize(reflect(-L, N)), V), 0.0f), 16));

			col = ambient + diffuse + specular;
		}

		float s = (float)glm::floor(glm::log((float)nowDepth) / glm::log(2.0f));
		color += col * pow(0.2f, s);

		//////////////////////////////////////////////////////
		//Ray nowRay;

		//nowRay = GetQueueFront(rayQueue, front);
		//Dequeue(rayQueue, front);

		//float distToTriangle;
		//float minDistToTriangle = 99999999.0f;
		//glm::vec3 minColor = glm::vec3(0, 0, 0);

		//for (int i = 0; i < triangleNum; i++)
		//{
		//	// intersect �� ���
		//	// reflect ray�� refract ray ����
		//	if (RayTriangleIntersect(nowRay, triangles[i], distToTriangle))
		//	{
		//		if (distToTriangle < 0.0f) {
		//			continue;
		//		}

		//		// ���� �տ� �ִ� �ȼ��� �׸���
		//		if (distToTriangle < minDistToTriangle)
		//		{
		//			minDistToTriangle = distToTriangle;

		//			for (int j = 0; j < lightNum; j++)
		//			{
		//				glm::vec3 hitPoint = nowRay.origin + nowRay.dir * distToTriangle;

		//				Ray shadowRay;
		//				shadowRay.origin = hitPoint;
		//				shadowRay.dir = normalize(lights[j].pos - hitPoint);

		//				bool isLighted = true;
		//				float tmp;
		//				for (int k = 0; k < triangleNum; k++)
		//				{
		//					if (i != k)
		//						if (RayTriangleIntersect(shadowRay, triangles[k], tmp))
		//							// ������ dir�� ��
		//							if (tmp > 0.0f)
		//								isLighted = false;
		//				}

		//				if (!isLighted)
		//				{
		//					continue;
		//				}

		//				glm::vec3 L = glm::normalize(lights[j].pos - hitPoint);
		//				glm::vec3 N = normalize(triangles[i].normal);
		//				glm::vec3 V = -nowRay.dir;

		//				glm::vec3 ambient = glm::vec3(0.2, 0.2, 0.2) * lights[j].color;
		//				glm::vec3 diffuse = glm::vec3(0.3, 0.3, 0.3) * lights[j].color * glm::max(0.0f, dot(N, L));
		//				glm::vec3 specular = glm::vec3(0.1, 0.8, 0.2) * lights[j].color * glm::max(0.0f, pow(glm::max(dot(normalize(reflect(-L, N)), V), 0.0f), 16));

		//				glm::vec3 col = ambient + diffuse + specular;

		//				minColor = col;
		//			}
		//		}
		//	}
		//}

		//float s = (float)glm::floor(glm::log((float)nowDepth) / glm::log(2.0f));
		//color += minColor * 0.1f;
	}

	return color;
}

// TODO view matrix�� ������ ��
__global__ void RayTraceD(
	glm::vec4* data,
	glm::mat4 view,
	Triangle* triangles, int triangleNum,
	Light* lights, int lightNum)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	Ray ray = GenerateCameraRay(blockIdx.x, threadIdx.x, view);

	Ray rayQueue[3];

	vec3 color = GenerateRayQueue(ray, rayQueue, triangles, triangleNum, lights, lightNum, 2);

	data[x] = glm::vec4(color, 1.0f);
}

//void RayTrace(glm::vec4* data, glm::mat4 view, Triangle* triangles, int triangleNum)
void RayTrace(glm::vec4* data, glm::mat4 view, const vector<Triangle> &triangles, const vector<Light>& lights)
{
	thrust::device_vector<Triangle> t = triangles;
	thrust::device_vector<Light> l = lights;

	size_t size;
	hipDeviceSetLimit(hipLimitMallocHeapSize, 10000 * sizeof(float));
	hipDeviceGetLimit(&size, hipLimitMallocHeapSize);

	RayTraceD << <WINDOW_HEIGHT, WINDOW_WIDTH >> > (
		data,
		view,
		t.data().get(),
		t.size(),
		l.data().get(),
		l.size()
		);
}